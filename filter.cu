#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "filter.h"
#include "zone_map.h"
#include <iomanip>
#include <iostream>
#include <sstream>

struct cmp_functor_dict
{
	const unsigned long long* source;
	bool *dest;
	const unsigned int *pars;

	cmp_functor_dict(const unsigned long long int* _source, bool * _dest,  const unsigned int * _pars):
		source(_source), dest(_dest), pars(_pars) {}

	template <typename IndexType>
	__host__ __device__
	void operator()(const IndexType & i) {

		unsigned int idx = pars[0];
		unsigned int cmp = pars[1];
		unsigned int bits = ((unsigned int*)source)[1];
		unsigned int fit_count = ((unsigned int*)source)[0];
		unsigned int int_sz = 64;

		//find the source index
		unsigned int src_idx = i/fit_count;
		// find the exact location
		unsigned int src_loc = i%fit_count;
		//right shift the values
		unsigned int shifted = ((fit_count-src_loc)-1)*bits;
		unsigned long long int tmp = source[src_idx+2]  >> shifted;
		// set  the rest of bits to 0
		tmp	= tmp << (int_sz - bits);
		tmp	= tmp >> (int_sz - bits);
		//printf("COMP1 %llu %d \n", tmp, idx);
		if(cmp == 4) { // ==
			if(tmp == idx)
				dest[i] = 1;
			else
				dest[i] = 0;
		}
		else  { // !=
			if(tmp == idx)
				dest[i] = 0;
			else
				dest[i] = 1;
		};
	}
};


struct gpu_regex
{
	char  *source;
	char *pattern;
	bool * dest;
	const unsigned int *len;

	gpu_regex(char * _source,char * _pattern, bool * _dest,
	          const unsigned int * _len):
		source(_source), pattern(_pattern), dest(_dest), len(_len) {}

	template <typename IndexType>
	__host__ __device__
	void operator()(const IndexType & i) {

		bool star = 0;
		int j = 0;
		char* s;
		char* p;
		char* str = source + len[0]*i;
		char* pat = pattern;

loopStart:
		for (s = str, p = pat; j < len[0] && *s; ++s, ++p, ++j) {
			switch (*p) {
			case '?':
				if (*s == '.')
					goto starCheck;
				break;
			case '%':
				star = 1;
				str = s, pat = p;
				do {
					++pat;
				}
				while (*pat == '%');
				if (!*pat) {
					dest[i] = 1;
					return;
				}
				goto loopStart;
			default
					:
				if (*s != *p)
					goto starCheck;
				break;
			} /* endswitch */
		} /* endfor */
		while (*p == '%')
			++p;
		dest[i] = !*p;
		return;

starCheck:
		if (!star) {
			dest[i] = 0;
			return;
		};
		str++;
		j++;
		goto loopStart;
	}
};



bool* filter(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a,
             unsigned int segment)
{

	stack<string> exe_type;
	stack<string> exe_value;
	stack<int_type*> exe_vectors;
	stack<unsigned int> exe_precision;
	stack<int_type> exe_nums;
	stack<bool*> bool_vectors;
	string  s1, s2, s1_val, s2_val;
	int_type n1, n2, res;

	for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

		string ss = op_type.front();
		//cout << endl << ss << " " <<  op_nums.size() << " " << op_nums_precision.size() << endl;

		if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("FLOAT") == 0
		        || ss.compare("STRING") == 0 || ss.compare("FIELD") == 0) {


			if (ss.compare("NUMBER") == 0) {
				exe_nums.push(op_nums.front());
				op_nums.pop();
				exe_type.push(ss);
				exe_precision.push(op_nums_precision.front());
				op_nums_precision.pop();
			}
			else
				if (ss.compare("NAME") == 0 || ss.compare("STRING") == 0) {
					exe_value.push(op_value.front());
					op_value.pop();
					exe_type.push(ss);
				}
				else
					if(ss.compare("FIELD") == 0) {
						size_t pos1 = op_value.front().find_first_of(".", 0);
						string tbl = op_value.front().substr(0,pos1);
						string field = op_value.front().substr(pos1+1, string::npos);
						op_value.pop();
						CudaSet *b = varNames.find(tbl)->second;
						auto val = b->h_columns_int[field][0];
						exe_nums.push(val);
						exe_type.push("NUMBER");
					}
		}
		else {
			if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
				// get 2 values from the stack

				s1 = exe_type.top();
				exe_type.pop();
				s2 = exe_type.top();
				exe_type.pop();


				if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
					s1_val = exe_value.top();
					exe_value.pop();
					s2_val = exe_value.top();
					exe_value.pop();

					int_type* t = get_vec(a, s1_val, exe_vectors);
					exe_type.push("NAME");
					exe_value.push("");
					exe_precision.push(0);

				}
				else
					if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
						s2_val = exe_value.top();
						exe_value.pop();
						s1_val = exe_value.top();
						exe_value.pop();
						int_type* t = get_vec(a, s1_val, exe_vectors);
						exe_type.push("NAME");
						exe_value.push("");
						exe_precision.push(0);
					}
					else
						if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();

							int_type* t = get_vec(a, s1_val, exe_vectors);

							exe_type.push("NAME");
							exe_value.push("");
							exe_precision.push(0);
						}
						else
							if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
								s1_val = exe_value.top();
								exe_value.pop();
								s2_val = exe_value.top();
								exe_value.pop();

								int_type* t = get_vec(a, s2_val, exe_vectors);

								exe_type.push("NAME");
								exe_value.push("");
								exe_precision.push(0);
							}
							else
								if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
									n1 = exe_nums.top();
									exe_nums.pop();
									n2 = exe_nums.top();
									exe_nums.pop();

									auto p1 = exe_precision.top();
									exe_precision.pop();
									auto p2 = exe_precision.top();
									exe_precision.pop();
									auto pres = precision_func(p1, p2, ss);
									exe_precision.push(pres);
									if(p1)
										n1 = n1*(unsigned int)pow(10,p1);
									if(p2)
										n2 = n2*(unsigned int)pow(10,p2);

									if (ss.compare("ADD") == 0 )
										res = n1+n2;
									else
										if (ss.compare("MUL") == 0 )
											res = n1*n2;
										else
											if (ss.compare("DIV") == 0 )
												res = n1/n2;
											else
												res = n1-n2;

									thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
									thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

									exe_type.push("NAME");
									exe_value.push("");
									exe_vectors.push(thrust::raw_pointer_cast(p));
								}
								else
									if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
										s1_val = exe_value.top();
										exe_value.pop();
										n1 = exe_nums.top();
										exe_nums.pop();
										auto p1 = exe_precision.top();
										exe_precision.pop();
										auto p2 = get_decimals(a, s1_val, exe_precision);

										int_type* t = get_vec(a, s1_val, exe_vectors);
										auto pres = precision_func(p1, p2, ss);
										exe_precision.push(pres);
										exe_type.push("NAME");
										exe_value.push("");
										exe_vectors.push(a->op(t,n1,ss,1, p1, p2));

									}
									else
										if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
											n1 = exe_nums.top();
											exe_nums.pop();
											s2_val = exe_value.top();
											exe_value.pop();
											auto p1 = exe_precision.top();
											exe_precision.pop();
											auto p2 = get_decimals(a, s2_val, exe_precision);

											int_type* t = get_vec(a, s2_val, exe_vectors);
											auto pres = precision_func(p2, p1, ss);
											exe_precision.push(pres);
											exe_type.push("NAME");
											exe_value.push("");
											exe_vectors.push(a->op(t,n1,ss,0, p2, p1));

										}
										else
											if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
												s1_val = exe_value.top();
												exe_value.pop();
												s2_val = exe_value.top();
												exe_value.pop();

												if (a->type[s1_val] == 0) {
													int_type* t1 = get_vec(a, s1_val, exe_vectors);
													int_type* t = get_vec(a, s2_val, exe_vectors);
													auto p1 = get_decimals(a, s1_val, exe_precision);
													auto p2 = get_decimals(a, s2_val, exe_precision);
													auto pres = precision_func(p1, p2, ss);
													exe_precision.push(pres);
													exe_type.push("NAME");
													exe_value.push("");
													exe_vectors.push(a->op(t,t1,ss,0,p2,p1));
												}
											}
			}

			else
				if (ss.compare("CMP") == 0) {


					int_type cmp_type = op_nums.front();
					op_nums.pop();

					s1 = exe_type.top();
					exe_type.pop();
					s2 = exe_type.top();
					exe_type.pop();

					if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
						n1 = exe_nums.top();
						exe_nums.pop();
						n2 = exe_nums.top();
						exe_nums.pop();
						exe_type.push("NAME");
						exe_value.push("");
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto pres = precision_func(p1, p2, ss);
						exe_precision.push(pres);
						if(p1)
							n1 = n1*(unsigned int)pow(10,p1);
						if(p2)
							n2 = n2*(unsigned int)pow(10,p2);

						bool_vectors.push(a->compare(n1,n2,cmp_type));
					}
					else
						if ((s1.compare("STRING") == 0 && s2.compare("NAME") == 0) ||
						        (s1.compare("NAME") == 0 && s2.compare("STRING") == 0))
						{
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();


							if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
								s1.swap(s2);
								s1_val.swap(s2_val);
							};


							void* d_res, *d_v;
							if(cmp_type != 7)
								hipMalloc((void **) &d_res, a->mRecCount);
							else
								hipMalloc((void **) &d_res, a->hostRecCount);
							thrust::device_ptr<bool> dd_res((bool*)d_res);

							hipMalloc((void **) &d_v, 8);
							thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
							thrust::counting_iterator<unsigned int> begin(0);

							if(s2_val.find(".") != string::npos) { //bitmap index
								auto pos1 = s2_val.find_first_of(".");
								auto pos2 = s2_val.find_last_of(".");
								auto set = s2_val.substr(pos1+1, (pos2-pos1)-1);
								auto col = s2_val.substr(pos2+1);
								auto len = data_dict[set][col].col_length;

								while(s1_val.length() < len)
									s1_val = s1_val + '\0';

								auto s1_hash = MurmurHash64A(&s1_val[0], len, hash_seed)/2;
								if(a->idx_dictionary_int[s2_val].find(s1_hash) != a->idx_dictionary_int[s2_val].end()) {
									dd_v[0] = a->idx_dictionary_int[s2_val][s1_hash];
									dd_v[1] = (unsigned int)cmp_type;
									cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
									thrust::for_each(begin, begin + a->mRecCount, ff);
								}
								else {
									hipMemset(d_res,0,a->mRecCount);
								}
							}
							else {

								auto s = a->string_map[s2_val];
								auto pos = s.find_first_of(".");
								auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

								dd_v[0] = len;
								dd_v[1] = (unsigned int)s1_val.length() + 1;

								if(cmp_type != 7) {
									thrust::device_vector<unsigned long long int> vv(1);
									while(s1_val.length() < len) {
										s1_val = s1_val + '\0';
									};

									vv[0] = MurmurHash64A(&s1_val[0], s1_val.length(), hash_seed)/2;

									string f1 = a->load_file_name + "." + s2_val + "." + to_string(segment) + ".hash";
									FILE* f = fopen(f1.c_str(), "rb" );
									unsigned long long int* buff = new unsigned long long int[a->mRecCount];
									unsigned int cnt;
									fread(&cnt, 4, 1, f);
									fread(buff, a->mRecCount*8, 1, f);
									fclose(f);
									thrust::device_vector<unsigned long long int> vals(a->mRecCount);
									thrust::copy(buff, buff+a->mRecCount, vals.begin());
									if(cmp_type == 4) //==
										thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::equal_to<unsigned long long int>());
									else
										if(cmp_type == 3) //!=
											thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::not_equal_to<unsigned long long int>());
									delete [] buff;

								}
								else {
									if(a->map_like.find(s2_val) == a->map_like.end()) {

										void* d_str;
										hipMalloc((void **) &d_str, len);
										hipMemset(d_str,0,len);
										hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);

										string f1 = a->load_file_name + "." + s2_val;
										FILE* f = fopen(f1.c_str(), "rb" );
										fseek(f, 0, SEEK_END);
										long fileSize = ftell(f);
										fseek(f, 0, SEEK_SET);

										unsigned int pieces = 1;
										if(fileSize > getFreeMem()/2)
											pieces = fileSize /(getFreeMem()/2) + 1;
										auto piece_sz = fileSize/pieces;
										ldiv_t ldivresult = ldiv(fileSize/pieces, len);
										if(ldivresult.rem != 0)
											piece_sz = fileSize/pieces + (len - ldivresult.rem);
										thrust::device_vector<char> dev(piece_sz);
										char* buff = new char[piece_sz];
										a->map_res[s2_val] = thrust::device_vector<unsigned int>();
										for(auto i = 0; i < pieces; i++) {

											if(i == pieces-1)
												piece_sz = fileSize - piece_sz*i;
											fread(buff, piece_sz, 1, f);
											hipMemcpy( thrust::raw_pointer_cast(dev.data()), (void*)buff, piece_sz, hipMemcpyHostToDevice);

											gpu_regex ff(thrust::raw_pointer_cast(dev.data()), (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
											thrust::for_each(begin, begin + piece_sz/len, ff);

											auto cnt = thrust::count(dd_res, dd_res + piece_sz/len, 1);
											auto offset = a->map_res[s2_val].size();
											a->map_res[s2_val].resize(a->map_res[s2_val].size() + cnt);
											thrust::copy_if(thrust::make_counting_iterator((unsigned int)(i*(piece_sz/len))), thrust::make_counting_iterator((unsigned int)((i+1)*(piece_sz/len))),
											                dd_res, a->map_res[s2_val].begin() + offset, thrust::identity<bool>());
										};

										fclose(f);
										delete [] buff;
										hipFree(d_str);
										thrust::sort(a->map_res[s2_val].begin(), a->map_res[s2_val].end());
										a->map_like[s2_val] = 1;


									};
									// now lets calc the current segments's matches
									hipMemset(d_res, 0, a->hostRecCount);
									binary_search(a->map_res[s2_val].begin(),a->map_res[s2_val].end(), a->d_columns_int[s2_val].begin(), a->d_columns_int[s2_val].end(), dd_res);
								};
							};

							hipFree(d_v);
							exe_type.push("NAME");
							bool_vectors.push((bool*)d_res);
						}

						else
							if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
								n1 = exe_nums.top();
								exe_nums.pop();
								s1_val = exe_value.top();
								exe_value.pop();

								if(s1_val.find(".") != string::npos) { //bitmap index
									void* d_v, *d_res;
									hipMalloc((void **) &d_v, 8);
									thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
									hipMalloc((void **) &d_res, a->mRecCount);
									if(a->idx_dictionary_int[s1_val].find(n1) != a->idx_dictionary_int[s1_val].end()) {
										dd_v[0] = a->idx_dictionary_int[s1_val][n1];
										dd_v[1] = (unsigned int)cmp_type;
										thrust::counting_iterator<unsigned int> begin(0);
										cmp_functor_dict ff(idx_vals[s1_val], (bool*)d_res, (unsigned int*)d_v);
										thrust::for_each(begin, begin + a->mRecCount, ff);
									}
									else {
										hipMemset(d_res,0,a->mRecCount);
									};
									exe_type.push("NAME");
									bool_vectors.push((bool*)d_res);
									hipFree(d_v);
								}
								else {
									int_type* t = get_vec(a, s1_val, exe_vectors);
									thrust::device_ptr<int_type> bp((int_type*)t);
									auto p2 = exe_precision.top();
									exe_precision.pop();
									auto p1 = get_decimals(a, s1_val, exe_precision);
									auto pres = std::max(p1, p2);
									exe_precision.push(pres);

									exe_type.push("NAME");
									bool_vectors.push(a->compare(t,n1,cmp_type, pres-p1, pres-p2));
								};
							}
							else
								if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
									cmp_type = reverse_op(cmp_type);
									n1 = exe_nums.top();
									exe_nums.pop();
									s2_val = exe_value.top();
									exe_value.pop();

									if(s2_val.find(".") != string::npos) { //bitmap index
										void* d_v, *d_res;
										hipMalloc((void **) &d_v, 8);
										thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
										hipMalloc((void **) &d_res, a->mRecCount);

										if(a->idx_dictionary_int[s2_val].find(n1) != a->idx_dictionary_int[s2_val].end()) {

											dd_v[0] = a->idx_dictionary_int[s2_val][n1];
											dd_v[1] = (unsigned int)cmp_type;

											thrust::counting_iterator<unsigned int> begin(0);
											cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
											thrust::for_each(begin, begin + a->mRecCount, ff);
										}
										else {
											hipMemset(d_res,0,a->mRecCount);
										};
										exe_type.push("NAME");
										bool_vectors.push((bool*)d_res);
										hipFree(d_v);
									}
									else {
										int_type* t = get_vec(a, s2_val, exe_vectors);
										auto p2 = exe_precision.top();
										exe_precision.pop();
										auto p1 = get_decimals(a, s2_val, exe_precision);
										auto pres = std::max(p1, p2);
										exe_precision.push(pres);
										exe_type.push("NAME");
										bool_vectors.push(a->compare(t,n1,cmp_type, p1, p2));
									};
								}

								else
									if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
										s1_val = exe_value.top();
										exe_value.pop();
										s2_val = exe_value.top();
										exe_value.pop();
										exe_type.push("NAME");

										int_type* t = get_vec(a, s1_val, exe_vectors);
										int_type* t1 = get_vec(a, s2_val, exe_vectors);
										auto p1 = get_decimals(a, s1_val, exe_precision);
										auto p2 = get_decimals(a, s2_val, exe_precision);
										auto pres = max(p1, p2);
										exe_precision.push(pres);
										bool_vectors.push(a->compare(t1,t,cmp_type, p2, p1));
									}
				}

				else
					if (ss.compare("AND") == 0) {
						bool* s3 = bool_vectors.top();
						bool_vectors.pop();
						bool* s2 = bool_vectors.top();
						bool_vectors.pop();
						exe_type.push("NAME");
						bool_vectors.push(a->logical_and(s2,s3));
					}
					else
						if (ss.compare("OR") == 0) {
							bool* s3 = bool_vectors.top();
							bool_vectors.pop();
							bool* s2 = bool_vectors.top();
							bool_vectors.pop();
							exe_type.push("NAME");
							bool_vectors.push(a->logical_or(s2,s3));
						}
						else {
							cout << "found nothing " << endl;
						}
		};
	};

	return bool_vectors.top();
}