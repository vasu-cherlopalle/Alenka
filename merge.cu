#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "merge.h"

using namespace std;

void process_error(int severity, string err);	// this should probably live in a utils header file


#if defined(_MSC_VER)
#define BIG_CONSTANT(x) (x)
// Other compilers
#else   // defined(_MSC_VER)
#define BIG_CONSTANT(x) (x##LLU)
#endif // !defined(_MSC_VER)

unsigned int hash_seed;

struct float_avg
{
	__host__  float_type operator()(const float_type &lhs, const int_type &rhs) const {
		return lhs/rhs;
	}
};

struct float_avg1
{
	__host__  float_type operator()(const int_type &lhs, const int_type &rhs) const {
		return ((float_type)lhs)/rhs;
	}
};

struct div100
{
	__host__  int_type operator()(const int_type &lhs, const int_type &rhs) const {
		return (lhs*100)/rhs;
	}
};

thrust::host_vector<unsigned long long int> h_merge;

using namespace std;
using namespace thrust::placeholders;


void create_c(CudaSet* c, CudaSet* b)
{
	c->not_compressed = 1;
	c->segCount = 1;
	c->columnNames = b->columnNames;
	h_merge.clear();
	c->cols = b->cols;
	c->type = b->type;
	c->decimal = b->decimal;
	c->decimal_zeroes = b->decimal_zeroes;
	c->grp_type = b->grp_type;
	c->ts_cols = b->ts_cols;

	for(unsigned int i=0; i < b->columnNames.size(); i++) {
		if (b->type[b->columnNames[i]] == 0) {
			c->h_columns_int[b->columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
			c->d_columns_int[b->columnNames[i]] = thrust::device_vector<int_type>();
			if(b->string_map.find(b->columnNames[i]) != b->string_map.end()) {
				c->string_map[b->columnNames[i]] = b->string_map[b->columnNames[i]];
			};
		}
		else
			if (b->type[b->columnNames[i]] == 1) {
				c->h_columns_float[b->columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
				c->d_columns_float[b->columnNames[i]] = thrust::device_vector<float_type>();
			}
			else {
				c->h_columns_char[b->columnNames[i]] = nullptr;
				c->d_columns_char[b->columnNames[i]] = nullptr;
				c->char_size[b->columnNames[i]] = b->char_size[b->columnNames[i]];
			};
	};
}

void add(CudaSet* c, CudaSet* b, queue<string> op_v3, map<string,string> aliases,
         vector<thrust::device_vector<int_type> >& distinct_tmp, vector<thrust::device_vector<int_type> >& distinct_val,
         vector<thrust::device_vector<int_type> >& distinct_hash, CudaSet* a)
{

	if (c->columnNames.empty()) {
		// create d_columns and h_columns
		create_c(c,b);
	}

	size_t cycle_sz = op_v3.size();

	vector<string> opv;
	for(unsigned int z = 0; z < cycle_sz; z++) {
		if(std::find(b->columnNames.begin(), b->columnNames.end(), aliases[op_v3.front()]) == b->columnNames.end()) { //sanity check
			cout << "Syntax error: alias " << op_v3.front() << endl;
			exit(0);
		};
		opv.push_back(aliases[op_v3.front()]);
		op_v3.pop();
	};


	// create hashes of groupby columns
	unsigned long long int* hashes = new unsigned long long int[b->mRecCount];
	unsigned long long int* sum = new unsigned long long int[cycle_sz*b->mRecCount];

	for(unsigned int z = 0; z < cycle_sz; z++) {
		// b->CopyColumnToHost(opv[z]);
		if(b->type[opv[z]] != 1) {  //int or string
			for(int i = 0; i < b->mRecCount; i++) {
				//memcpy(&sum[i*cycle_sz + z], &b->h_columns_int[opv[z]][i], 8);
				sum[i*cycle_sz + z] = b->h_columns_int[opv[z]][i];
				//cout << "CPY to " << i*cycle_sz + z << " " << opv[z] << " " << b->h_columns_int[opv[z]][i] <<   endl;
				//cout << "SET " << sum[i*cycle_sz + z] << endl;
			};
		}
		else {  //float
			for(int i = 0; i < b->mRecCount; i++) {
				memcpy(&sum[i*cycle_sz + z], &b->h_columns_float[opv[z]][i], 8);
			};
		};
	};

	for(int i = 0; i < b->mRecCount; i++) {
		hashes[i] = MurmurHash64A(&sum[i*cycle_sz], 8*cycle_sz, hash_seed);
		//cout << "hash " << hashes[i] << " " << i*cycle_sz << " "  << sum[i*cycle_sz] << " " << sum[i*cycle_sz + 1] << endl;
	};

	delete [] sum;
	thrust::device_vector<unsigned long long int> d_hashes(b->mRecCount);
	thrust::device_vector<unsigned int> v(b->mRecCount);
	thrust::sequence(v.begin(), v.end(), 0, 1);
	thrust::copy(hashes, hashes+b->mRecCount, d_hashes.begin());

	// sort the results by hash
	thrust::sort_by_key(d_hashes.begin(), d_hashes.end(), v.begin());

	void* d_tmp;
	CUDA_SAFE_CALL(hipMalloc((void **) &d_tmp, b->mRecCount*int_size));

	for(unsigned int i = 0; i < b->columnNames.size(); i++) {

		if(b->type[b->columnNames[i]] == 0 || b->type[b->columnNames[i]] == 2) {
			thrust::device_ptr<int_type> d_tmp_int((int_type*)d_tmp);
			thrust::gather(v.begin(), v.end(), b->d_columns_int[b->columnNames[i]].begin(), d_tmp_int);
			thrust::copy(d_tmp_int, d_tmp_int + b->mRecCount, b->h_columns_int[b->columnNames[i]].begin());
		}
		else
			if(b->type[b->columnNames[i]] == 1) {
				thrust::device_ptr<float_type> d_tmp_float((float_type*)d_tmp);
				thrust::gather(v.begin(), v.end(), b->d_columns_float[b->columnNames[i]].begin(), d_tmp_float);
				thrust::copy(d_tmp_float, d_tmp_float + b->mRecCount, b->h_columns_float[b->columnNames[i]].begin());
			}
	};
	hipFree(d_tmp);

	thrust::host_vector<unsigned long long int> hh = d_hashes;
	char* tmp = new char[max_char(b)*(c->mRecCount + b->mRecCount)];
	c->resize(b->mRecCount);

	//lets merge every column

	for(unsigned int i = 0; i < b->columnNames.size(); i++) {

		if(b->type[b->columnNames[i]] != 1) {

			thrust::merge_by_key(h_merge.begin(), h_merge.end(),
			                     hh.begin(), hh.end(),
			                     c->h_columns_int[c->columnNames[i]].begin(), b->h_columns_int[b->columnNames[i]].begin(),
			                     thrust::make_discard_iterator(), (int_type*)tmp);
			memcpy(thrust::raw_pointer_cast(c->h_columns_int[c->columnNames[i]].data()), (int_type*)tmp, (h_merge.size() + b->mRecCount)*int_size);
		}
		else {
			thrust::merge_by_key(h_merge.begin(), h_merge.end(),
			                     hh.begin(), hh.end(),
			                     c->h_columns_float[c->columnNames[i]].begin(), b->h_columns_float[b->columnNames[i]].begin(),
			                     thrust::make_discard_iterator(), (float_type*)tmp);
			memcpy(thrust::raw_pointer_cast(c->h_columns_float[c->columnNames[i]].data()), (float_type*)tmp, (h_merge.size() + b->mRecCount)*float_size);
		}
	};


	//merge the keys
	thrust::merge(h_merge.begin(), h_merge.end(),
	              hh.begin(), hh.end(), (unsigned long long int*)tmp);

	size_t cpy_sz = h_merge.size() + b->mRecCount;
	h_merge.resize(h_merge.size() + b->mRecCount);
	thrust::copy((unsigned long long int*)tmp, (unsigned long long int*)tmp + cpy_sz, h_merge.begin());

	delete [] tmp;
	delete [] hashes;

	//cout << endl << "end b and c " << b->mRecCount << " " << c->mRecCount << endl;
	//for(int i = 0; i < h_merge.size();i++)
	//cout << "H " << h_merge[i] << endl;

	/*   bool dis_exists = 0;
	    for(unsigned int j=0; j < c->mColumnCount; j++) {
	        if (c->grp_type[j] == 6)
	            dis_exists = 1;
	    };

	    if (dis_exists) {
	        bool grp_scanned = 0;
	        thrust::device_ptr<bool> d_di(a->grp);
	        thrust::device_ptr<unsigned int> d_dii = thrust::device_malloc<unsigned int>(a->mRecCount);
	        thrust::identity<bool> op;
	        thrust::transform(d_di, d_di+a->mRecCount, d_dii, op);

	        thrust::device_ptr<int_type> tmp = thrust::device_malloc<int_type>(a->mRecCount);

	        unsigned int dist_count = 0;

	        for(unsigned int j=0; j < c->mColumnCount; j++) {

	            if (c->grp_type[j] == 6) {

	                if(!grp_scanned) {

	                    d_dii[a->mRecCount-1] = 0;
	                    thrust::inclusive_scan(d_dii, d_dii + a->mRecCount, d_dii);
	                    thrust::gather(d_dii, d_dii + a->mRecCount, hashes.begin(), tmp);	// now hashes are in tmp
	                    grp_scanned = 1;
	                };
	                unsigned int offset = distinct_val[dist_count].size();

	                distinct_val[dist_count].resize(distinct_val[dist_count].size() + a->mRecCount);
	                distinct_hash[dist_count].resize(distinct_hash[dist_count].size() + a->mRecCount);

	                thrust::copy(distinct_tmp[dist_count].begin(), distinct_tmp[dist_count].begin() + a->mRecCount, distinct_val[dist_count].begin() + offset);
	                thrust::copy(tmp, tmp + a->mRecCount, distinct_hash[dist_count].begin() + offset);

	                thrust::stable_sort_by_key(distinct_val[dist_count].begin(), distinct_val[dist_count].end(), distinct_hash[dist_count].begin());
	                thrust::stable_sort_by_key(distinct_hash[dist_count].begin(), distinct_hash[dist_count].end(), distinct_val[dist_count].begin());

	                ZipIterator new_last = thrust::unique(thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].begin(), distinct_val[dist_count].begin())),
	                                                      thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].end(), distinct_val[dist_count].end())));

	                IteratorTuple t = new_last.get_iterator_tuple();
	                distinct_val[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());
	                distinct_hash[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());

	                dist_count++;

	            };
	        };
	        thrust::device_free(tmp);
	        thrust::device_free(d_dii);
	    };
		*/


}


void count_simple(CudaSet* c)
{
	int_type count;

	for(unsigned int i = 0; i < c->columnNames.size(); i++) {
		if(c->grp_type[c->columnNames[i]] == 0) { // COUNT
			count = thrust::reduce(c->h_columns_int[c->columnNames[i]].begin(), c->h_columns_int[c->columnNames[i]].begin() + c->mRecCount);
			c->h_columns_int[c->columnNames[i]][0] = count;
		};
	};


	if (c->mRecCount != 0) {

		for(unsigned int k = 0; k < c->columnNames.size(); k++) {
			if(c->grp_type[c->columnNames[k]] == 1) {   // AVG
				if(c->type[c->columnNames[k]] == 0 || c->type[c->columnNames[k]] == 2) {
					int_type sum  = thrust::reduce(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount);
					c->h_columns_int[c->columnNames[k]][0] = sum/count;
				}
				if(c->type[c->columnNames[k]] == 1) {
					float_type sum  = thrust::reduce(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount);
					c->h_columns_float[c->columnNames[k]][0] = sum/count;
				};
			}
			else
				if(c->grp_type[c->columnNames[k]] == 2) {   // SUM
					if(c->type[c->columnNames[k]] == 0 || c->type[c->columnNames[k]] == 2) {
						int_type sum  = thrust::reduce(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount);
						c->h_columns_int[c->columnNames[k]][0] = sum;
					}
					if(c->type[c->columnNames[k]] == 1) {
						float_type sum  = thrust::reduce(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount);
						c->h_columns_float[c->columnNames[k]][0] = sum;
					};

				}
		};
	}
	c->mRecCount = 1;
};


void count_avg(CudaSet* c,  vector<thrust::device_vector<int_type> >& distinct_hash)
{
	string countstr;
	thrust::equal_to<unsigned long long int> binary_pred;
	thrust::maximum<unsigned long long int> binary_op_max;
	thrust::minimum<unsigned long long int> binary_op_min;

	for(unsigned int i = 0; i < c->columnNames.size(); i++) {
		if(c->grp_type[c->columnNames[i]] == 0) { // COUNT
			countstr = c->columnNames[i];
			break;
		};
	};


	thrust::host_vector<bool> grp;
	size_t res_count;

	if(h_merge.size()) {
		grp.resize(h_merge.size());
		thrust::adjacent_difference(h_merge.begin(), h_merge.end(), grp.begin());
		res_count = h_merge.size() - thrust::count(grp.begin(), grp.end(), 0);
	};


	if (c->mRecCount != 0) {

		//unsigned int dis_count = 0;
		if (h_merge.size()) {
			int_type* tmp =  new int_type[res_count];
			for(unsigned int k = 0; k < c->columnNames.size(); k++)	{

				if(c->grp_type[c->columnNames[k]] <= 2) { //sum || avg || count
					if (c->type[c->columnNames[k]] == 0) { // int
						// check for overflow
						// convert to double, reduce, check if larger than max 64 bit int

						float_type* tmp1 =  new float_type[c->mRecCount];
						float_type* tmp_res = new float_type[res_count];

						for(int z = 0; z < c->mRecCount ; z++)
							tmp1[z] = (float_type)(c->h_columns_int[c->columnNames[k]][z]);

						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), tmp1,
						                      thrust::make_discard_iterator(), tmp_res);

						double max_overflow = 0;
						for(int z = 0; z < res_count; z++) {
							if (tmp_res[z] > 9223372036854775807.0) {
								if(tmp_res[z] - 9223372036854775807.0 > max_overflow)
									max_overflow = tmp_res[z];
							};
						};
						if(max_overflow) {
							unsigned pw = ceil(log10(max_overflow/9223372036854775807.0));
							thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].end(), thrust::make_constant_iterator((int_type)pow(10, pw)), c->h_columns_int[c->columnNames[k]].begin(), thrust::divides<int_type>());
							c->decimal_zeroes[c->columnNames[k]] = c->decimal_zeroes[c->columnNames[k]] - pw;
						};

						delete [] tmp1;
						delete [] tmp_res;

						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							float_type* tmp1 =  new float_type[res_count];
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), tmp1);
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::copy(tmp1, tmp1 + res_count, c->h_columns_float[c->columnNames[k]].begin());
							delete [] tmp1;
						};
				}
				if(c->grp_type[c->columnNames[k]] == 4) { //min
					if (c->type[c->columnNames[k]] == 0 ) { // int
						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_min);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_min);
						};
				}
				if(c->grp_type[c->columnNames[k]] == 5) { //max
					if (c->type[c->columnNames[k]] == 0 ) { // int
						int_type* tmp =  new int_type[res_count];
						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_max);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
						delete [] tmp;
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_max);
						};
				}
				else
					if(c->grp_type[c->columnNames[k]] == 3) { //no group function
						if (c->type[c->columnNames[k]] == 0 || c->type[c->columnNames[k]] == 2) { // int
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_max);
							c->h_columns_int[c->columnNames[k]].resize(res_count);
							thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
						}
						else
							if (c->type[c->columnNames[k]] == 1 ) { // float
								c->h_columns_float[c->columnNames[k]].resize(res_count);
								thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
								                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_max);
							}
					};
			};
			c->mRecCount = res_count;
			delete [] tmp;
		};

		for(unsigned int k = 0; k < c->columnNames.size(); k++)	{
			if(c->grp_type[c->columnNames[k]] == 1) {   // AVG

				if (c->type[c->columnNames[k]] == 0 ) { // int

					if(c->decimal_zeroes[c->columnNames[k]] <= 2) {
						thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount,
						                  c->h_columns_int[countstr].begin(), c->h_columns_int[c->columnNames[k]].begin(), div100());
						c->decimal_zeroes[c->columnNames[k]] = c->decimal_zeroes[c->columnNames[k]] + 2;
					}
					else {
						thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount,
						                  c->h_columns_int[countstr].begin(), c->h_columns_int[c->columnNames[k]].begin(), thrust::divides<int_type>());
					};
					c->grp_type[c->columnNames[k]] = 3;
				}
				else {              // float
					thrust::transform(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount,
					                  c->h_columns_int[countstr].begin(), c->h_columns_float[c->columnNames[k]].begin(), float_avg());
				};
			}
			else
				if(c->grp_type[c->columnNames[k]] == 6) {
					/*   unsigned int res_count = 0;

					   thrust::host_vector<int_type> h_hash = distinct_hash[dis_count];
					   int_type curr_val = h_hash[0];
					   unsigned int cycle_sz = h_hash.size();

					   for(unsigned int i = 0; i < cycle_sz; i++) {
					       if (h_hash[i] == curr_val) {
					           res_count++;
					           if(i == cycle_sz-1) {
					               c->h_columns_int[c->columnNames[k]][mymap[h_hash[i]]] = res_count;
					           };
					       }
					       else {
					           unsigned int idx = mymap[h_hash[i-1]];
					           c->h_columns_int[c->columnNames[k]][idx] = res_count;
					           curr_val = h_hash[i];
					           res_count = 1;
					       };
					   };
					   dis_count++;*/
				}
				else
					if(c->grp_type[c->columnNames[k]] == 2) {

					};
		};

	};

	c->segCount = 1;
	c->maxRecs = c->mRecCount;
};

