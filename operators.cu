#include "hip/hip_runtime.h"
#include "operators.h"
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/set_operations.h>
#include "moderngpu/src/moderngpu/kernel_join.hxx"


struct is_even
   {
     __host__ __device__
     bool operator()(const int &x)
     {
       return (x % 2) == 0;
     }
   };

using namespace mgpu;
using namespace std;
using namespace thrust::placeholders;


size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
unsigned int distinct_cnt = 0;
unsigned int join_col_cnt = 0;
unsigned int join_tab_cnt = 0;
unsigned int tab_cnt = 0;
queue<string> op_join;
queue<char> join_type;
queue<char> join_eq_type;
unsigned int partition_count;
map<string,unsigned int> mystat;
map<unsigned int, unsigned int> join_and_cnt;
map<string, map<string, bool> > used_vars;
bool save_dict = 0;

thrust::device_vector<unsigned char> scratch;
map<string, string> filter_var; 
thrust::device_vector<int> ranj;
unsigned long long int currtime;
standard_context_t context;	


void check_used_vars()
{
    for (auto it=data_dict.begin() ; it != data_dict.end(); ++it ) {
        auto s = (*it).second;
        auto vars(op_value);
        while(!vars.empty()) {
            if(s.count(vars.front()) != 0) {
                used_vars[(*it).first][vars.front()] = 1;
            };
            vars.pop();
        }
    };
}


void emit_name(const char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

void emit_limit(const int val)
{
    op_nums.push(val);
}


void emit_string(const char *str)
{   // remove the float_type quotes
	if(str[0] == '"') {
		string sss(str,1, strlen(str)-2);		
		op_value.push(sss);
	}
	else {	
		string sss(str);		
		op_value.push(sss);		
	};
	op_type.push("STRING");
}

void emit_string_grp(const char *str, const char *str_grp) 
{
	emit_string(str);
	grp_val = str_grp;	
};

void emit_fieldname(const char* name1, const char* name2)
{
    string s1(name1);
    string s2(name2);
    op_type.push("FIELD");
    op_value.push(s1 + "." + s2);
};

void emit_number(const int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);
	op_nums_precision.push(0);
}

void emit_float(const float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

void emit_decimal(const char* str)
{
    op_type.push("NUMBER");
    string s1(str);
	unsigned int precision;
	auto pos = s1.find(".");
	if(pos == std::string::npos)
		precision = 0;
	else {
		precision = (s1.length() - pos) -1;
		s1.erase(pos,1);
	};	
	op_nums.push(stoi(s1));
    op_nums_precision.push(precision);
}

void emit_mul()
{
    op_type.push("MUL");
}

void emit_add()
{
    op_type.push("ADD");
}

void emit_div()
{
    op_type.push("DIV");
}

unsigned int misses = 0;

void emit_and()
{
    op_type.push("AND");
    join_col_cnt++;
}

void emit_eq()
{
    op_type.push("JOIN");
	join_eq_type.push('E');
    if(misses == 0) {
        join_and_cnt[tab_cnt] = join_col_cnt;
        misses = join_col_cnt;
        join_col_cnt = 0;
        tab_cnt++;
    }
    else {
        misses--;
    }
}

void emit_neq()
{
    op_type.push("JOIN");
	join_eq_type.push('N');
    if(misses == 0) {
        join_and_cnt[tab_cnt] = join_col_cnt;
        misses = join_col_cnt;
        join_col_cnt = 0;
        tab_cnt++;
    }
    else {
        misses--;
    }
}


void emit_distinct()
{
    op_type.push("DISTINCT");
    distinct_cnt++;
}

void emit_year()
{
    op_type.push("YEAR");
}

void emit_month()
{
    op_type.push("MONTH");
}


void emit_day()
{
    op_type.push("DAY");
}

void emit_cast()
{
    op_type.push("CAST");
}


void emit_or()
{
    op_type.push("OR");
}


void emit_minus()
{
    op_type.push("MINUS");
}

void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}

void emit(const char *s, ...)
{
}

void emit_var(const char *s, const int c, const char *f, const char* ref, const char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
}

void emit_var_asc(const char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

void emit_var_desc(const char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}

void emit_sort(const char *s, const int p)
{
    op_sort.push(s);
    partition_count = p;
}

void emit_presort(const char *s)
{
    op_presort.push(s);
}


void emit_varchar(const char *s, const int c, const char *f, const int d, const char *ref, const char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
}

void emit_vardecimal(const char *s, const int c, const char *f, const int scale, const int precision)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(precision);
    cols.push(c);	
}

void emit_sel_name(const char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

void emit_count()
{
    op_type.push("COUNT");
}

void emit_sum()
{
    op_type.push("SUM");
}


void emit_average()
{
    op_type.push("AVG");
}

void emit_min()
{
    op_type.push("MIN");
}

void emit_max()
{
    op_type.push("MAX");
}

void emit_join_tab(const char *s, const char tp)
{
    op_join.push(s);
    join_tab_cnt++;
    join_type.push(tp);
};


void order_inplace_host(CudaSet* a, stack<string> exe_type, set<string> field_names,  bool update_str)
{
    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    char* temp = new char[a->mRecCount*max_char(a)];
    stack<string> exe_type1(exe_type), exe_value;

    while(!exe_type1.empty()) {
        exe_value.push("ASC");
        exe_type1.pop();
    };

    // sort on host

    for(;!exe_type.empty(); exe_type.pop(),exe_value.pop()) {
        if (a->type[exe_type.top()] != 1)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else 
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
    };

	for (auto it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] != 1) {
            thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_int[*it].data(), (int_type*)temp);
            thrust::copy((int_type*)temp, (int_type*)temp + a->mRecCount, a->h_columns_int[*it].data());
        }
        else  {
            thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_float[*it].data(), (float_type*)temp);
            thrust::copy((float_type*)temp, (float_type*)temp + a->mRecCount, a->h_columns_float[*it].data());
        }
    };

    delete [] temp;
    delete [] permutation;
}


void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{
	if(scratch.size() < a->mRecCount*4)
		scratch.resize(a->mRecCount*4);
	thrust::device_ptr<unsigned int> permutation((unsigned int*)thrust::raw_pointer_cast(scratch.data()));	
    thrust::sequence(permutation, permutation+a->mRecCount,0,1);
    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
	
	if(a->grp.size() < a->mRecCount*8)
		a->grp.resize(a->mRecCount*8);
 	unsigned int bits;
	
    for(; !exe_type.empty(); exe_type.pop()) {	
	
		if(cpy_bits.empty())
			bits = 0;
		else	
			bits = cpy_bits[exe_type.top()];			

        if (a->type[exe_type.top()] != 1) {
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, a->mRecCount, "ASC", (int_type*)thrust::raw_pointer_cast(a->grp.data()), bits);
		}	
        else
            update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, a->mRecCount,"ASC", (float_type*)thrust::raw_pointer_cast(a->grp.data()), bits);			
    };	
	
    for (auto it=field_names.begin(); it!=field_names.end(); ++it) {
		if(cpy_bits.empty())
			bits = 0;
		else	
			bits = cpy_bits[*it];
			
        if (a->type[*it] != 1) {
            apply_permutation(a->d_columns_int[*it], raw_ptr, a->mRecCount, (int_type*)thrust::raw_pointer_cast(a->grp.data()), bits);		
        }
        else {
            apply_permutation(a->d_columns_float[*it], raw_ptr, a->mRecCount, (float_type*)thrust::raw_pointer_cast(a->grp.data()), bits);			
		};				
    };
}

bool check_star_join(const string j1)
{
    auto op_vals(op_value);

    for(auto i=0; i < sel_count; i++) {
        op_vals.pop();
        op_vals.pop();
    };

    if(join_tab_cnt > 0) {

        while(op_vals.size()) {
            if (std::find(varNames[j1]->columnNames.begin(), varNames[j1]->columnNames.end(), op_vals.front()) != varNames[j1]->columnNames.end()) {
                op_vals.pop();
                op_vals.pop();
            }
            else {
                return 0;
            };
        };
        if(join_tab_cnt == 1) {
            if(!check_bitmap_file_exist(varNames[j1], varNames[op_join.front()])) {
                return 0;
            };
        };
        return 1;
    }
    else
        return 0;
}


void star_join(const char *s, const string j1)
{	
    map<string,bool> already_copied;
    queue<string> op_left;
    CudaSet* left = varNames.find(j1)->second;

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(auto i=0; i < sel_count; i++) {
        if(std::find(left->columnNames.begin(), left->columnNames.end(), op_value.front()) != left->columnNames.end())
            op_left.push(op_value.front());
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };
    auto op_sel_s(op_sel), op_sel_s_as(op_sel_as), op_g(op_value);
    CudaSet* c = new CudaSet(op_sel_s, op_sel_s_as);

    string f1, f2;
    map<string, string> key_map;
    map<string, char> sort_map;
    map<string, string> r_map;
	
    for(auto i = 0; i < join_tab_cnt; i++) {

        f1 = op_g.front();
        op_g.pop();
        f2 = op_g.front();
        op_g.pop();
        r_map[f1] = f2;
		

        queue<string> op_jj(op_join);
        for(auto z = 0; z < (join_tab_cnt-1) - i; z++)
            op_jj.pop();
		

        size_t rcount;
        queue<string> op_vd(op_g), op_alt(op_sel);
        unsigned int jc = join_col_cnt;
		
        while(jc) {
            jc--;
            op_vd.pop();
            op_alt.push(op_vd.front());
            op_vd.pop();
        };
		
        key_map[op_jj.front()] = f1;

        CudaSet* right = varNames.find(op_jj.front())->second;
        if(!check_bitmaps_exist(left, right)) {
            cout << "Required bitmap on table " << op_jj.front() << " doesn't exists" << endl;
            exit(0);
        };
		
        queue<string> second;
        while(!op_alt.empty()) {
            if(f2.compare(op_alt.front()) != 0 && std::find(right->columnNames.begin(), right->columnNames.end(), op_alt.front()) != right->columnNames.end()) {
                second.push(op_alt.front());
                //cout << "col " << op_alt.front() << " " << op_jj.front() <<  endl;
                op_left.push(f1);
            };
            op_alt.pop();
        };
        if(!second.empty()) {
            right->filtered = 0;
            right->mRecCount = right->maxRecs;
            load_queue(second, right, "", rcount, 0, right->segCount, 0,0); // put all used columns into GPU
        };
    };
	
    queue<string> idx;
    set<string> already_loaded;
    bool right_cpy = 0;
    for (unsigned int i = 0; i < left->segCount; i++) {
        std::clock_t start2 = std::clock();
        if(verbose)
            cout << "segment " << i << " " << getFreeMem() <<  endl;

        idx = left->fil_value;
        already_loaded.clear();
        while(!idx.empty()) {
            //load the index
            if(idx.front().find(".") != string::npos && (already_loaded.find(idx.front()) == already_loaded.end())) {
                //extract table name and colname from index name
                already_loaded.insert(idx.front());
                size_t pos1 = idx.front().find_first_of(".", 0);
                size_t pos2 = idx.front().find_first_of(".", pos1+1);
                CudaSet* r = varNames.find(idx.front().substr(pos1+1, pos2-pos1-1))->second;
                char a;
				//cout << "loading index " << idx.front() << endl;
                a = left->loadIndex(idx.front(), i);
                sort_map[idx.front().substr(pos1+1, pos2-pos1-1)] = a;
            };
            idx.pop();
        };

        left->filtered = 0;
        size_t cnt_c = 0;
        allocColumns(left, left->fil_value);
        copyColumns(left, left->fil_value, i, cnt_c);
        bool* res = filter(left->fil_type, left->fil_value, left->fil_nums, left->fil_nums_f, left->fil_nums_precision, left, i);
        thrust::device_ptr<bool> star((bool*)res);
        size_t cnt = thrust::count(star, star + (unsigned int)left->mRecCount, 1);
        //cout << "join res " << cnt << " out of " << left->mRecCount << endl;
        thrust::host_vector<unsigned int> prm_vh(cnt);
        thrust::device_vector<unsigned int> prm_v(cnt);
        thrust::host_vector<unsigned int> prm_tmp(cnt);
        thrust::device_vector<unsigned int> prm_tmp_d(cnt);
        //std::cout<< "seg filter " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;

        if(cnt) { //gather

            //start1 = std::clock();
            left->prm_d.resize(cnt);
            thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)left->mRecCount-1),
                            star, left->prm_d.begin(), thrust::identity<bool>());
            thrust::device_free(star);
            prm_vh = left->prm_d;

            size_t offset = c->mRecCount;
            c->resize_join(cnt);
            queue<string> op_sel1(op_sel_s);
            void* temp;
            CUDA_SAFE_CALL(hipMalloc((void **) &temp, cnt*max_char(c)));
			hipMemset(temp,0,cnt*max_char(c));
            CudaSet *t;
            unsigned int cnt1, bits;
            int_type lower_val;
            thrust::device_vector<unsigned int> output(cnt);
            //std::cout<< "seg start " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;

            while(!op_sel1.empty()) {

                if(std::find(left->columnNames.begin(), left->columnNames.end(), op_sel1.front()) !=  left->columnNames.end()) {

                    if(left->filtered)
                        t = varNames[left->source_name];
                    else
                        t = left;

                    if(left->type[op_sel1.front()] <= 1) {

                        if(ssd && !interactive) {
                            //start1 = std::clock();
                            lower_val = t->readSsdSegmentsFromFile(i, op_sel1.front(), offset, prm_vh, c);
                            //std::cout<<  "SSD L SEEK READ " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
                        }
                        else {
                            t->readSegmentsFromFile(i, op_sel1.front());
                            void* h;
							
                            if(!interactive) {
                                if(left->type[op_sel1.front()] == 0)
                                    h = t->h_columns_int[op_sel1.front()].data();
                                else
                                    h = t->h_columns_float[op_sel1.front()].data();
                            }
                            else {
                                string ff = t->load_file_name + "." + op_sel1.front()+ "." + to_string(i);
                                h = buffers[ff];
                            };
                            cnt1 = ((unsigned int*)h)[0];//bytes
                            lower_val = ((int_type*)(((unsigned int*)h)+1))[0];
                            bits = ((unsigned int*)((char*)h + cnt1))[8];
                            //cout << cnt1 << " " << lower_val << " " << bits << " " << left->type[op_sel1.front()] << endl;

                            if(bits == 8) {
                                if(left->type[op_sel1.front()] == 0) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                }
                                else {
                                    int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), ptr + offset);
                                };
                            }
                            else if(bits == 16) {
                                if(left->type[op_sel1.front()] == 0) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                }
                                else {
                                    int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), ptr + offset);
                                };
                            }
                            else if(bits == 32) {
                                if(left->type[op_sel1.front()] == 0) {									
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                }
                                else {
                                    int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), ptr + offset);
                                }
                            }
                            else if(bits == 64) {
                                if(left->type[op_sel1.front()] == 0) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(),  (int_type*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                }
                                else {
                                    int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                    thrust::gather(prm_vh.begin(), prm_vh.end(),  (int_type*)((unsigned int*)h + 6), ptr + offset);
                                };
                            };
                        };

                        if(left->type[op_sel1.front()] != 1)
                            thrust::transform( c->h_columns_int[op_sel1.front()].begin() + offset,  c->h_columns_int[op_sel1.front()].begin() + offset + cnt,
                                               thrust::make_constant_iterator(lower_val), c->h_columns_int[op_sel1.front()].begin() + offset, thrust::plus<int_type>());
                        else {
                            int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                            thrust::transform(ptr + offset, ptr + offset + cnt,
                                              thrust::make_constant_iterator(lower_val), ptr + offset, thrust::plus<int_type>());
                            thrust::transform(ptr + offset, ptr + offset + cnt, c->h_columns_float[op_sel1.front()].begin() + offset, long_to_float());
                        };

                    }
                    else { //gather string. There are no strings in fact tables.

                    };
                }
                else {
                    for(auto it = key_map.begin(); it != key_map.end(); it++) {
                        CudaSet* r = varNames.find(it->first)->second;

                        if(std::find(r->columnNames.begin(), r->columnNames.end(), op_sel1.front()) !=  r->columnNames.end()) {

                            if(i == 0) {
                                if(data_dict[varNames[it->first]->load_file_name][op_sel1.front()].col_type == 2) {
                                    //cout << "SET " << op_sel1.front() << " to " << varNames[it->first]->load_file_name + "." + op_sel1.front() << endl;
                                    c->string_map[op_sel1.front()] = varNames[it->first]->load_file_name + "." + op_sel1.front();
                                };
                            }

                            if(left->filtered)
                                t = varNames[left->source_name];
                            else
                                t = left;

                            if(ssd && !interactive) {
                                //start1 = std::clock();
                                lower_val = t->readSsdSegmentsFromFileR(i, key_map[it->first], prm_vh, prm_tmp);
                                //std::cout<<  "SSD R SEEK READ " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
                            }
                            else {
                                t->readSegmentsFromFile(i, key_map[it->first]);
                                void* h;
                                if(!interactive) {
                                    h = t->h_columns_int[key_map[it->first]].data();
                                }
                                else {
                                    string ff = t->load_file_name + "." + key_map[it->first] + "." + to_string(i);
                                    h = buffers[ff];
                                };
                                cnt1 = ((unsigned int*)h)[0];
                                lower_val = ((int_type*)(((unsigned int*)h)+1))[0];
                                bits = ((unsigned int*)((char*)h + cnt1))[8];
                                //cout << cnt1 << " " << lower_val << " " << bits << endl;

                                if(bits == 8) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), prm_tmp.begin());
                                }
                                else if(bits == 16) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), prm_tmp.begin());
                                }
                                else if(bits == 32) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), prm_tmp.begin());
                                }
                                else if(bits == 64) {
                                    thrust::gather(prm_vh.begin(), prm_vh.end(),  (int_type*)((unsigned int*)h + 6), prm_tmp.begin());
                                };
                            };


                            if(lower_val != 1)
                                thrust::transform(prm_tmp.begin(), prm_tmp.end(), thrust::make_constant_iterator(lower_val-1), prm_tmp.begin(), thrust::plus<unsigned int>());
                            if(sort_map[r->source_name] == '1') { // sorted consecutive starting with 1 dimension keys
                                prm_tmp_d = prm_tmp;
                                //cout << "PATH 1 " << endl;
                            }
                            else {
                                //cout << "PATH 2 " << r->source_name << endl;
                                output = prm_tmp;

                                if(r->d_columns_int[r_map[key_map[it->first]]].size() == 0) {
                                    r->d_columns_int[r_map[key_map[it->first]]].resize(r->maxRecs);
                                };
                                if(right_cpy == 0) {
                                    r->CopyColumnToGpu(r_map[key_map[it->first]]);
                                };

                                thrust::lower_bound(r->d_columns_int[r_map[key_map[it->first]]].begin(), r->d_columns_int[r_map[key_map[it->first]]].end(),
                                                    output.begin(), output.end(),
                                                    prm_tmp_d.begin());

                            };

                            if(r->type[op_sel1.front()] != 1) {
                                thrust::device_ptr<int_type> d_tmp((int_type*)temp);
                                thrust::gather(prm_tmp_d.begin(), prm_tmp_d.end(), r->d_columns_int[op_sel1.front()].begin(), d_tmp);
                                thrust::copy(d_tmp, d_tmp + cnt, c->h_columns_int[op_sel1.front()].begin() + offset);
                            }
                            else {
                                thrust::device_ptr<float_type> d_tmp((float_type*)temp);
                                thrust::gather(prm_tmp_d.begin(), prm_tmp_d.end(), r->d_columns_float[op_sel1.front()].begin(), d_tmp);
                                thrust::copy(d_tmp, d_tmp + cnt, c->h_columns_float[op_sel1.front()].begin() + offset);
                            };
                            break;
                        };
                    };
                };
                op_sel1.pop();
                //std::cout<<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            hipFree(temp);
            right_cpy = 1;
        };
        //std::cout<< "SEG " << i << " "  <<  ( ( std::clock() - start2 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        //unload the segment indexes :
        idx = left->fil_value;
        already_loaded.clear();
        while(!idx.empty()) {
            if(idx.front().find(".") != string::npos && (already_loaded.find(idx.front()) == already_loaded.end())) {
                //extract table name and colname from index name
                already_loaded.insert(idx.front());
                size_t pos1 = idx.front().find_first_of(".", 0);
                size_t pos2 = idx.front().find_first_of(".", pos1+1);
                CudaSet* r = varNames.find(idx.front().substr(pos1+1, pos2-pos1-1))->second;
                string f1 = idx.front() + "." + to_string(i);
                auto it = index_buffers.find(f1);
                if(it != index_buffers.end()) {
                    hipHostFree(index_buffers[f1]);
                    index_buffers.erase(it);
                };
            };
            idx.pop();
        };
    };

    //if(verbose)
    //    std::cout<< "star join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;


    while(!op_join.empty()) {
        varNames[op_join.front()]->deAllocOnDevice();
        op_join.pop();
    };

    varNames[s] = c;
    c->maxRecs = c->mRecCount;

    if(verbose)
        cout << endl << "join count " << c->mRecCount << endl;
};


void emit_join(const char *s, const char *j1, const int grp, const int start_seg, const int end_seg)
{
    //cout << "emit_join " <<  s << " " << join_tab_cnt << " " << op_join.front() <<  endl;
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(j1) == mystat.end() && data_dict.count(j1) == 0) {
            process_error(2, "Join : couldn't find variable " + string(j1) );
        };
        if (mystat.find(op_join.front()) == mystat.end() && data_dict.count(op_join.front()) == 0) {
            process_error(2, "Join : couldn't find variable " + op_join.front() );
        };
        mystat[s] = statement_count;
        mystat[j1] = statement_count;
		if(filter_var.find(j1) != filter_var.end()) {
			mystat[filter_var[j1]] = statement_count;
		};
        check_used_vars();
        while(!op_join.empty()) {
            mystat[op_join.front()] = statement_count;
			if(filter_var.find(op_join.front()) != filter_var.end()) {
				mystat[filter_var[op_join.front()]] = statement_count;
			};			
            op_join.pop();			
        };
        return;
    };

    queue<string> op_m(op_value);

    if(check_star_join(j1)) {
        if(verbose)
            cout << "executing star join !! " << endl;
        star_join(s, j1);
    }
    else {
        if(join_tab_cnt > 1) {
            string tab_name;
            for(unsigned int i = 1; i <= join_tab_cnt; i++) {

                if(i == join_tab_cnt)
                    tab_name = s;
                else
                    tab_name = s + to_string(i);

                string j, j2;
                if(i == 1) {
                    j2 = op_join.front();
                    op_join.pop();
                    j = op_join.front();
                    op_join.pop();
                }
                else {
                    if(!op_join.empty()) {
                        j = op_join.front();
                        op_join.pop();
                    }
                    else
                        j = j1;
                    j2 = s + to_string(i-1);
                };
                emit_multijoin(tab_name, j, j2, i, s, start_seg, end_seg);
                op_value = op_m;
            };
        }
        else {
            emit_multijoin(s, j1, op_join.front(), 1, s, start_seg, end_seg);
            op_join.pop();
        };
    };


    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_m.front());
        op_m.pop();
        op_sel_as.push(op_m.front());
        op_m.pop();
    };
    while(!op_sel_as.empty()) {
        //cout << "alias " << op_sel.front() << " : " << op_sel_as.front() << endl;
        if(op_sel.front() != op_sel_as.front()) {
            if(varNames[s]->type[op_sel.front()] == 0) {
                varNames[s]->h_columns_int[op_sel_as.front()] = varNames[s]->h_columns_int[op_sel.front()];
                varNames[s]->h_columns_int.erase(op_sel.front());
                varNames[s]->d_columns_int[op_sel_as.front()] = varNames[s]->d_columns_int[op_sel.front()];
                varNames[s]->d_columns_int.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 0;
                varNames[s]->type.erase(op_sel.front());
            }
            else if(varNames[s]->type[op_sel.front()] == 1) {
                varNames[s]->h_columns_float[op_sel_as.front()] = varNames[s]->h_columns_float[op_sel.front()];
                varNames[s]->h_columns_float.erase(op_sel.front());
                varNames[s]->d_columns_float[op_sel_as.front()] = varNames[s]->d_columns_float[op_sel.front()];
                varNames[s]->d_columns_float.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 1;
                varNames[s]->type.erase(op_sel.front());
                varNames[s]->decimal.erase(op_sel.front());
            }
            else {
                varNames[s]->h_columns_char[op_sel_as.front()] = varNames[s]->h_columns_char[op_sel.front()];
                varNames[s]->h_columns_char.erase(op_sel.front());
                varNames[s]->d_columns_char[op_sel_as.front()] = varNames[s]->d_columns_char[op_sel.front()];
                varNames[s]->d_columns_char.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 2;
                varNames[s]->type.erase(op_sel.front());
                varNames[s]->char_size[op_sel_as.front()] = varNames[s]->char_size[op_sel.front()];
                varNames[s]->char_size.erase(op_sel.front());
            };
            varNames[s]->decimal[op_sel_as.front()] = varNames[s]->decimal[op_sel.front()];
            auto it = std::find(varNames[s]->columnNames.begin(), varNames[s]->columnNames.end(), op_sel.front());
            *it = op_sel_as.front();
        };
        op_sel_as.pop();
        op_sel.pop();
    };


    clean_queues();

    if(mystat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(op_join.size()) {
        if(mystat[op_join.front()] == statement_count && op_join.front().compare(j1) != 0) {
            varNames[op_join.front()]->free();
            varNames.erase(op_join.front());
        };
    };
}

template<typename T, typename P>
void p_gather(thrust::host_vector<int>& h_tmp, T* h, P* dest)
{
	for(int i = 0; i < h_tmp.size(); i++) {
		dest[i] = h[h_tmp[i]];
	};	
};	



void emit_multijoin(const string s, const string j1, const string j2, const unsigned int tab, const char* res_name, const int start_segment, const int end_segment)
{

    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
        if(varNames.find(j1) == varNames.end())
            cout << "Couldn't find j1 " << j1 << endl;
        if(varNames.find(j2) == varNames.end())
            cout << "Couldn't find j2 " << j2 << " here " << endl;

        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };

    queue<string> op_sel_s(op_sel);
    queue<string> op_sel_s_as(op_sel_as);
    queue<string> op_g(op_value);

    if(tab > 0) {
        for(unsigned int z = 0; z < join_tab_cnt - tab; z++) {
            for(unsigned int j = 0; j < join_and_cnt[z]*2 + 2; j++) {
                op_sel_s.push(op_g.front());
                op_sel_s_as.push(op_g.front());
                op_g.pop();
            };
        };
    };

    string f1 = op_g.front();
    op_g.pop();
    string f2 = op_g.front();
    op_g.pop();

    if (verbose)
        cout << "JOIN " << s <<  " " <<  f1 << " " << f2 << " " << getFreeMem() <<  " " << phase_copy << endl;

    std::clock_t start1 = std::clock();
    CudaSet* c = new CudaSet(right, left, op_sel_s, op_sel_s_as);

    if ((left->mRecCount == 0 && !left->filtered) || (right->mRecCount == 0 && !right->filtered)) {
        c = new CudaSet(left, right, op_sel_s, op_sel_s_as);
        varNames[res_name] = c;
        clean_queues();
        return;
    };

    if(join_tab_cnt > 1 && tab < join_tab_cnt)
        c->tmp_table = 1;
    else
        c->tmp_table = 0;

    string colname1, colname2;
    string tmpstr;
    if (std::find(left->columnNames.begin(), left->columnNames.end(), f1) != left->columnNames.end()) {
        colname1 = f1;
        if (std::find(right->columnNames.begin(), right->columnNames.end(), f2) != right->columnNames.end()) {
            colname2 = f2;
        }
        else {
            process_error(2, "Couldn't find column " + f2 );
        };
    }
    else if (std::find(right->columnNames.begin(), right->columnNames.end(), f1) != right->columnNames.end()) {
        colname2 = f1;
        tmpstr = f1;
        f1 = f2;
        if (std::find(left->columnNames.begin(), left->columnNames.end(), f2) != left->columnNames.end()) {
            colname1 = f2;
            f2 = tmpstr;
        }
        else {
            process_error(2, "Couldn't find column " +f2 );
        };
    }
    else {
        process_error(2, "Couldn't find column " + f1);
    };


    if (!((left->type[colname1] == 0 && right->type[colname2]  == 0) || (left->type[colname1] == 2 && right->type[colname2]  == 2)
            || (left->type[colname1] == 1 && right->type[colname2]  == 1 && left->decimal[colname1] && right->decimal[colname2]))) {
        process_error(2, "Joins on floats are not supported ");
    };


    //bool decimal_join = 0;
    //if (left->type[colname1] == 1 && right->type[colname2]  == 1)
    //    decimal_join = 1;

    queue<string> op_vd(op_g);
    queue<string> op_g1(op_g);
    queue<string> op_alt(op_sel);
	
    unsigned int jc = join_and_cnt[join_tab_cnt - tab];
    while(jc) {
        jc--;
        op_vd.pop();
        op_alt.push(op_vd.front());
        op_vd.pop();
    };

    size_t rcount = 0, cnt_r;
    queue<string> cc;

    if (left->type[colname1]  == 2) {
        left->d_columns_int[colname1] = thrust::device_vector<int_type>();
    }
    else {
        cc.push(f1);
        allocColumns(left, cc);
    };

    left->hostRecCount = left->mRecCount;

    size_t cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
    queue<string> lc(cc);
    thrust::device_vector<unsigned int> v_l(left->maxRecs);
    //MGPU_MEM(int) aIndicesDevice, bIndicesDevice, intersectionDevice;	
    stack<string> exe_type;
    set<string> field_names;
    exe_type.push(f2);
    for(unsigned int i = 0; i < right->columnNames.size(); i++) {
        if (std::find(c->columnNames.begin(), c->columnNames.end(), right->columnNames[i]) != c->columnNames.end() || right->columnNames[i] == f2 || join_and_cnt[join_tab_cnt - tab]) {
            field_names.insert(right->columnNames[i]);
        };
    };

    thrust::device_vector<int> p_tmp;    
    unsigned int start_part = 0;
	bool prejoin = 0;

    while(start_part < right->segCount) {

        right->deAllocOnDevice();
		std::clock_t start12 = std::clock();
        if(right->not_compressed || (!right->filtered && getFreeMem() < right->columnNames.size()*right->hostRecCount*8*2)) {
            cnt_r = load_right(right, f2, op_g1, op_alt, rcount, start_part, start_part+1);
            start_part = start_part+1;			
        }
        else {
            cnt_r = load_right(right, f2, op_g1, op_alt, rcount, start_part, right->segCount);			
            start_part = right->segCount;
			
			for(unsigned int i=0; i < right->columnNames.size(); i++) {
				if (right->type[right->columnNames[i]] != 1) {
					right->d_columns_int[right->columnNames[i]].shrink_to_fit();
				}
				else 
					right->d_columns_float[right->columnNames[i]].shrink_to_fit();
			};    			
        };

        right->mRecCount = cnt_r;
        bool order = 1;
		
		
        if(!right->presorted_fields.empty() && right->presorted_fields.front() == f2) {
            order = 0;
            //cout << "No need to sort " << endl;
            if (right->d_columns_int[f2][0] == 1 && right->d_columns_int[f2][right->d_columns_int[f2].size()-1] == right->d_columns_int[f2].size())
                right->sort_check = '1';
            else {
                right->sort_check = '0';
            };
        };
		
        if(order) {
            if(thrust::is_sorted(right->d_columns_int[f2].begin(), right->d_columns_int[f2].end())) {
                if (right->d_columns_int[f2][0] == 1 && right->d_columns_int[f2][right->d_columns_int[f2].size()-1] == right->d_columns_int[f2].size()) {
                    right->sort_check = '1';
                }
                else {
                    right->sort_check = '0';
                };
            }
            else {
				//cout << "sorting " << endl;
				size_t tot_size = right->mRecCount*8*right->columnNames.size();
				if (getFreeMem() > tot_size*1.5) {
					order_inplace(right, exe_type, field_names, 0);					
				}
				else {
					for (auto it=field_names.begin(); it!=field_names.end(); ++it) {
						//cout << "sorting " << *it << endl;
						if(right->type[*it] != 1) {
							if(right->h_columns_int[*it].size() < right->mRecCount)
								right->h_columns_int[*it].resize(right->mRecCount);
							thrust::copy(right->d_columns_int[*it].begin(), right->d_columns_int[*it].begin() +	right->mRecCount, right->h_columns_int[*it].begin());			
						}	
						else {
							if(right->type[*it] == 1) {						
								if(right->h_columns_float[*it].size() < right->mRecCount)
									right->h_columns_float[*it].resize(right->mRecCount);
							};		
							thrust::copy(right->d_columns_float[*it].begin(), right->d_columns_float[*it].begin() +	right->mRecCount, right->h_columns_float[*it].begin());			
						};		
					};	
					order_inplace_host(right, exe_type, field_names, 0);
				    for (auto it=field_names.begin(); it!=field_names.end(); ++it) {
						if(right->type[*it] != 1) 
							thrust::copy(right->h_columns_int[*it].begin(), right->h_columns_int[*it].begin() + right->mRecCount, right->d_columns_int[*it].begin());
						else	
							thrust::copy(right->h_columns_float[*it].begin(), right->h_columns_float[*it].begin() + right->mRecCount, right->d_columns_float[*it].begin());
					};
				};		
            };
        };

	//std::cout<< "join right load time " <<  ( ( std::clock() - start12 ) / (double)CLOCKS_PER_SEC ) <<  " " << getFreeMem() << '\n';				

        int e_segment;
        if(end_segment == -1) {
            e_segment  = left->segCount;
        }
        else
            e_segment = end_segment;

        for (unsigned int i = start_segment; i < e_segment; i++) {
		
            if(verbose)
                //cout << "segment " << i <<  '\xd';
                cout << "segment " << i <<  endl;
            cnt_l = 0;
			
            copyColumns(left, lc, i, cnt_l);	
            cnt_l = left->mRecCount;
			auto join_eq_type1(join_eq_type);

            if (cnt_l) {							
				
		
                // sort the left index column, save the permutation vector, it might be needed later				
                thrust::device_ptr<int_type> d_col((int_type*)thrust::raw_pointer_cast(left->d_columns_int[colname1].data()));
                thrust::sequence(v_l.begin(), v_l.begin() + cnt_l,0,1);

                bool do_sort = 1;
                if(!left->sorted_fields.empty()) {
                    if(left->sorted_fields.front() == f1) {
                        do_sort = 0;
                    };
                }
                else if(!left->presorted_fields.empty()) {
                    if(left->presorted_fields.front() == f1) {
                        do_sort = 0;
                    };
                };

                if(do_sort) {
                    thrust::sort_by_key(d_col, d_col + cnt_l, v_l.begin());
				}	
                else if(verbose)
                    cout << "No need of sorting " << endl;
				
				if(prejoin) {
					//res_count = SetOpKeys<MgpuSetOpIntersection, true>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
					//										thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
					//										&intersectionDevice, *context, false);	
					//if(!res_count)
					//	continue;     
				};		
				
					
				

                if (left->d_columns_int[colname1][0] > right->d_columns_int[colname2][cnt_r-1] ||
                        left->d_columns_int[colname1][cnt_l-1] < right->d_columns_int[colname2][0]) {
                    if(verbose)
                        cout << endl << "skipping after copying " << endl;
                    continue;
                };
                //else
                //    cout << "JOINING " << left->d_columns_int[colname1][0] << ":" << left->d_columns_int[colname1][cnt_l-1] << " AND " << right->d_columns_int[colname2][0] << ":" << right->d_columns_int[colname2][cnt_r-1] << endl;

                //cout << "joining " << left->d_columns_int[colname1][0] << " : " << left->d_columns_int[colname1][cnt_l-1] << " and " << right->d_columns_int[colname2][0] << " : " << right->d_columns_int[colname2][cnt_r-1] << endl;

                char join_kind = join_type.front();
				std::clock_t start11 = std::clock();
				mem_t<int2> res;
				
                if (join_kind == 'I' || join_kind == '1' || join_kind == '2' || join_kind == '3' || join_kind == '4') {
                    //res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                    //            thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                    //            &aIndicesDevice, &bIndicesDevice,
                    //            mgpu::less<int_type>(), *context);




					res = inner_join(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l, 
									thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r, less_t<int_type>(), context);				
									
				};					
				   
				res_count = res.size();
				
               /* else if(join_kind == 'L')
                    res_count = RelationalJoin<MgpuJoinKindLeft>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                &aIndicesDevice, &bIndicesDevice,
                                mgpu::less<int_type>(), *context);
                else if(join_kind == 'R')
                    res_count = RelationalJoin<MgpuJoinKindRight>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                &aIndicesDevice, &bIndicesDevice,
                                mgpu::less<int_type>(), *context);
                else if(join_kind == 'O')
                    res_count = RelationalJoin<MgpuJoinKindOuter>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                &aIndicesDevice, &bIndicesDevice,
                                mgpu::less<int_type>(), *context);
				*/
				if(verbose)	
					std::cout<< "join time " <<  ( ( std::clock() - start11 ) / (double)CLOCKS_PER_SEC ) <<  '\n';				

                if(verbose)
                    cout << "RES " << res_count << endl;
				if(res_count == 0)
					prejoin = 1; 				
				
				
                thrust::device_ptr<int> d_res1 = thrust::device_malloc<int>(res_count);
                thrust::device_ptr<int> d_res2 = thrust::device_malloc<int>(res_count);
				
				thrust::counting_iterator<unsigned int> begin(0);
				split_int2 ff(thrust::raw_pointer_cast(d_res1), thrust::raw_pointer_cast(d_res2), res.data());
				thrust::for_each(begin, begin + res_count, ff);
				

				
                if(res_count) {
                    p_tmp.resize(res_count);
                    thrust::sequence(p_tmp.begin(), p_tmp.end(),-1);
                    thrust::gather_if(d_res1, d_res1+res_count, d_res1, v_l.begin(), p_tmp.begin(), _1 >= 0);					
                };				
				

                // check if the join is a multicolumn join
                unsigned int mul_cnt = join_and_cnt[join_tab_cnt - tab];
                while(mul_cnt) {
					
                    mul_cnt--;
                    queue<string> mult(op_g);
                    string f3 = mult.front();
                    mult.pop();
                    string f4 = mult.front();
                    mult.pop();

                    //cout << "ADDITIONAL COL JOIN " << f3 << " " << f4 << " " << join_eq_type.front() << endl;

                    queue<string> rc;
                    rc.push(f3);
					
                    allocColumns(left, rc);
                    size_t offset = 0;
                    copyColumns(left, rc, i, offset, 0, 0);
                    rc.pop();

                    if (res_count) {
                        thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(res_count);
						
						if(right->d_columns_int[f4].size() == 0)
							load_queue(rc, right, f4, rcount, 0, right->segCount, 0, 0);                   


                        if (left->type[f3] == 1 && right->type[f4]  == 1) {                        	
							thrust::transform(make_permutation_iterator(left->d_columns_float[f3].begin(), p_tmp.begin()),
											  make_permutation_iterator(left->d_columns_float[f3].begin(), p_tmp.end()),
											  make_permutation_iterator(right->d_columns_float[f4].begin(), d_res2),
											  d_add, float_equal_to());
                        }
                        else {                        	
							if(join_eq_type1.front() != 'N') 
								thrust::transform(make_permutation_iterator(left->d_columns_int[f3].begin(), p_tmp.begin()),
												  make_permutation_iterator(left->d_columns_int[f3].begin(), p_tmp.end()),
												  make_permutation_iterator(right->d_columns_int[f4].begin(), d_res2),
												  d_add, thrust::equal_to<int_type>());
							else  {
								thrust::transform(make_permutation_iterator(left->d_columns_int[f3].begin(), p_tmp.begin()),
												  make_permutation_iterator(left->d_columns_int[f3].begin(), p_tmp.end()),
												  make_permutation_iterator(right->d_columns_int[f4].begin(), d_res2),
												  d_add, thrust::not_equal_to<int_type>());								
							};				  
                        };

						if (join_kind == 'I' || join_kind == '1' || join_kind == '2' || join_kind == '3' || join_kind == '4') {  // result count changes only in case of an inner join
                            unsigned int new_cnt = thrust::count(d_add, d_add+res_count, 1);
                            thrust::stable_partition(d_res2, d_res2 + res_count, d_add, thrust::identity<unsigned int>());
                            thrust::stable_partition(p_tmp.begin(), p_tmp.end(), d_add, thrust::identity<unsigned int>());                            
                            res_count = new_cnt;
						}
						else { //otherwise we consider it a valid left join result with non-nulls on the left side and nulls on the right side
							thrust::transform(d_res2, d_res2 + res_count, d_add , d_res2, set_minus());
						};
						thrust::device_free(d_add);
					};
					if(!join_eq_type1.empty())
						join_eq_type1.pop();
                };
				
				
				while(!join_eq_type1.empty())
					join_eq_type1.pop();
				
				//cout << "MUL res_count " << res_count << endl;			
				
				
				if(join_kind == '1') { //LEFT SEMI
					thrust::sort(p_tmp.begin(), p_tmp.begin() + res_count);
					auto new_end = thrust::unique(p_tmp.begin(), p_tmp.begin() + res_count);
					res_count = new_end - p_tmp.begin();
				}
				else if(join_kind == '2'){ // RIGHT SEMI
					thrust::sort(d_res2, d_res2 + res_count);
					auto new_end = thrust::unique(d_res2, d_res2 + res_count);
					res_count = new_end - d_res2;
					auto old_sz = ranj.size();
					ranj.resize(ranj.size() + res_count);
					thrust::copy(d_res2, d_res2 + res_count, ranj.begin() + old_sz);
					thrust::sort(ranj.begin(), ranj.end());
					auto ra_cnt = thrust::unique(ranj.begin(), ranj.end());
					ranj.resize(ra_cnt-ranj.begin());						
				}
				else if(join_kind == '3'){ // ANTI JOIN LEFT
					thrust::counting_iterator<int> iter(0);
					thrust::device_vector<int> rr(cnt_l);					
					auto new_end = thrust::set_difference(iter, iter+cnt_l, p_tmp.begin(), p_tmp.begin() + res_count, rr.begin());
					res_count = new_end - rr.begin();
					thrust::copy(rr.begin(), new_end, p_tmp.begin());
				}
				else if(join_kind == '4'){ // ANTI JOIN RIGHT
	
					thrust::sort(d_res2, d_res2 + res_count);			
					auto new_end = thrust::unique(d_res2, d_res2 + res_count);			
					auto cnt = new_end - d_res2;
					thrust::device_vector<int> seq(cnt + ranj.size());		
					
					//auto new_end = thrust::set_difference(seq.begin(), seq.end(), d_res2, d_res2 + res_count, rr.begin());
					auto new_end1 = thrust::set_union(d_res2, d_res2 + cnt, ranj.begin(), ranj.end(), seq.begin());					
					auto s_cnt = new_end1 - seq.begin();
					thrust::sort(seq.begin(), seq.begin() + s_cnt);
					auto end_seq = thrust::unique(seq.begin(), seq.begin() + s_cnt);
					auto u_cnt = end_seq - seq.begin();					
					ranj.resize(u_cnt);
					thrust::copy(seq.begin(), seq.begin() + u_cnt, ranj.begin());
					
					thrust::sort(ranj.begin(), ranj.end());
					auto ra_cnt = thrust::unique(ranj.begin(), ranj.end());
					ranj.resize(ra_cnt-ranj.begin());								
				}
				

                tot_count = tot_count + res_count;
				//cout << "tot " << tot_count << endl;

				//std::clock_t start12 = std::clock();
                if(res_count && join_kind != '4' && join_kind != '2') {          		
					
                    offset = c->mRecCount;
                    queue<string> op_sel1(op_sel_s);					
					c->resize_join(res_count);					
					if(scratch.size() < res_count*int_size)
						scratch.resize(res_count*int_size);
					thrust::fill(scratch.begin(), scratch.begin() + res_count*int_size, 0);										
                    std::map<string,bool> processed;
					
                    while(!op_sel1.empty()) {

                        if (processed.find(op_sel1.front()) != processed.end()) {
                            op_sel1.pop();
                            continue;
                        }
                        else
                            processed[op_sel1.front()] = 1;

                        while(!cc.empty())
                            cc.pop();

                        cc.push(op_sel1.front());

                        if(std::find(left->columnNames.begin(), left->columnNames.end(), op_sel1.front()) !=  left->columnNames.end() && join_kind != '2') {												
								allocColumns(left, cc);
								copyColumns(left, cc, i, k, 0, 0);						
								//gather
								if(left->type[op_sel1.front()] != 1 ) {
									thrust::device_ptr<int_type> d_tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));
									thrust::gather(p_tmp.begin(), p_tmp.begin() + res_count, left->d_columns_int[op_sel1.front()].begin(), d_tmp);
									thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);
								}
								else {
									thrust::device_ptr<float_type> d_tmp((float_type*)thrust::raw_pointer_cast(scratch.data()));
									thrust::gather(p_tmp.begin(), p_tmp.begin() + res_count, left->d_columns_float[op_sel1.front()].begin(), d_tmp);
									thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
								};

								if(op_sel1.front() != colname1)
									left->deAllocColumnOnDevice(op_sel1.front());
							//};
                        }
                        else if(std::find(right->columnNames.begin(), right->columnNames.end(), op_sel1.front()) !=  right->columnNames.end()) {

                            //gather
                            if(right->type[op_sel1.front()] != 1) {
                                thrust::device_ptr<int_type> d_tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));
                                thrust::gather(d_res2, d_res2 + res_count, right->d_columns_int[op_sel1.front()].begin(), d_tmp);
                                thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);								
                            }
                            else {
                                thrust::device_ptr<float_type> d_tmp((float_type*)thrust::raw_pointer_cast(scratch.data()));
                                thrust::gather(d_res2, d_res2 + res_count, right->d_columns_float[op_sel1.front()].begin(), d_tmp);
                                thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
                            }
                        }
                        else {
                        };
                        op_sel1.pop();
                    };
                };
				thrust::device_free(d_res1);
				thrust::device_free(d_res2);
            };
        };
		
		if(join_type.front() == '4') {
			thrust::device_vector<int> st(cnt_r);
			thrust::sequence(st.begin(), st.end(),0,1);
			thrust::device_vector<int> r(cnt_r);
			auto new_end = thrust::set_difference(st.begin(), st.end(), ranj.begin(), ranj.end(), r.begin());	
			ranj.resize(0);	
			res_count = new_end - r.begin();
			tot_count = res_count;
			
			queue<string> op_sel1(op_sel_s);					
			c->resize_join(res_count);					
			if(scratch.size() < res_count*int_size)
				scratch.resize(res_count*int_size);
			thrust::fill(scratch.begin(), scratch.begin() + res_count*int_size, 0);										
			std::map<string,bool> processed;
			
			while(!op_sel1.empty()) {
				if (processed.find(op_sel1.front()) != processed.end()) {
					op_sel1.pop();
					continue;
				}
				else
					processed[op_sel1.front()] = 1;

				while(!cc.empty())
					cc.pop();

				cc.push(op_sel1.front());			
				thrust::device_ptr<int_type> d_tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));
				thrust::gather(r.begin(), r.end(), right->d_columns_int[op_sel1.front()].begin(), d_tmp);
				thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin());				
				op_sel1.pop();
			};		
		}
		else if(join_type.front() == '2') {
			res_count = ranj.size();
			tot_count = res_count;		
			queue<string> op_sel1(op_sel_s);					
			c->resize_join(res_count);					
			if(scratch.size() < res_count*int_size)
				scratch.resize(res_count*int_size);
			thrust::fill(scratch.begin(), scratch.begin() + res_count*int_size, 0);										
			std::map<string,bool> processed;
			
			while(!op_sel1.empty()) {
				if (processed.find(op_sel1.front()) != processed.end()) {
					op_sel1.pop();
					continue;
				}
				else
					processed[op_sel1.front()] = 1;

				while(!cc.empty())
					cc.pop();

				cc.push(op_sel1.front());			
				thrust::device_ptr<int_type> d_tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));
				thrust::gather(ranj.begin(), ranj.end(), right->d_columns_int[op_sel1.front()].begin(), d_tmp);
				thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin());				
				op_sel1.pop();
			};	
			ranj.resize(0);			
		};		
    };	

    left->deAllocOnDevice();
    right->deAllocOnDevice();
    c->deAllocOnDevice();

    varNames[s] = c;
    c->mRecCount = tot_count;
    c->hostRecCount = tot_count;
    c->name = s;

    if(verbose)
        cout << "tot res " << tot_count << " " << getFreeMem() << endl;

    if(right->tmp_table == 1) {
        right->free();
        varNames.erase(j2);
    }
    else {
        if(mystat[j2] == statement_count) {
            right->free();
            varNames.erase(j2);
        };
    };
    if(mystat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };	
	
    join_type.pop();	
	if(!join_eq_type.empty())
		join_eq_type.pop();
	
    size_t tot_size = tot_count*8*c->columnNames.size();		
    if (getFreeMem() > tot_size) {
        c->maxRecs = tot_count;
        c->segCount = 1;
    }
    else {		
        c->segCount = ((tot_size)/getFreeMem() + 1);
        c->maxRecs = c->hostRecCount - (c->hostRecCount/c->segCount)*(c->segCount-1);
    };	

    if(verbose)
        std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;	

}


void order_on_host(CudaSet *a, CudaSet* b, queue<string> names, stack<string> exe_type, stack<string> exe_value)
{
    unsigned int tot = 0;
    if(!a->not_compressed) { //compressed
        allocColumns(a, names);

        unsigned int c = 0;
        size_t cnt = 0;
        for(unsigned int i = 0; i < a->segCount; i++) {
            copyColumns(a, names, (a->segCount - i) - 1, cnt);	//uses segment 1 on a host	to copy data from a file to gpu
            if (a->mRecCount) {
                a->CopyToHost((c - tot) - a->mRecCount, a->mRecCount);
                tot = tot + a->mRecCount;
            };
        };
    }
    else
        tot = a->mRecCount;

    b->resize(tot); //resize host arrays
    a->mRecCount = tot;

    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    size_t maxSize =  a->mRecCount;
    char* temp;
    temp = new char[maxSize*max_char(a)];

    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {

        if (a->type[exe_type.top()] == 0)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_char_permutation(a, exe_type.top(), permutation, exe_value.top(),  temp, 1);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if (a->type[a->columnNames[i]] != 1) {
            apply_permutation_host(a->h_columns_int[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_int[a->columnNames[i]].data());
        }
        else
            apply_permutation_host(a->h_columns_float[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_float[a->columnNames[i]].data());
    };

    delete [] temp;
    delete [] permutation;
}



void emit_order(const char *s, const char *f, const int e, const int ll)
{
    if(ll == 0)
        statement_count++;
	
    if (scan_state == 0 && ll == 0) {
        if (mystat.find(f) == mystat.end() && data_dict.count(f) == 0) {
            process_error(2, "Order : couldn't find variable " + string(f));
        };
        mystat[s] = statement_count;
        mystat[f] = statement_count;
		if(filter_var.find(f) != filter_var.end()) 
			mystat[filter_var[f]] = statement_count;

        return;
    };

    if (scan_state == 0) {
        check_used_vars();
        return;
    };

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;
    stack<string> exe_type, exe_value;

    if(verbose)
        cout << "ORDER: " << s << " " << f << endl;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
        if(std::find(a->columnNames.begin(), a->columnNames.end(), exe_type.top()) == a->columnNames.end()) {
            process_error(2, "Couldn't find name " + exe_type.top());
        };

    };

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };

    queue<string> names;
    for (unsigned int i = 0; i < a->columnNames.size() ; i++ )
        names.push(a->columnNames[i]);

    CudaSet *b = a->copyDeviceStruct();

    //lets find out if our data set fits into a GPU
    size_t mem_available = getFreeMem();
    size_t rec_size = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[a->columnNames[i]] == 0)
            rec_size = rec_size + int_size;
        else if(a->type[a->columnNames[i]] == 1)
            rec_size = rec_size + float_size;
        else
            rec_size = rec_size + a->char_size[a->columnNames[i]];
    };
    bool fits;
    if (rec_size*a->mRecCount > (mem_available/2)) // doesn't fit into a GPU
        fits = 0;
    else fits = 1;

    if(!fits) {
        order_on_host(a, b, names, exe_type, exe_value);
    }
    else {
        // initialize permutation to [0, 1, 2, ... ,N-1]

        size_t rcount;
        if(a->filtered) {
            CudaSet *t = varNames[a->source_name];
            a->mRecCount = t->mRecCount;
            a->hostRecCount = a->mRecCount;
        };

        a->mRecCount = load_queue(names, a, op_vx.front(), rcount, 0, a->segCount);

		if(scratch.size() < a->mRecCount)
			scratch.resize(a->mRecCount*4);
		thrust::device_ptr<unsigned int> permutation((unsigned int*)thrust::raw_pointer_cast(scratch.data())); 
        thrust::sequence(permutation, permutation+(a->mRecCount));
        unsigned int* perm_ptr = thrust::raw_pointer_cast(permutation);

        void* temp;
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char(a)));

        if(a->filtered)
            varNames[a->source_name]->hostRecCount = varNames[a->source_name]->mRecCount;
        else
            a->hostRecCount = a->mRecCount;;


        if(a->filtered)
            varNames[a->source_name]->mRecCount = varNames[a->source_name]->hostRecCount;
        else
            a->mRecCount = a->hostRecCount;

        for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
            if (a->type[exe_type.top()] == 0 && a->string_map.find(exe_type.top()) == a->string_map.end())
                update_permutation(a->d_columns_int[exe_type.top()], perm_ptr, a->mRecCount, exe_value.top(), (int_type*)temp, 64);
            else if (a->type[exe_type.top()] == 1)
                update_permutation(a->d_columns_float[exe_type.top()], perm_ptr, a->mRecCount,exe_value.top(), (float_type*)temp, 64);
            else {
                //get strings to device
                update_char_permutation(a, exe_type.top(), perm_ptr, exe_value.top(), temp, 0);
            };
        };

        b->resize(a->mRecCount); //resize host arrays
        b->mRecCount = a->mRecCount;

        for (unsigned int i = 0; i < a->mColumnCount; i++) {
            if (a->type[a->columnNames[i]] != 1) {
                apply_permutation(a->d_columns_int[a->columnNames[i]], perm_ptr, a->mRecCount, (int_type*)temp, 64);
            }
            else
                apply_permutation(a->d_columns_float[a->columnNames[i]], perm_ptr, a->mRecCount, (float_type*)temp, 64);
        };

        for(unsigned int i = 0; i < a->mColumnCount; i++) {
            if(a->type[a->columnNames[i]] != 1) {
                thrust::copy(a->d_columns_int[a->columnNames[i]].begin(), a->d_columns_int[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_int[a->columnNames[i]].begin());
            }
            else
                thrust::copy(a->d_columns_float[a->columnNames[i]].begin(), a->d_columns_float[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_float[a->columnNames[i]].begin());
        };

        b->deAllocOnDevice();
        a->deAllocOnDevice();
        hipFree(temp);
    };

    varNames[s] = b;
    b->segCount = 1;
    b->not_compressed = 1;
    b->string_map = a->string_map;

    if(mystat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


void emit_select(const char *s, const char *f, const int grp_cnt)
{	
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(f) == mystat.end() && data_dict.count(f) == 0) {
            process_error(2, "Select : couldn't find variable " + string(f) );
        };
        mystat[s] = statement_count;
        mystat[f] = statement_count;
		if(filter_var.find(f) != filter_var.end()) 
			mystat[filter_var[f]] = statement_count;
		
        check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        cout << "Couldn't find1 " << f << endl;
        process_error(2, "Couldn't find(1) " + string(f) );
        return;
    };

    queue<string> op_v1(op_value);
    while(op_v1.size() > grp_cnt)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < grp_cnt; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };


    CudaSet *a;
    if(varNames.find(f) != varNames.end())
        a = varNames.find(f)->second;
    else {
        process_error(2, "Couldn't find " + string(f) );
    };

    if(a->mRecCount == 0 && !a->filtered) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        c->name = s;
        clean_queues();
        if(verbose)
            cout << "SELECT " << s << " count : 0,  Mem " << getFreeMem() << endl;
        return;
    };

    if(verbose)
        cout << "SELECT " << s << " " << f << " " << getFreeMem() << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;

    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            tt = op_v.front();
            op_v.pop();
            if(!op_v.empty()) {
                if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end()) {
                    if(aliases.count(tt) == 0) {
                        aliases[tt] = op_v.front();
                    };
                }
                else {
                    while(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end() && !op_v.empty()) {
                        op_v.pop();
                    };
                };
            };
        };
        if(!op_v.empty())
            op_v.pop();
    };

    op_v = op_value;
    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            field_names.insert(op_v.front());
        };
        op_v.pop();
    };

    for (auto it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };

    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;

    CudaSet *b, *c;

    if(a->segCount <= 1)
        setSegments(a, op_vx);
    allocColumns(a, op_vx);

    unsigned int cycle_count;
    if(a->filtered)
        cycle_count = varNames[a->source_name]->segCount;
    else
        cycle_count = a->segCount;

    size_t ol_count = a->mRecCount, cnt;
    a->hostRecCount = a->mRecCount;
    b = new CudaSet(0, col_count);
    b->name = "tmp b in select";
    bool c_set = 0;

    //size_t tmp_size = a->mRecCount;
    //if(a->segCount > 1)
    //    tmp_size = a->maxRecs;

    vector<thrust::device_vector<int_type> > distinct_val; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_hash; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_tmp;

    /* for(unsigned int i = 0; i < distinct_cnt; i++) {
         distinct_tmp.push_back(thrust::device_vector<int_type>(tmp_size));
         distinct_val.push_back(thrust::device_vector<int_type>());
         distinct_hash.push_back(thrust::device_vector<int_type>());
     };
    */

    bool one_liner;	
	if (grp_cnt != 0)
		phase_copy = 1;	

    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
        if(verbose)
            cout << "segment " << i << " select mem " << getFreeMem() << endl;
        std::clock_t start3 = std::clock();

        cnt = 0;
        copyColumns(a, op_vx, i, cnt);
		
        if(a->mRecCount) {
            if (grp_cnt != 0) {			
				bool not_srt_and_eq = 0;
				stack<string> op_vv(op_v2);
				while(!op_vv.empty()) {
					if(!min_max_eq[op_vv.top()])
						not_srt_and_eq = 1;
					op_vv.pop();	
				};
				if(not_srt_and_eq) {
					order_inplace(a, op_v2, field_names, 1);	
					a->GroupBy(op_v2);				
				}
				else {
					if(a->grp.size() != 1)
						a->grp.resize(1);	
					a->grp[0] = 1;
					a->grp_count = 1;	
				};	
            }
			else
				a->grp_count = 0;			
			
			copyFinalize(a, op_vx,0);
			
					
            one_liner = select(op_type,op_value,op_nums, op_nums_f, op_nums_precision, a,b, distinct_tmp);	

            if(i == 0)
                std::reverse(b->columnNames.begin(), b->columnNames.end());


            if (!c_set && b->mRecCount > 0) {
                c = new CudaSet(0, col_count);
                create_c(c,b);
                c_set = 1;
                c->name = s;
            };			

            if (grp_cnt && cycle_count > 1  && b->mRecCount > 0) {
                add(c,b,op_v3, aliases, distinct_tmp, distinct_val, distinct_hash, a);
            }
            else {
                //copy b to c
                unsigned int c_offset = c->mRecCount;
                c->resize(b->mRecCount);

                for(unsigned int j=0; j < b->columnNames.size(); j++) {
                    if (b->type[b->columnNames[j]] == 0) {
                        thrust::copy(b->d_columns_int[b->columnNames[j]].begin(), b->d_columns_int[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_int[b->columnNames[j]].begin() + c_offset);
                    }
                    else if (b->type[b->columnNames[j]] == 1) {
                        thrust::copy(b->d_columns_float[b->columnNames[j]].begin(), b->d_columns_float[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_float[b->columnNames[j]].begin() + c_offset);
                    };
                };
            };
			//std::cout<< "add time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) <<  '\n';				
        };
        std::cout<< "cycle sel time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
    };
	phase_copy = 0;

    a->mRecCount = ol_count;
    a->mRecCount = a->hostRecCount;
    a->deAllocOnDevice();
    b->deAllocOnDevice();
	a->grp.resize(0);
	a->grp.shrink_to_fit();
	for(auto i = 0; i < alloced_mem.size(); i++) {
		hipFree(alloced_mem[i]);
		alloced_mem.pop_back();
	};

    if(!c_set) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        c->name = s;
        clean_queues();
        return;
    };
	
    if (grp_cnt) {
        count_avg(c, distinct_hash);
    }
    else {
        if(one_liner) {
            count_simple(c);
        };
    };
	
    c->maxRecs = c->mRecCount;
    c->hostRecCount = c->mRecCount;
    c->string_map = b->string_map;
    c->name = s;
    c->keep = 1;
    if(verbose)
        cout << "select res " << c->mRecCount << endl;
		
    size_t tot_size = c->maxRecs*8*c->columnNames.size();	
    if (getFreeMem() < tot_size*3) {	
        c->segCount = ((tot_size*3)/getFreeMem() + 1);
        c->maxRecs = c->hostRecCount - (c->hostRecCount/c->segCount)*(c->segCount-1);
	};		

    clean_queues();
    varNames[s] = c;
    b->free();
    varNames[s]->keep = 1;

    if(mystat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(mystat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
    if(verbose)
        std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';

}


void emit_insert(const char *f, const char* s) {
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(f) == mystat.end() && data_dict.count(f) == 0) {
            process_error(2, "Insert : couldn't find variable " + string(f));
        };
        if (mystat.find(s) == mystat.end() && data_dict.count(s) == 0) {
            process_error(2, "Insert : couldn't find variable " + string(s) );
        };
        check_used_vars();
        mystat[f] = statement_count;
        mystat[s] = statement_count;
        clean_queues();
        return;
    };


    if(varNames.find(f) == varNames.end() || varNames.find(s) == varNames.end()) {
        clean_queues();
        return;
    };

    if(verbose)
        cout << "INSERT " << f << " " << s << endl;
    insert_records(f,s);
    clean_queues();


};


void emit_delete(const char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(f) == mystat.end()  && data_dict.count(f) == 0) {
            process_error(2, "Delete : couldn't find variable " + string(f));
        };
        mystat[f] = statement_count;
        check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    delete_records(f);
    cout << "DELETE " << f <<  endl;
    clean_queues();

}

void emit_case()
{
    op_case = 1;
    if (scan_state == 1)
        cout << "emit case " << endl;
    //extract releveant values and pass to modified filter
    // get a bool vector back
    /*						while(!op_type.empty())
    						{
    						cout << "CASE type " << op_type.front() << endl;
    						op_type.pop();
    						}
    */
}



void emit_create_index(const char *index_name, const char *table, const char *column)
{
	if (scan_state != 0) {
		FILE *f;
		string s1(table);
		string s3 = s1 + ".key";
		f = fopen(s3.c_str(), "w");
		fputs(column,f);
		fclose(f);
   };
}


void emit_create_interval(const char *interval_name, const char *table, const char *lcolumn, const char *rcolumn)
{
	if (scan_state != 0) {
		FILE *f;
		string s1(table);
		string s3 = s1 + ".interval";
		f = fopen(s3.c_str(), "w");
		fputs(lcolumn,f);
		fputc('|',f);
		fputs(rcolumn,f);
		fclose(f);
   };
}



void emit_create_bitmap_index(const char *index_name, const char *ltable, const char *rtable, const char *rcolumn, const char *lid, const char *rid)
{
    statement_count++;
    if (scan_state == 0) {
        emit_name(rcolumn);
        emit_sel_name(rcolumn);
        emit_name(lid);
        emit_name(rid);
        check_used_vars();
		mystat[rtable] = std::numeric_limits<unsigned int>::max();
        mystat[ltable] = std::numeric_limits<unsigned int>::max();		
    }
    else {
		cout << ltable << " " << rtable << " " << rid << " " << lid << endl;
        emit_name(rcolumn);
        emit_sel_name(rcolumn);
        emit_name(lid);
        emit_name(rid);
        check_used_vars();	
		
        if(varNames.find(ltable) == varNames.end())
            cout << "Couldn't find  " << ltable << endl;
        if(varNames.find(rtable) == varNames.end())
            cout << "Couldn't find  " << rtable << endl;
		
				
        CudaSet* left = varNames.find(ltable)->second;
		CudaSet* right = varNames.find(rtable)->second;
		
		queue<string> op_vx;
		op_vx.push(rcolumn);op_vx.push(rid);
		allocColumns(right, op_vx);		
		right->CopyColumnToGpu(rid, 0, 0);
		right->CopyColumnToGpu(rcolumn, 0, 0);
		op_vx.pop();op_vx.pop();
		op_vx.push(lid);
		allocColumns(left, op_vx);		
		
        for(int i = 0; i < left->segCount; i++) {
			
			left->CopyColumnToGpu(lid, i, 0);			
	
			thrust::device_vector<unsigned int> output(left->mRecCount);
			thrust::lower_bound(right->d_columns_int[rid].begin(), right->d_columns_int[rid].begin() + right->mRecCount, 
							    left->d_columns_int[lid].begin(), left->d_columns_int[lid].begin() + left->mRecCount, output.begin());

            string str = std::string(ltable) + std::string(".") + std::string(rtable) + std::string(".") + std::string(rcolumn) + std::string(".") + to_string(i);

			thrust::device_vector<int_type> res(left->mRecCount);
			thrust::host_vector<int_type> res_h(left->mRecCount);
			
            if(right->type[rcolumn] == 0) {
                thrust::gather(output.begin(), output.begin() + left->mRecCount,  right->d_columns_int[rcolumn].begin() , res.begin());
                thrust::copy(res.begin(), res.begin() + left->mRecCount, res_h.begin());				
                compress_int(str, res_h);
            }
            else if(right->type[rcolumn] == 1) {
            }
            else { //strings
                string f1 = right->load_file_name + "." + rcolumn + ".0.hash"; //need to change it in case if there are dimensions tables larger than 1 segment ?
                FILE* f = fopen(f1.c_str(), "rb" );
                unsigned int cnt;
                fread(&cnt, 4, 1, f);
				if(res_h.size() < cnt)
					res_h.resize(cnt);
				if(res.size() < cnt)
					res.resize(cnt);				
                fread(res_h.data(), cnt*8, 1, f);
				res = res_h;
                fclose(f);

				thrust::device_vector<int_type> output1(left->mRecCount);
				thrust::gather(output.begin(), output.begin() + left->mRecCount ,
								res.begin(), output1.begin());				
							
                thrust::copy(output1.begin(), output1.begin() + left->mRecCount, res_h.begin());
                compress_int(str, res_h);
            };
        };		
    };
}

void emit_display(const char *f, const char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(f) == mystat.end() && data_dict.count(f) == 0) {
            process_error(2, "Filter : couldn't find variable " + string(f) );
        };
        mystat[f] = statement_count;
   		if(filter_var.find(f) != filter_var.end()) 
			mystat[filter_var[f]] = statement_count;
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;
    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Display(limit, 0, 1);
    clean_queues();
    if(mystat[f] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };

}


void emit_filter(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(f) == mystat.end() && data_dict.count(f) == 0) {
            process_error(1, "Filter : couldn't find variable " + string(f));
        };
        mystat[s] = statement_count;
        mystat[f] = statement_count;
		filter_var[s] = f;
        // check possible use of other variables in filters
        queue<string> op(op_value);
        while(!op.empty()) {
            size_t pos1 = op.front().find_first_of(".", 0);
            if(pos1 != string::npos) {
                mystat[op.front().substr(0,pos1)] = statement_count;
            };
            op.pop();
        };

        check_used_vars();
        clean_queues();
        return;
    };

	
    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;

    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
        if(verbose)
            cout << "INLINE FILTER " << f << endl;
        b = a->copyDeviceStruct();

        b->name = s;
        b->sorted_fields = a->sorted_fields;
        b->presorted_fields = a->presorted_fields;
        //save the stack
        b->fil_s = s;
        b->fil_f = f;
        b->fil_type = op_type;
        b->fil_value = op_value;
        b->fil_nums = op_nums;		
        b->fil_nums_f = op_nums_f;		
		b->fil_nums_precision = op_nums_precision;
        b->filtered = 1;
        b->tmp_table = a->tmp_table;
        b->string_map = a->string_map;
        if(a->filtered) {

            b->source_name = a->source_name;
            b->fil_f = a->fil_f;
            while(!a->fil_value.empty()) {
                b->fil_value.push(a->fil_value.front());
                a->fil_value.pop();
            };

            while(!a->fil_type.empty()) {
                b->fil_type.push(a->fil_type.front());
                a->fil_type.pop();
            };
            b->fil_type.push("AND");

            while(!a->fil_nums.empty()) {
                b->fil_nums.push(a->fil_nums.front());
                a->fil_nums.pop();
            };
			
			while(!a->fil_nums_precision.empty()) {
				b->fil_nums_precision.push(a->fil_nums_precision.front());
				a->fil_nums_precision.pop();				
			};	

            while(!a->fil_nums_f.empty()) {
                b->fil_nums_f.push(a->fil_nums_f.front());
                a->fil_nums_f.pop();
            };
            a->filtered = 0;
            varNames.erase(f);
        }
        else
            b->source_name = f;
        b->maxRecs = a->maxRecs;
        b->prm_d.resize(a->maxRecs);
    };
    b->hostRecCount = a->hostRecCount;
    clean_queues();


    if (varNames.count(s) > 0)
        varNames[s]->free();
    varNames[s] = b;

    if(mystat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
}

void emit_store(const char *s, const char *f, const char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(s) == mystat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s) );
        };
        mystat[s] = statement_count;
		if(filter_var.find(f) != filter_var.end()) 
			mystat[filter_var[f]] = statement_count;
        clean_queues();
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;
    if(verbose)
        cout << "STORE: " << s << " " << f << " " << sep << endl;

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0, 0);

    if(mystat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };
};


void emit_store_binary(const char *s, const char *f, const bool append)
{
    statement_count++;
    if (scan_state == 0) {
        if (mystat.find(s) == mystat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s));
        };
        mystat[s] = statement_count;
		if(filter_var.find(f) != filter_var.end()) 
			mystat[filter_var[f]] = statement_count;
        clean_queues();
        return;
    };
	
	cout << "Append " << append << endl;

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(mystat[f] == statement_count)
        a->deAllocOnDevice();

    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
    total_count = 0;
    total_segments = 0;
	a->maxRecs = 0;
	
    if(fact_file_loaded) {
        a->Store(f,"", limit, 1, append);
    }
    else {
        FILE* file_p;
        if(a->text_source) {
            file_p = fopen(a->load_file_name.c_str(), "rb");
            if (!file_p) {
                process_error(2, "Could not open file " + a->load_file_name );
            };
        };		

		thrust::device_vector<char> d_readbuff;
		thrust::device_vector<char*> dest(a->mColumnCount);
		thrust::device_vector<unsigned int> ind(a->mColumnCount);
		thrust::device_vector<unsigned int> dest_len(a->mColumnCount);	
		
        while(!fact_file_loaded) {
            if(verbose)
                cout << "LOADING " << a->load_file_name << " mem: " << getFreeMem() << endl;
            if(a->text_source)
                fact_file_loaded = a->LoadBigFile(file_p, d_readbuff, dest, ind, dest_len);
			if(a->maxRecs < a->mRecCount)
				a->maxRecs = a->mRecCount;
            a->Store(f,"", limit, 1, append);
        };
    };
    a->writeSortHeader(f);

    if(mystat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};


void emit_load_binary(const char *s, const char *f, const int d)
{
	
    statement_count++;
    if (scan_state == 0) {				
        mystat[s] = statement_count;
        return;
    };	
	
    if(verbose)
        printf("BINARY LOAD: %s %s \n", s, f);	
			
	std::clock_t start1 = std::clock();	
    CudaSet *a;
    unsigned int segCount, maxRecs;
    string f1(f);
    f1 += "." + namevars.front() + ".header";

    FILE* ff = fopen(f1.c_str(), "rb");
    if(!ff) {
        process_error(2, "Couldn't open file " + f1);
    };
    size_t totRecs;
    fread((char *)&totRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);


    if(verbose)
        cout << "Reading " << totRecs << " records" << endl;

    a = new CudaSet(namevars, typevars, sizevars, cols, totRecs, f, maxRecs);

    a->segCount = segCount;
    a->keep = true;
    a->name = s;
    varNames[s] = a;

    if(mystat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
	std::cout<< "load time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}


void emit_load(const char *s, const char *f, const int d, const char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        mystat[s] = statement_count;
        return;
    };

    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count);
    a->keep = true;
    a->not_compressed = 1;
    a->load_file_name = f;	
    a->separator = sep;
    varNames[s] = a;
    fact_file_loaded = 0;

    if(mystat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };
}


void emit_show_tables()
{
    if (scan_state == 1) {
        for (auto it=data_dict.begin() ; it != data_dict.end(); ++it ) {
            cout << (*it).first << endl;
        };
    };

    return;
}

void emit_drop_table(const char* table_name)
{
    if (scan_state == 1) {

        map<string, map<string, col_data> >::iterator iter;
        if((iter = data_dict.find(table_name)) != data_dict.end()) {
            auto s = (*iter).second;
            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                int seg = 0;
                string f_name = (*iter).first + "." + (*it).first + "." + to_string(seg);
                while(!remove(f_name.c_str())) {
                    seg++;
                    f_name = (*iter).first + "." + (*it).first + "." + to_string(seg);
                };
                f_name = (*iter).first + "." + (*it).first + ".header";
                remove(f_name.c_str());
            };
        };
        string s_name = (*iter).first + ".presort";
        remove(s_name.c_str());
        s_name = (*iter).first + ".sort";
        remove(s_name.c_str());

        if(data_dict.find(table_name) != data_dict.end()) {
            data_dict.erase(table_name);
        };
        save_dict = 1;
    };

    return;
}


void emit_describe_table(const char* table_name)
{
    if (scan_state == 1) {
        map<string, map<string, col_data> >::iterator iter;
        if((iter = data_dict.find(table_name)) != data_dict.end()) {
            auto s = (*iter).second;
            for (auto it=s.begin() ; it != s.end(); ++it ) {
                if ((*it).second.col_type == 0) {
					if((*it).second.col_length) {
						if((*it).second.col_length != UINT_MAX)
							cout << (*it).first << " decimal with precision of " << (*it).second.col_length << endl;
						else
							cout << (*it).first << " timestamp" << endl;
					}	
					else	
						cout << (*it).first << " integer" << endl;
                }
                else if ((*it).second.col_type == 1) {
                    cout << (*it).first << " float" << endl;
                }
                else if ((*it).second.col_type == 3) {
                    cout << (*it).first << " decimal" << endl;
                }
                else {
                    cout << (*it).first << " char(" << (*it).second.col_length << ")" << endl;
                };
            };
        };
    };
    return;
}


void yyerror(char *s, ...)
{
    extern int yylineno;
    extern char *yytext;

    fprintf(stderr, "%d: error: ", yylineno);
    cout << yytext << endl;
    error_cb(1, s);
}


void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
	while(!op_nums_precision.empty()) op_nums_precision.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();
    while(!op_sort.empty()) op_sort.pop();
    while(!op_presort.empty()) op_presort.pop();
    while(!join_type.empty()) join_type.pop();
	while(!join_eq_type.empty()) join_eq_type.pop();
		
    op_case = 0;
    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = 0;
    distinct_cnt = 0;
    join_tab_cnt = 0;
    tab_cnt = 0;
    join_and_cnt.clear();
}

void load_vars()
{	
    if(used_vars.size() == 0) {
        //cout << "Error, no valid column names have been found " << endl;
        //exit(0);
    }
    else {
        for (auto it=used_vars.begin(); it != used_vars.end(); ++it ) {

            while(!namevars.empty()) namevars.pop();
            while(!typevars.empty()) typevars.pop();
            while(!sizevars.empty()) sizevars.pop();
            while(!cols.empty()) cols.pop();
            if(mystat.count((*it).first) != 0) {
                auto c = (*it).second;
                for (auto sit=c.begin() ; sit != c.end(); ++sit ) {
                    //cout << "name " << (*sit).first << " " << data_dict[(*it).first][(*sit).first].col_length << endl;
                    namevars.push((*sit).first);
                    if(data_dict[(*it).first][(*sit).first].col_type == 0) {
						if(data_dict[(*it).first][(*sit).first].col_length == 0) {
							typevars.push("int");
						}	
						else {
							if(data_dict[(*it).first][(*sit).first].col_length == UINT_MAX)
								typevars.push("timestamp");
							else
								typevars.push("decimal");
						}	
					}	
                    else if(data_dict[(*it).first][(*sit).first].col_type == 1)
                        typevars.push("float");
                    else typevars.push("char");
                    sizevars.push(data_dict[(*it).first][(*sit).first].col_length);
                    cols.push(0);
                };
                emit_load_binary((*it).first.c_str(), (*it).first.c_str(), 0);
            };
        };
    };	
}




void process_error(int severity, string err) {
    switch (severity) {
    case 1:
        err = "(Warning) " + err;
        break;
    case 2:
        err = "(Fatal) " + err;
        break;
    default:
        err = "(Aborting) " + err;
        break;
    }
    error_cb(severity, err.c_str());            // send the error to the c based callback
}


void alenkaInit(char ** av)
{
    process_count = 1000000000;
    verbose = 0;
    scan_state = 1;
    statement_count = 0;
    clean_queues();	
    //context = CreateCudaDevice(0, nullptr, true);
}


void alenkaClose()
{
    statement_count = 0;

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
}




