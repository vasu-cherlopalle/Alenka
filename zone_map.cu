#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "zone_map.h"

void process_error(int severity, string err);   // this should probably live in a utils header file

bool fh_equal_to(const float_type x, const float_type y)
{
	return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
}


bool fh_less(const float_type x, const float_type y)
{
	return ((y-x) > EPSILON);
}


bool fh_greater(const float_type x, const float_type y)
{
	return ((x-y) > EPSILON);
}


bool fh_greater_equal_to(const float_type x, const float_type y)
{
	return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}


bool fh_less_equal_to(const float_type x, const float_type y)
{
	return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}

char host_logical_and(char column1, char column2)
{
	//cout << "AND " << column1 << " " << column2 << endl;
	if (column1 == 'A' && column2 == 'A')
		return 'A';
	else
		if (column1 == 'N' || column2 == 'N') {
			return 'N';
		}
		else
			return 'R';

}


char host_logical_or(char column1, char column2)
{
	//cout << "OR " << column1 << " " << column2 << endl;
	if (column1 == 'A' && column2 == 'A')
		return 'A';
	else
		if (column1 == 'N' && column2 == 'N')
			return 'N';
		else
			return 'R';

}



char host_compare(int_type s, int_type d, int_type op_type)
{
	char res = 'N';

	if (op_type == 2 && d>s ) // >
		res = 'A';
	else
		if (op_type == 1 && d<s)  // <
			res = 'A';
		else
			if (op_type == 6 && d>=s) // >=
				res = 'A';
			else
				if (op_type == 5 && d<=s)  // <=
					res = 'A';
				else
					if (op_type == 4 && d==s)// =
						res = 'A';
					else // !=
						if(d!=s)
							res = 'A';

	return res;
}


char host_compare(float_type s, float_type d, int_type op_type)
{
	char res = 'N';

	if (op_type == 2 && (d-s) > EPSILON) // >
		res = 'A';
	else
		if (op_type == 1 && (s-d) > EPSILON)  // <
			res = 'A';
		else
			if (op_type == 6 && ((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) // >=
				res = 'A';
			else
				if (op_type == 5 && ((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON)))  // <=
					res = 'A';
				else
					if (op_type == 4 && ((d-s) < EPSILON) && ((d-s) > -EPSILON))// =
						res = 'A';
					else // !=
						if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON)))
							res = 'A';

	return res;
}


char host_compare(int_type* column1, int_type d, int_type op_type)
{
	char res = 'R';

	//cout << "CMP " << column1[0] << " " << column1[1] << " " << d << " " << op_type << endl;

	if (op_type == 2) {   // >
		if (column1[1] <= d)
			res = 'N';
		else
			if (column1[0] > d)
				res = 'A';
	}
	else
		if (op_type == 1) { // <
			if (column1[0] >= d)
				res = 'N';
			else
				if (column1[1] < d)
					res = 'A';
		}
		else
			if (op_type == 6) {  // >=
				if (column1[1] < d)
					res = 'N';
				else
					if (column1[0] >= d)
						res = 'A';
			}
			else
				if (op_type == 5) { // <=
					if (column1[0] > d)
						res = 'N';
					else
						if (column1[1] <= d)
							res = 'A';
				}
				else
					if (op_type == 4 && column1[0] == d && column1[1] == d) { // =
						res = 'A';
					};
	//cout << "res " << res << endl;

	return res;
}

char host_compare(float_type* column1, float_type d, int_type op_type)
{
	char res = 'R';
	//cout << "CMP " << column1[0] << " " << column1[1] << " with " << d << endl;

	if (op_type == 2) { // >
		if(fh_less_equal_to(column1[1],d)) {
			res = 'N';
		}
		else
			if(fh_greater(column1[0],d)) {
				res = 'A';
			};
	}
	else
		if (op_type == 1) { // <
			if(fh_less(column1[1],d)) {
				res = 'A';
			}
			else
				if(fh_greater_equal_to(column1[0],d)) {
					res = 'N';
				};
		}
		else
			if (op_type == 6) { // >=
				if(fh_greater_equal_to(column1[0],d)) {
					res = 'A';
				}
				else
					if(fh_less(column1[1],d)) {
						res = 'N';
					};
			}
			else
				if (op_type == 5) { // <=
					if(fh_less_equal_to(column1[1],d)) {
						res = 'A';
					}
					else
						if(fh_greater(column1[0],d)) {
							res = 'N';
						};
				}
				else
					if (op_type == 4 && fh_equal_to(column1[0],d) && fh_equal_to(column1[1],d)) // =
						res = 'A';

	//cout << "res " << res << endl;
	return res;

}


char host_compare(int_type* column1, int_type* column2, int_type op_type)
{
	char res = 'R';

	if (op_type == 2) { // >
		if(column1[0] > column2[1])
			res = 'A';
		else
			if(column1[1] <= column2[0])
				res = 'N';
	}
	else
		if (op_type == 1) { // <
			if(column1[1] < column2[0])
				res = 'A';
			else
				if(column1[0] >= column2[1])
					res = 'N';
		}
		else
			if (op_type == 6) { // >=
				if(column1[0] >= column2[1])
					res = 'A';
				else
					if(column1[1] < column2[0])
						res = 'N';
			}
			else
				if (op_type == 5) { // <=
					if(column1[1] <= column2[0])
						res = 'A';
					else
						if(column1[0] > column2[1])
							res = 'N';
				}
				else
					if (op_type == 4  && column1[0] == column2[1] && column1[1] == column2[0]) // =
						res = 'A';

	return res;


}

char host_compare(float_type* column1, float_type* column2, int_type op_type)
{
	char res = 'R';

	if (op_type == 2) { // >
		if(fh_greater(column1[0],column2[1]))
			res = 'A';
		else
			if(fh_less_equal_to(column1[1],column2[0]))
				res = 'N';
	}
	else
		if (op_type == 1) { // <
			if(fh_less(column1[1],column2[0]))
				res = 'A';
			else
				if(fh_greater_equal_to(column1[0],column2[1]))
					res = 'N';
		}
		else
			if (op_type == 6) { // >=
				if(fh_greater_equal_to(column1[1],column2[0]))
					res = 'A';
				else
					if(fh_less(column1[1],column2[0]))
						res = 'N';
			}
			else
				if (op_type == 5) { // <=
					if(fh_less_equal_to(column1[1],column2[0]))
						res = 'A';
					else
						if(fh_greater(column1[0],column2[1]))
							res = 'N';
				}
				else
					if (op_type == 4  && fh_equal_to(column1[0], column2[1]) && fh_equal_to(column1[1],column2[0])) // =
						res = 'A';

	return res;
}


char host_compare(float_type* column1, int_type* column2, int_type op_type)
{
	char res = 'R';

	if (op_type == 2) { // >
		if(fh_greater(column1[0],(float_type)column2[1]))
			res = 'A';
		else
			if(fh_less_equal_to(column1[1],(float_type)column2[0]))
				res = 'N';
	}
	else
		if (op_type == 1) { // <
			if(fh_less(column1[1],(float_type)column2[0]))
				res = 'A';
			else
				if(fh_greater_equal_to(column1[0],(float_type)column2[1]))
					res = 'N';
		}
		else
			if (op_type == 6) { // >=
				if(fh_greater_equal_to(column1[1],(float_type)column2[0]))
					res = 'A';
				else
					if(fh_less(column1[1],(float_type)column2[0]))
						res = 'N';
			}
			else
				if (op_type == 5) { // <=
					if(fh_less_equal_to(column1[1],(float_type)column2[0]))
						res = 'A';
					else
						if(fh_greater(column1[0],(float_type)column2[1]))
							res = 'N';
				}
				else
					if (op_type == 4  && fh_equal_to(column1[0],(float_type) column2[1]) && fh_equal_to(column1[1],(float_type)column2[0])) // =
						res = 'A';

	return res;
}



float_type* host_op(int_type* column1, float_type* column2, string op_type, int reverse)
{

	float_type* temp = (float_type*)malloc(2*float_size);
	temp[0] = (float_type)column1[0];
	temp[1] = (float_type)column1[1];

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp[0] = temp[0] * column2[0];
			temp[1] = temp[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = temp[0] + column2[0];
				temp[1] = temp[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column2[0] - temp[0];
					temp[1] = column2[1] - temp[1];
				}
				else {
					temp[0] = column2[0] / temp[0];
					temp[1] = column2[1] / temp[1];
				}
	}
	else {
		if (op_type.compare("MUL") == 0) {
			temp[0] = temp[0] * column2[0];
			temp[1] = temp[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = temp[0] + column2[0];
				temp[1] = temp[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = temp[0] - column2[0];
					temp[1] = temp[1] - column2[1];
				}
				else {
					temp[0] = temp[0] / column2[0];
					temp[1] = temp[1] / column2[1];
				}
	};

	return temp;
}




int_type* host_op(int_type* column1, int_type* column2, string op_type, int reverse)
{
	int_type* temp = (int_type*)malloc(2*int_size);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * column2[0];
			temp[1] = column1[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + column2[0];
				temp[1] = column1[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column1[0] - column2[0];
					temp[1] = column1[1] - column2[1];
				}
				else {
					temp[0] = column1[0] / column2[0];
					temp[1] = column1[1] / column2[1];
				}
	}
	else  {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * column2[0];
			temp[1] = column1[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + column2[0];
				temp[1] = column1[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column2[0] - column1[0];
					temp[1] = column2[1] - column1[1];
				}
				else {
					temp[0] = column2[0] / column1[0];
					temp[1] = column2[1] / column1[1];
				}
	}

	return temp;

}

float_type* host_op(float_type* column1, float_type* column2, string op_type, int reverse)
{
	float_type* temp = (float_type*)malloc(2*float_size);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * column2[0];
			temp[1] = column1[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + column2[0];
				temp[1] = column1[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column1[0] - column2[0];
					temp[1] = column1[1] - column2[1];
				}
				else {
					temp[0] = column1[0] / column2[0];
					temp[1] = column1[1] / column2[1];
				}
	}
	else  {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * column2[0];
			temp[1] = column1[1] * column2[1];
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + column2[0];
				temp[1] = column1[1] + column2[1];
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column2[0] - column1[0];
					temp[1] = column2[1] - column1[1];
				}
				else {
					temp[0] = column2[0] / column1[0];
					temp[1] = column2[1] / column1[1];
				}
	}

	return temp;

}

int_type* host_op(int_type* column1, int_type d, string op_type, int reverse)
{
	int_type* temp = (int_type*)malloc(2*int_size);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * d;
			temp[1] = column1[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + d;
				temp[1] = column1[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column1[0] - d;
					temp[1] = column1[1] - d;
				}
				else {
					temp[0] = column1[0] / d;
					temp[1] = column1[1] / d;
				}
	}
	else {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * d;
			temp[1] = column1[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + d;
				temp[1] = column1[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = d - column1[0];
					temp[1] = d - column1[1];
				}
				else {
					temp[0] = d / column1[0];
					temp[1] = d / column1[1];
				}

	};
	return temp;

}

float_type* host_op(int_type* column1, float_type d, string op_type, int reverse)
{
	float_type* temp = (float_type*)malloc(2*float_size);
	temp[0] = (float_type)column1[0];
	temp[1] = (float_type)column1[1];

	float_type* temp1 = (float_type*)malloc(2*float_size);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp1[0] = temp[0] * d;
			temp1[1] = temp[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp1[0] = temp[0] + d;
				temp1[1] = temp[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp1[0] = temp[0] - d;
					temp1[1] = temp[1] - d;
				}
				else {
					temp1[0] = temp[0] / d;
					temp1[1] = temp[1] / d;
				}
	}
	else  {
		if (op_type.compare("MUL") == 0) {
			temp1[0] = temp[0] * d;
			temp1[1] = temp[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp1[0] = temp[0] + d;
				temp1[1] = temp[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp1[0] = d - temp[0];
					temp1[1] = d - temp[1];
				}
				else {
					temp1[0] = d / temp[0];
					temp1[1] = d / temp[1];
				}
	};

	free(temp);
	return temp1;

}



float_type* host_op(float_type* column1, float_type d, string op_type,int reverse)
{
	float_type* temp = (float_type*)malloc(2*float_size);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * d;
			temp[1] = column1[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + d;
				temp[1] = column1[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = column1[0] - d;
					temp[1] = column1[1] - d;
				}
				else {
					temp[0] = column1[0] / d;
					temp[1] = column1[1] / d;
				}
	}
	else {
		if (op_type.compare("MUL") == 0) {
			temp[0] = column1[0] * d;
			temp[1] = column1[1] * d;
		}
		else
			if (op_type.compare("ADD") == 0) {
				temp[0] = column1[0] + d;
				temp[1] = column1[1] + d;
			}
			else
				if (op_type.compare("MINUS") == 0) {
					temp[0] = d - column1[0];
					temp[1] = d - column1[1];
				}
				else {
					temp[0] = d / column1[0];
					temp[1] = d / column1[1];
				}
	};

	return temp;

}


unsigned int precision_func(unsigned int& p1, unsigned int& p2, string op) {

	if (op.compare("DIV") != 0 ) {
		unsigned int res;
		if (op.compare("MUL") != 0 ) {
			if(p1 > p2) {
				res = p1;
				p2 = p1-p2;
				p1 = 0;
			}
			else {
				res = p1;
				p1 = p2-p1;
				p2 = 0;
			};
			return res;
		}
		else {
			//std::swap(p1,p2);
			res = p1+p2;
			p1 = 0;
			p2 = 0;
			return res;
		};
	}
	else {
		if(p1 == p2) {
			p1 = p1+4;
			p2 = 0;
			return p1;
		}
		else {
			if(p1 > p2) {
				p1 = p1 + (p1-p2) + 4;
				p2 = 0;
				return p1;
			}
			else {
				p2 = p2 + (p2-p1) + 4;
				p1 = 0;
				return p2;
			}
		}
	};
}



//CudaSet a contains two records - with all minimum and maximum values of the segment
//We need to determine if this segment needs to be processed
//The check takes place in host's memory

char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a, unsigned int segment)
{

	stack<string> exe_type;
	stack<string> exe_value;
	stack<int_type*> exe_vectors;
	stack<float_type*> exe_vectors_f;
	stack<int_type> exe_nums;
	stack<char> bool_vectors;
	stack<unsigned int> exe_precision;
	string  s1, s2, s1_val, s2_val;
	int_type n1, n2, res;

	if(a->not_compressed)
		return 'R';

	//first we need to set all host arrays [0] and [1] of t to min and max values of appropriate files
	set<string> uniques;
	queue<string> fields(op_value);
	CudaSet *t;
	FILE* f;
	unsigned int cnt;
	string f1;


	while(!fields.empty()) {
		if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{

			if(a->filtered)
				t = varNames[a->source_name];
			else
				t = a;

			// copy t min and max values to a only if int, decimal or float
			if(t->type[fields.front()] <= 1) {

				f1 = t->load_file_name + "." + fields.front() + "." + to_string(segment);
				f = fopen (f1.c_str() , "rb" );
				if(!f) {
					cout << "Error opening " << f1 << " file " << endl;
					exit(0);
				};

				fread((char *)&cnt, 4, 1, f);
				if (t->type[fields.front()] == 0) {
					a->h_columns_int[fields.front()].resize(2);
					fread((char *)&a->h_columns_int[fields.front()][0], 8, 1, f);
					fread((char *)&a->h_columns_int[fields.front()][1], 8, 1, f);
					fseek(f, 8+cnt, SEEK_CUR);
					fread((char *)&a->mRecCount, 4, 1, f);
					//cout << endl << "ZONE " << a->mRecCount << endl;
					fread((char *)&cnt, 4, 1, f);
					//cout << "file " << f1 << " " << segment << " " << a->h_columns_int[fields.front()][0] << ":" << a->h_columns_int[fields.front()][1] << endl;
				}
				else  {
					long long int t;
					a->h_columns_float[fields.front()].resize(2);
					fread((char *)&t, 8, 1, f);
					a->h_columns_float[fields.front()][0] = (float_type)t/100.0;
					fread((char *)&t, 8, 1, f);
					a->h_columns_float[fields.front()][1] = (float_type)t/100.0;
					//cout << "file " << f1 << " " << segment << " " << a->h_columns_float[a->type_index[colIndex]][0] << ":" << a->h_columns_float[a->type_index[colIndex]][1] << endl;
				};
				fclose(f);
			};
		};
		uniques.insert(fields.front());
		fields.pop();
	};


	for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

		string ss = op_type.front();
		//cout << ss << endl;

		if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("FLOAT") == 0
		        || ss.compare("STRING") == 0) {

			exe_type.push(ss);
			if (ss.compare("NUMBER") == 0) {
				exe_nums.push(op_nums.front());
				op_nums.pop();
				exe_precision.push(op_nums_precision.front());
				op_nums_precision.pop();
			}
			else
				if (ss.compare("NAME") == 0)  {
					if(var_exists(a, op_value.front())) {
						exe_value.push(op_value.front());
						op_value.pop();
					}
					else {
						process_error(1, "Couldn't find column " + op_value.front());
						//cout << "Couldn't find column " << op_value.front() << endl;
						//exit(0);
					};
				}
				else
					if (ss.compare("STRING") == 0) {
						exe_value.push(op_value.front());
						op_value.pop();
					}

		}
		else {
			if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
				// get 2 values from the stack
				s1 = exe_type.top();
				exe_type.pop();
				s2 = exe_type.top();
				exe_type.pop();


				if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
					n1 = exe_nums.top();
					exe_nums.pop();
					n2 = exe_nums.top();
					exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = precision_func(p1, p2, ss);
					exe_precision.push(pres);

					if(p1)
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2)
						n2 = n2*(unsigned int)pow(10,p2);

					if (ss.compare("ADD") == 0 )
						res = n1+n2;
					else
						if (ss.compare("MUL") == 0 )
							res = n1*n2;
						else
							if (ss.compare("DIV") == 0 )
								res = n1/n2;
							else
								res = n1-n2;

					exe_type.push("NUMBER");
					exe_nums.push(res);
				}



				else
					if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
						s1_val = exe_value.top();
						exe_value.pop();
						s2_val = exe_value.top();
						exe_value.pop();
						int_type val;
						int_type* t = get_host_vec(a, s1_val, exe_vectors);


						exe_type.push("NAME");
						exe_value.push("");
						exe_precision.push(0);
					}
					else
						if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = get_host_vec(a, s2_val, exe_vectors);
							//cout << "name " << s2_val << endl;


							exe_type.push("NAME");
							exe_value.push("");
							exe_precision.push(0);
						}

						else
							if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
								s1_val = exe_value.top();
								exe_value.pop();
								n1 = exe_nums.top();
								exe_nums.pop();
								auto p2 = exe_precision.top();
								exe_precision.pop();
								auto p1 = get_decimals(a, s1_val, exe_precision);
								int_type* t = get_host_vec(a, s1_val, exe_vectors);
								auto pres = precision_func(p1, p2, ss);
								exe_precision.push(pres);
								if(p1) {
									t[0] = t[0]*(unsigned int)pow(10,p1);
									t[1] = t[1]*(unsigned int)pow(10,p1);
								};
								if(p2) {
									n1 = n1*(unsigned int)pow(10,p2);
								};

								exe_type.push("NAME");
								exe_value.push("");
								exe_vectors.push(host_op(t,n1,ss,1));

							}
							else
								if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
									n1 = exe_nums.top();
									exe_nums.pop();
									s2_val = exe_value.top();
									exe_value.pop();
									auto p2 = exe_precision.top();
									exe_precision.pop();
									auto p1 = get_decimals(a, s2_val, exe_precision);

									int_type* t = get_host_vec(a, s2_val, exe_vectors);
									auto pres = precision_func(p1, p2, ss);
									exe_precision.push(pres);
									if(p1) {
										t[0] = t[0]*(unsigned int)pow(10,p1);
										t[1] = t[1]*(unsigned int)pow(10,p1);
									};
									if(p2) {
										n1 = n1*(unsigned int)pow(10,p2);
									};

									exe_type.push("NAME");
									exe_value.push("");
									exe_vectors.push(host_op(t,n1,ss,0));

								}
								else
									if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {

										s1_val = exe_value.top();
										exe_value.pop();
										s2_val = exe_value.top();
										exe_value.pop();

										auto p1 = get_decimals(a, s1_val, exe_precision);
										auto p2 = get_decimals(a, s2_val, exe_precision);

										int_type* t = get_host_vec(a, s1_val, exe_vectors);
										int_type* s3 = get_host_vec(a, s2_val, exe_vectors);;

										exe_type.push("NAME");
										exe_value.push("");
										auto pres = precision_func(p1, p2, ss);
										exe_precision.push(pres);
										if(p1) {
											t[0] = t[0]*(unsigned int)pow(10,p1);
											t[1] = t[1]*(unsigned int)pow(10,p1);
										};
										if(p2) {
											s3[0] = s3[0]*(unsigned int)pow(10,p2);
											s3[1] = s3[1]*(unsigned int)pow(10,p2);
										};

										exe_vectors.push(host_op(t,s3,ss,1));
										if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) == a->columnNames.end())
											delete [] t;
										if(std::find(a->columnNames.begin(), a->columnNames.end(), s2_val) == a->columnNames.end())
											delete [] s3;
									}
			}

			else
				if (ss.compare("CMP") == 0) {

					int_type cmp_type = op_nums.front();
					op_nums.pop();

					s1 = exe_type.top();
					exe_type.pop();
					s2 = exe_type.top();
					exe_type.pop();


					if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
						n1 = exe_nums.top();
						exe_nums.pop();
						n2 = exe_nums.top();
						exe_nums.pop();
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto pres = std::max(p1, p2);
						exe_precision.push(pres);
						exe_type.push("NAME");
						exe_value.push("");

						if(p1)
							n1 = n1*(unsigned int)pow(10,pres-p1);
						if(p2)
							n2 = n2*(unsigned int)pow(10,pres-p2);
						bool_vectors.push(host_compare(n1,n2,cmp_type));
					}

					else
						if (s1.compare("STRING") == 0 && s2.compare("NAME") == 0) {
							time_t tt;
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = get_host_vec(a, s2_val, exe_vectors);
							auto pos = s1_val.find("date()");
							bool_vectors.push('R');
							exe_type.push("NAME");
							exe_value.push("");
						}
						else
							if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
								s2_val = exe_value.top();
								exe_value.pop();
								s1_val = exe_value.top();
								exe_value.pop();
								int_type val;
								time_t tt;
								int_type* t = get_host_vec(a, s1_val, exe_vectors);

								bool_vectors.push('R');
								exe_type.push("NAME");
								exe_value.push("");
							}

							else
								if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
									n1 = exe_nums.top();
									exe_nums.pop();
									s1_val = exe_value.top();
									exe_value.pop();
									auto p2 = exe_precision.top();
									exe_precision.pop();

									auto p1 = get_decimals(a, s1_val, exe_precision);
									int_type* t = get_host_vec(a, s1_val, exe_vectors);
									auto pres = std::max(p1, p2);
									exe_precision.push(pres);

									if(p1) {
										t[0] = t[0]*(unsigned int)pow(10,pres-p1);
										t[1] = t[1]*(unsigned int)pow(10,pres-p1);
									};
									if(p2) {
										n1 = n1*(unsigned int)pow(10,pres-p2);
									};
									exe_type.push("NAME");
									exe_value.push("");
									bool_vectors.push(host_compare(t,n1,cmp_type));
								}
								else
									if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
										cmp_type = reverse_op(cmp_type);
										n1 = exe_nums.top();
										exe_nums.pop();
										s2_val = exe_value.top();
										exe_value.pop();
										auto p2 = exe_precision.top();
										exe_precision.pop();
										auto p1 = get_decimals(a, s2_val, exe_precision);

										int_type* t = get_host_vec(a, s2_val, exe_vectors);
										auto pres = std::max(p1, p2);
										exe_precision.push(pres);
										if(p1) {
											t[0] = t[0]*(unsigned int)pow(10,pres-p1);
											t[1] = t[1]*(unsigned int)pow(10,pres-p1);
										};
										if(p2) {
											n1 = n1*(unsigned int)pow(10,pres-p2);
										};
										exe_type.push("NAME");
										exe_value.push("");
										bool_vectors.push(host_compare(t,n1,cmp_type));
									}

									else  {
										return 'R';
									}
				}

				else
					if (ss.compare("AND") == 0) {
						char s3 = bool_vectors.top();
						bool_vectors.pop();
						char s2 = bool_vectors.top();
						bool_vectors.pop();
						exe_type.push("NAME");
						bool_vectors.push(host_logical_and(s2,s3));
					}
					else
						if (ss.compare("OR") == 0) {
							char s3 = bool_vectors.top();
							bool_vectors.pop();
							char s2 = bool_vectors.top();
							bool_vectors.pop();
							exe_type.push("NAME");
							bool_vectors.push(host_logical_or(s2,s3));
						}
						else {
							if(ss.compare("JOIN") == 0)
								process_error(2, "operation = is not valid");
							//cout << "operation = is not valid" << endl;
							else
								process_error(2, "operation " + string(ss)+ " is not valid");
							//cout << "operation " << ss << " is not valid" << endl;
							exit(0);	// never gets here
						}
		};
	};


	return bool_vectors.top();

}

