#include "hip/hip_runtime.h"
/*
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include <ctime>
#include <time.h>
#include "cm.h"
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"
#include "filter.h"
#include "callbacks.h"
#include "zone_map.h"

#ifdef _WIN64
#define atoll(S) _atoi64(S)
#define fseek(S, S1, S2) _fseeki64(S, S1, S2)
#include <windows.h>
#else
#include <unistd.h>
#endif

using namespace std;
using namespace thrust::placeholders;

size_t total_count = 0, total_max;
clock_t tot;
unsigned int total_segments = 0, old_segments;
size_t process_count;
size_t alloced_sz = 0;
bool fact_file_loaded = 1;
bool verbose;
bool interactive, ssd, delta, star;
unsigned int prs;
void* d_v = nullptr;
void* s_v = nullptr;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_type;
bool op_case = 0;
string grp_val;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<unsigned int> op_nums_precision;
queue<string> col_aliases;
map<string, map<string, col_data> > data_dict;
map<unsigned int, map<unsigned long long int, size_t> > char_hash;

map<string, char*> index_buffers;
map<string, unsigned long long int*> idx_vals;
map<string, char*> buffers;
map<string, size_t> buffer_sizes;
size_t total_buffer_size;
queue<string> buffer_names;

void* alloced_tmp;
bool alloced_switch = 0;
map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string, unsigned int> cpy_bits;
map<string, long long int> cpy_init_val;
char* readbuff = nullptr;
thrust::device_vector<unsigned int> rcol_matches;
thrust::device_vector<int_type> rcol_dev;

struct f_equal_to
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
	}
};


struct f_less
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return ((y-x) > EPSILON);
	}
};

struct f_greater
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return ((x-y) > EPSILON);
	}
};

struct f_greater_equal_to
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
	}
};

struct f_less_equal
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
	}
};

struct f_not_equal_to
{
	__host__ __device__
	bool operator()(const float_type x, const float_type y)
	{
		return ((x-y) > EPSILON) || ((x-y) < -EPSILON);
	}
};


struct long_to_float_type
{
	__host__ __device__
	float_type operator()(const int_type x)
	{
		return (float_type)x;
	}
};

template <typename T>
struct power_functor : public thrust::unary_function<T,T>
{
	unsigned int a;

	__host__ __device__
	power_functor(unsigned int a_) {
		a = a_;
	}

	__host__ __device__
	T operator()(T x)
	{
		return x*(unsigned int)pow((double)10,(double)a);
	}
};

struct is_zero
{
	__host__ __device__
	bool operator()(const int &x)
	{
		return x == 0;
	}
};


int get_utc_offset() {

	time_t zero = 24*60*60L;
	struct tm * timeptr;
	int gmtime_hours;

	/* get the local time for Jan 2, 1900 00:00 UTC */
	timeptr = localtime( &zero );
	gmtime_hours = timeptr->tm_hour;

	/* if the local time is the "day before" the UTC, subtract 24 hours
	  from the hours to get the UTC offset */
	if( timeptr->tm_mday < 2 )
		gmtime_hours -= 24;

	return gmtime_hours;

}

/*
  the utc analogue of mktime,
  (much like timegm on some systems)
*/
time_t tm_to_time_t_utc( struct tm * timeptr ) {

	/* gets the epoch time relative to the local time zone,
	and then adds the appropriate number of seconds to make it UTC */
	return mktime( timeptr ) + get_utc_offset() * 3600;

}


/*class power_functor {

    unsigned int a;

    public:

        power_functor(unsigned int a_) { a = a_; }

        __host__ __device__ int_type operator()(int_type x) const
        {
            return x*(unsigned int)pow((double)10,(double)a);
        }
};
*/


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void write_compressed_char(string file_name, unsigned int index, size_t mCount);
size_t getFreeMem();
size_t getTotalSystemMemory();
void process_error(int severity, string err);

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs)
	: mColumnCount(0), mRecCount(0)
{
	initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
	source = 1;
	text_source = 1;
	fil_f = nullptr;
	fil_s = nullptr;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name, unsigned int max)
	: mColumnCount(0),  mRecCount(0)
{
	maxRecs = max;
	initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
	source = 1;
	text_source = 0;
	fil_f = nullptr;
	fil_s = nullptr;
};

CudaSet::CudaSet(const size_t RecordCount, const unsigned int ColumnCount)
{
	initialize(RecordCount, ColumnCount);
	keep = false;
	source = 0;
	text_source = 0;
	fil_f = nullptr;
	fil_s = nullptr;
};


CudaSet::CudaSet(queue<string> op_sel, const queue<string> op_sel_as)
{
	initialize(op_sel, op_sel_as);
	keep = false;
	source = 0;
	text_source = 0;
	fil_f = nullptr;
	fil_s = nullptr;
};

CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
	initialize(a,b, op_sel, op_sel_as);
	keep = false;
	source = 0;
	text_source = 0;
	fil_f = nullptr;
	fil_s = nullptr;
};


CudaSet::~CudaSet()
{
	free();
};


void CudaSet::allocColumnOnDevice(string colname, size_t RecordCount)
{
	if (type[colname] != 1 ) {
		d_columns_int[colname].resize(RecordCount);
	}
	else
		d_columns_float[colname].resize(RecordCount);
};


void CudaSet::resize_join(size_t addRecs)
{
	mRecCount = mRecCount + addRecs;
	for(unsigned int i=0; i < columnNames.size(); i++) {
		if(type[columnNames[i]] != 1) {
			h_columns_int[columnNames[i]].resize(mRecCount);
		}
		else
			h_columns_float[columnNames[i]].resize(mRecCount);
	};
};


void CudaSet::resize(size_t addRecs)
{
	mRecCount = mRecCount + addRecs;
	for(unsigned int i=0; i < columnNames.size(); i++) {
		if(type[columnNames[i]] != 1) {
			h_columns_int[columnNames[i]].resize(mRecCount);
		}
		else {
			h_columns_float[columnNames[i]].resize(mRecCount);
		}
	};
};

void CudaSet::deAllocColumnOnDevice(string colname)
{
	if (type[colname] != 1 && !d_columns_int.empty() && d_columns_int.find(colname) != d_columns_int.end()) {
		if(d_columns_int[colname].size() > 0) {
			d_columns_int[colname].resize(0);
			d_columns_int[colname].shrink_to_fit();
		};
	}
	else
		if (type[colname] == 1 && !d_columns_float.empty()) {
			if (d_columns_float[colname].size() > 0) {
				d_columns_float[colname].resize(0);
				d_columns_float[colname].shrink_to_fit();
			};
		};
};

void CudaSet::allocOnDevice(size_t RecordCount)
{
	for(unsigned int i=0; i < columnNames.size(); i++)
		allocColumnOnDevice(columnNames[i], RecordCount);
};

void CudaSet::deAllocOnDevice()
{
	for(unsigned int i=0; i < columnNames.size(); i++) {
		deAllocColumnOnDevice(columnNames[i]);
	};

	if(prm_d.size()) {
		prm_d.resize(0);
		prm_d.shrink_to_fit();
	};

	for (auto it=d_columns_int.begin(); it != d_columns_int.end(); ++it ) {
		if(it->second.size() > 0) {
			it->second.resize(0);
			it->second.shrink_to_fit();
		};
	};

	for (auto it=d_columns_float.begin(); it != d_columns_float.end(); ++it ) {
		if(it->second.size() > 0) {
			it->second.resize(0);
			it->second.shrink_to_fit();
		};
	};

	if(filtered) { // dealloc the source
		if(varNames.find(source_name) != varNames.end()) {
			varNames[source_name]->deAllocOnDevice();
		};
	};
};


void CudaSet::resizeDeviceColumn(size_t RecCount, string colname)
{
	if (type[colname] != 1) {
		d_columns_int[colname].resize(RecCount);
	}
	else
		d_columns_float[colname].resize(RecCount);
};

void CudaSet::resizeDevice(size_t RecCount)
{
	for(unsigned int i=0; i < columnNames.size(); i++) {
		resizeDeviceColumn(RecCount, columnNames[i]);
	};
};

bool CudaSet::onDevice(string colname)
{
	if (type[colname] != 1) {
		if (!d_columns_int.empty() && d_columns_int[colname].size())
			return 1;
	}
	else
		if (!d_columns_float.empty() && d_columns_float[colname].size())
			return 1;
	return 0;
}

CudaSet* CudaSet::copyDeviceStruct()
{

	CudaSet* a = new CudaSet(mRecCount, mColumnCount);
	a->not_compressed = not_compressed;
	a->segCount = segCount;
	a->maxRecs = maxRecs;
	a->columnNames = columnNames;
	a->ts_cols = ts_cols;
	a->cols = cols;
	a->type = type;
	a->char_size = char_size;
	a->decimal = decimal;
	a->decimal_zeroes = decimal_zeroes;

	for(unsigned int i=0; i < columnNames.size(); i++) {
		if(a->type[columnNames[i]] == 0) {
			a->d_columns_int[columnNames[i]] = thrust::device_vector<int_type>();
			a->h_columns_int[columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
		}
		else
			if(a->type[columnNames[i]] == 1) {
				a->d_columns_float[columnNames[i]] = thrust::device_vector<float_type>();
				a->h_columns_float[columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
			}
			else {
				a->h_columns_char[columnNames[i]] = nullptr;
				a->d_columns_char[columnNames[i]] = nullptr;
			};
	};
	a->load_file_name = load_file_name;
	a->mRecCount = 0;
	return a;
}

int_type CudaSet::readSsdSegmentsFromFile(unsigned int segNum, string colname, size_t offset, thrust::host_vector<unsigned int>& prm_vh, CudaSet* dest)
{
	string f1 = load_file_name + "." + colname + "." + to_string(segNum);
	FILE* f = fopen(f1.c_str(), "rb" );
	if(!f) {
		cout << "Error opening " << f1 << " file " << endl;
		exit(0);
	};

	unsigned int cnt, bits;
	int_type lower_val;

	unsigned short int val_s_r[4096/2];
	char val_c_r[4096];
	unsigned int val_i_r[4096/4];
	unsigned long long int val_l_r[4096/8];
	unsigned int idx;
	bool idx_set = 0;

	fread(&cnt, 4, 1, f);
	fread(&lower_val, 8, 1, f);
	fseek(f, cnt - (8+4) + 32, SEEK_CUR);
	fread(&bits, 4, 1, f);
	//cout << "lower_val bits " << lower_val << " " << bits << endl;

	if(type[colname] == 0) {
		//cout << "lower_val bits " << lower_val << " " << bits << endl;

		for(unsigned int i = 0; i < prm_vh.size(); i++) {

			if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
				fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
				idx = prm_vh[i];
				idx_set = 1;

				if(bits == 8) {
					fread(&val_c_r[0], 4096, 1, f);
					dest->h_columns_int[colname][i + offset] = val_c_r[0];
				}
				else
					if(bits == 16) {
						fread(&val_s_r, 4096, 1, f);
						dest->h_columns_int[colname][i + offset] = val_s_r[0];
					}
				if(bits == 32) {
					fread(&val_i_r, 4096, 1, f);
					dest->h_columns_int[colname][i + offset] = val_i_r[0];
				}
				if(bits == 84) {
					fread(&val_l_r, 4096, 1, f);
					dest->h_columns_int[colname][i + offset] = val_l_r[0];
				}
			}
			else {
				if(bits == 8) {
					dest->h_columns_int[colname][i + offset] = val_c_r[prm_vh[i]-idx];
				}
				else
					if(bits == 16) {
						dest->h_columns_int[colname][i + offset] = val_s_r[prm_vh[i]-idx];
					}
				if(bits == 32) {
					dest->h_columns_int[colname][i + offset] = val_i_r[prm_vh[i]-idx];
				}
				if(bits == 84) {
					dest->h_columns_int[colname][i + offset] = val_l_r[prm_vh[i]-idx];
				}
			};
		};
	}
	else
		if(type[colname] == 1) {

			for(unsigned int i = 0; i < prm_vh.size(); i++) {
				if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
					fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
					idx = prm_vh[i];
					idx_set = 1;
					fread(val_c_r, 4096, 1, f);
					memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[0], bits/8);
				}
				else {
					memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[(prm_vh[i]-idx)*(bits/8)], bits/8);
				};
			};

		}
		else {
			//no strings in fact tables
		};
	fclose(f);
	return lower_val;
}

int_type CudaSet::readSsdSegmentsFromFileR(unsigned int segNum, string colname, thrust::host_vector<unsigned int>& prm_vh, thrust::host_vector<unsigned int>& dest)
{
	string f1 = load_file_name + "." + colname + "." + to_string(segNum);
	FILE* f = fopen(f1.c_str(), "rb" );
	if(!f) {
		cout << "Error opening " << f1 << " file " << endl;
		exit(0);
	};

	unsigned int cnt, bits;
	int_type lower_val;
	fread(&cnt, 4, 1, f);
	fread(&lower_val, 8, 1, f);
	fseek(f, cnt - (8+4) + 32, SEEK_CUR);
	fread(&bits, 4, 1, f);

	unsigned short int val_s_r[4096/2];
	char val_c_r[4096];
	unsigned int val_i_r[4096/4];
	unsigned long long int val_l_r[4096/8];
	unsigned int idx;
	bool idx_set = 0;

	for(unsigned int i = 0; i < prm_vh.size(); i++) {

		if(!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
			fseek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
			idx = prm_vh[i];
			idx_set = 1;

			if(bits == 8) {
				fread(val_c_r, 4096, 1, f);
				dest[i] = val_c_r[0];
			}
			else
				if(bits == 16) {
					fread(val_s_r, 4096, 1, f);
					dest[i] = val_s_r[0];
				}
			if(bits == 32) {
				fread(val_i_r, 4096, 1, f);
				dest[i] = val_i_r[0];
			}
			if(bits == 84) {
				fread(val_l_r, 4096, 1, f);
				dest[i] = val_l_r[0];
			}
		}
		else {
			if(bits == 8) {
				dest[i] = val_c_r[prm_vh[i]-idx];
			}
			else
				if(bits == 16) {
					dest[i] = val_s_r[prm_vh[i]-idx];
				}
			if(bits == 32) {
				dest[i] = val_i_r[prm_vh[i]-idx];
			}
			if(bits == 84) {
				dest[i] = val_l_r[prm_vh[i]-idx];
			}
		};
	};
	fclose(f);
	return lower_val;
}

std::clock_t tot_disk;

void CudaSet::readSegmentsFromFile(unsigned int segNum, string colname)
{
	string f1 = load_file_name + "." + colname + "." + to_string(segNum);
	if(type[colname] == 2)
		f1 = f1 + ".idx";

	std::clock_t start1 = std::clock();

	if(interactive) { //check if data are in buffers
		if(buffers.find(f1) == buffers.end()) { // add data to buffers
			FILE* f = fopen(f1.c_str(), "rb" );
			if(!f) {
				process_error(3, "Error opening " + string(f1) +" file " );
			};
			fseek(f, 0, SEEK_END);
			long fileSize = ftell(f);
			while(total_buffer_size + fileSize > getTotalSystemMemory() && !buffer_names.empty()) { //free some buffers
				//delete [] buffers[buffer_names.front()];
				hipHostFree(buffers[buffer_names.front()]);
				total_buffer_size = total_buffer_size - buffer_sizes[buffer_names.front()];
				buffer_sizes.erase(buffer_names.front());
				buffers.erase(buffer_names.front());
				buffer_names.pop();
			};
			fseek(f, 0, SEEK_SET);

			char* buff;
			hipHostAlloc((void**) &buff, fileSize,hipHostMallocDefault);
			fread(buff, fileSize, 1, f);
			fclose(f);
			buffers[f1] = buff;
			buffer_sizes[f1] = fileSize;
			buffer_names.push(f1);
			total_buffer_size = total_buffer_size + fileSize;
			buffer_names.push(f1);
			cout << "added buffer " << f1 << " " << fileSize << endl;
		};
		// get data from buffers
		if(type[colname] != 1) {
			unsigned int cnt = ((unsigned int*)buffers[f1])[0];
			if(cnt > h_columns_int[colname].size()/8 + 10)
				h_columns_int[colname].resize(cnt/8 + 10);
		}
		else {
			unsigned int cnt = ((unsigned int*)buffers[f1])[0];
			if(cnt > h_columns_float[colname].size()/8 + 10)
				h_columns_float[colname].resize(cnt/8 + 10);
		}
	}
	else {

		FILE* f = fopen(f1.c_str(), "rb" );
		if(!f) {
			cout << "Error opening " << f1 << " file " << endl;
			exit(0);
		};

		if(type[colname] != 1) {
			if(1 > h_columns_int[colname].size())
				h_columns_int[colname].resize(1);
			fread(h_columns_int[colname].data(), 4, 1, f);
			unsigned int cnt = ((unsigned int*)(h_columns_int[colname].data()))[0];
			if(cnt/8+10 > h_columns_int[colname].size()) {
				h_columns_int[colname].resize(cnt + 10);
			};
			size_t rr = fread((unsigned int*)(h_columns_int[colname].data()) + 1, 1, cnt+52, f);
			if(rr != cnt+52) {
				char buf[1024];
				sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
				process_error(3, string(buf));
			};
		}
		else  {
			if(1 > h_columns_float[colname].size())
				h_columns_float[colname].resize(1);
			fread(h_columns_float[colname].data(), 4, 1, f);
			unsigned int cnt = ((unsigned int*)(h_columns_float[colname].data()))[0];
			if(cnt/8+10 > h_columns_float[colname].size())
				h_columns_float[colname].resize(cnt + 10);
			size_t rr = fread((unsigned int*)(h_columns_float[colname].data()) + 1, 1, cnt+52, f);
			if(rr != cnt+52) {
				char buf[1024];
				sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
				process_error(3, string(buf));
			};
		}
		fclose(f);
	};
	tot_disk =  tot_disk + (std::clock() - start1);
};

void CudaSet::CopyColumnToGpu(string colname,  unsigned int segment, size_t offset)
{

	if(not_compressed) 	{
		// calculate how many records we need to copy
		if(segment < segCount-1) {
			mRecCount = maxRecs;
		}
		else {
			mRecCount = hostRecCount - maxRecs*(segCount-1);
		};

		if(type[colname] != 1) {
			if(!alloced_switch) {
				thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_columns_int[colname].begin() + offset);
			}
			else {
				thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
				thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_col);
			};
		}
		else {
			if(!alloced_switch) {
				thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_columns_float[colname].begin() + offset);
			}
			else {
				thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
				thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_col);
			};
		}
	}
	else {
		readSegmentsFromFile(segment,colname);
		if(!d_v)
			CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
		if(!s_v)
			CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

		string f1;
		if(type[colname] == 2) {
			f1 = load_file_name + "." + colname + "." + to_string(segment) + ".idx";
		}
		else {
			f1 = load_file_name + "." + colname + "." + to_string(segment);
		};

		if(type[colname] != 1) {
			if(!alloced_switch) {
				if(buffers.find(f1) == buffers.end()) {
					mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), h_columns_int[colname].data(), d_v, s_v, colname);
				}
				else {
					mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), buffers[f1], d_v, s_v, colname);
				};
			}
			else {
				if(buffers.find(f1) == buffers.end()) {
					mRecCount = pfor_decompress(alloced_tmp, h_columns_int[colname].data(), d_v, s_v, colname);
				}
				else {
					mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
				};
			};
		}
		else  {
			if(decimal[colname]) {
				if(!alloced_switch) {
					if(buffers.find(f1) == buffers.end()) {
						mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , h_columns_float[colname].data(), d_v, s_v, colname);
					}
					else {
						mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , buffers[f1], d_v, s_v, colname);
					};
					if(!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + offset));
						thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin(), long_to_float());
					};
				}
				else {
					if(buffers.find(f1) == buffers.end()) {
						mRecCount = pfor_decompress(alloced_tmp, h_columns_float[colname].data(), d_v, s_v, colname);
					}
					else {
						mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
					};
					if(!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
						thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
						thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
					};
					//for(int i = 0; i < mRecCount;i++)
					//cout << "DECOMP " << (float_type)(d_col_int[i]) << " " << d_col_float[i] << endl;

				};
			}
			//else // uncompressed float
			// will have to fix it later so uncompressed data will be written by segments too
		}
	};
}


void CudaSet::CopyColumnToGpu(string colname) // copy all segments
{
	if(not_compressed) {
		if(type[colname] != 1)
			thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mRecCount, d_columns_int[colname].begin());
		else
			thrust::copy(h_columns_float[colname].begin(), h_columns_float[colname].begin() + mRecCount, d_columns_float[colname].begin());
	}
	else {
		if(!d_v)
			CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
		if(!s_v)
			CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

		size_t cnt = 0;
		string f1;

		for(unsigned int i = 0; i < segCount; i++) {

			readSegmentsFromFile(i,colname);

			if(type[colname] == 2) {
				f1 = load_file_name + "." + colname + "." + to_string(i) + ".idx";
			}
			else {
				f1 = load_file_name + "." + colname + "." + to_string(i);
			};


			if(type[colname] == 0) {
				if(buffers.find(f1) == buffers.end()) {
					mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), h_columns_int[colname].data(), d_v, s_v, colname);
				}
				else {
					mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), buffers[f1], d_v, s_v, colname);
				};

			}
			else
				if(type[colname] == 1) {
					if(decimal[colname]) {
						if(buffers.find(f1) == buffers.end()) {
							mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , h_columns_float[colname].data(), d_v, s_v, colname);
						}
						else {
							mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , buffers[f1], d_v, s_v, colname);
						};
						if(!phase_copy) {
							thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt));
							thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin() + cnt, long_to_float());
						};
					}
					// else  uncompressed float
					// will have to fix it later so uncompressed data will be written by segments too
				};
			cnt = cnt + mRecCount;

			//totalRecs = totals + mRecCount;
		};

		mRecCount = cnt;
	};
}

void CudaSet::CopyColumnToHost(string colname, size_t offset, size_t RecCount)
{

	if(type[colname] != 1) {
		thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin() + RecCount, h_columns_int[colname].begin() + offset);
	}
	else
		thrust::copy(d_columns_float[colname].begin(), d_columns_float[colname].begin() + RecCount, h_columns_float[colname].begin() + offset);
}


void CudaSet::CopyColumnToHost(string colname)
{
	CopyColumnToHost(colname, 0, mRecCount);
}

void CudaSet::CopyToHost(size_t offset, size_t count)
{
	for(unsigned int i = 0; i < columnNames.size(); i++) {
		CopyColumnToHost(columnNames[i], offset, count);
	};
}

float_type* CudaSet::get_float_type_by_name(string name)
{
	return thrust::raw_pointer_cast(d_columns_float[name].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
	return thrust::raw_pointer_cast(d_columns_int[name].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
	return thrust::raw_pointer_cast(h_columns_float[name].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
	return thrust::raw_pointer_cast(h_columns_int[name].data());
}



void CudaSet::GroupBy(stack<string> columnRef)
{
	thrust::device_vector<bool> grp_dev(mRecCount);

	thrust::fill(grp_dev.begin(), grp_dev.end(), 0);
	if(scratch.size() < mRecCount)
		scratch.resize(mRecCount*sizeof(bool));
	thrust::device_ptr<bool> d_group((bool*)thrust::raw_pointer_cast(scratch.data()));
	d_group[mRecCount-1] = 0;


	for(int i = 0; i < columnRef.size(); columnRef.pop()) {

		unsigned int bits;
		if(cpy_bits.empty())
			bits = 0;
		else
			bits = cpy_bits[columnRef.top()];

		if(bits == 8) {
			thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
			thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned char>());
		}
		else
			if(bits == 16) {
				thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
				thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned short int>());
			}
			else
				if(bits == 32) {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned int>());
				}
				else {
					thrust::transform(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount - 1,
					                  d_columns_int[columnRef.top()].begin()+1, d_group, thrust::not_equal_to<int_type>());
				};
		thrust::transform(d_group, d_group+mRecCount, grp_dev.begin(), grp_dev.begin(), thrust::logical_or<bool>());
	};
	grp_count = thrust::count(grp_dev.begin(), grp_dev.end(), 1) + 1;
	//cout << "grp count " << grp_count << endl;
	grp.resize(grp_count);
	if(grp_count > 1)
		thrust::copy_if(thrust::make_counting_iterator((unsigned int)1), thrust::make_counting_iterator((unsigned int)grp_dev.size()),
		                grp_dev.begin(), grp.begin()+1, thrust::identity<bool>());
	grp[0] = 0;
};




void CudaSet::addDeviceColumn(int_type* col, string colname, size_t recCount)
{
	if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
		columnNames.push_back(colname);
		type[colname] = 0;
		d_columns_int[colname] = thrust::device_vector<int_type>(recCount);
		h_columns_int[colname] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >(recCount);
	}
	else {  // already exists, my need to resize it
		if(d_columns_int[colname].size() < recCount) {
			d_columns_int[colname].resize(recCount);
		};
		if(h_columns_int[colname].size() < recCount) {
			h_columns_int[colname].resize(recCount);
		};
	};
	// copy data to d columns
	thrust::device_ptr<int_type> d_col((int_type*)col);
	thrust::copy(d_col, d_col+recCount, d_columns_int[colname].begin());
	thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin()+recCount, h_columns_int[colname].begin());
};

void CudaSet::addDeviceColumn(float_type* col, string colname, size_t recCount, bool is_decimal)
{
	if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
		columnNames.push_back(colname);
		type[colname] = 1;
		d_columns_float[colname] = thrust::device_vector<float_type>(recCount);
		h_columns_float[colname] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(recCount);
	}
	else {  // already exists, my need to resize it
		if(d_columns_float[colname].size() < recCount)
			d_columns_float[colname].resize(recCount);
		if(h_columns_float[colname].size() < recCount)
			h_columns_float[colname].resize(recCount);
	};

	decimal[colname] = is_decimal;
	thrust::device_ptr<float_type> d_col((float_type*)col);
	thrust::copy(d_col, d_col+recCount, d_columns_float[colname].begin());
};

void CudaSet::gpu_perm(queue<string> sf, thrust::device_vector<unsigned int>& permutation) {

	permutation.resize(mRecCount);
	thrust::sequence(permutation.begin(), permutation.begin() + mRecCount,0,1);
	unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
	void* temp;

	CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*8));
	string sort_type = "ASC";

	while(!sf.empty()) {

		if (type[sf.front()] == 0) {
			update_permutation(d_columns_int[sf.front()], raw_ptr, mRecCount, sort_type, (int_type*)temp, 64);
		}
		else
			if (type[sf.front()] == 1) {
				update_permutation(d_columns_float[sf.front()], raw_ptr, mRecCount, sort_type, (float_type*)temp, 64);
			}
			else {
				thrust::host_vector<unsigned int> permutation_h = permutation;
				char* temp1 = new char[char_size[sf.front()]*mRecCount];
				update_permutation_char_host(h_columns_char[sf.front()], permutation_h.data(), mRecCount, sort_type, temp1, char_size[sf.front()]);
				delete [] temp1;
				permutation = permutation_h;
			};
		sf.pop();
	};
	hipFree(temp);
}


void CudaSet::compress(string file_name, size_t offset, unsigned int check_type, unsigned int check_val, size_t mCount, const bool append)
{
	string str(file_name);
	thrust::device_vector<unsigned int> permutation;
	long long int oldCount;
	bool int_check = 0;

	void* d;
	CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

	total_count = total_count + mCount;
	if (mCount > total_max && op_sort.empty()) {
		total_max = mCount;
	};

	if(!total_segments && append) {
		string s= file_name + "." + columnNames[0] + ".header";
		ifstream binary_file(s.c_str(),ios::binary);
		if(binary_file) {
			binary_file.read((char *)&oldCount, 8);
			binary_file.read((char *)&total_segments, 4);
			binary_file.read((char *)&maxRecs, 4);
			if(total_max < maxRecs)
				total_max = maxRecs;
			binary_file.close();
			total_count = oldCount + mCount;
		};
	};


	if(!op_sort.empty()) { //sort the segment
		gpu_perm(op_sort, permutation);
	};

	// here we need to check for partitions and if partition_count > 0 -> create partitions
	if(mCount < partition_count || partition_count == 0)
		partition_count = 1;
	unsigned int partition_recs = mCount/partition_count;

	if(!op_sort.empty()) {
		if(total_max < partition_recs)
			total_max = partition_recs;
	};


	total_segments++;
	old_segments = total_segments;
	size_t new_offset;
	for(unsigned int i = 0; i < columnNames.size(); i++) {
		std::clock_t start1 = std::clock();
		string colname = columnNames[i];
		str = file_name + "." + colname;
		curr_file = str;
		str += "." + to_string(total_segments-1);
		new_offset = 0;

		if(type[colname] == 0) {
			thrust::device_ptr<int_type> d_col((int_type*)d);
			if(!op_sort.empty()) {
				thrust::gather(permutation.begin(), permutation.end(), d_columns_int[colname].begin(), d_col);

				for(unsigned int p = 0; p < partition_count; p++) {
					str = file_name + "." + colname;
					curr_file = str;
					str += "." + to_string(total_segments-1);
					if (p < partition_count - 1) {
						pfor_compress( (int_type*)d + new_offset, partition_recs*int_size, str, h_columns_int[colname], 0);
					}
					else {
						pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*int_size, str, h_columns_int[colname], 0);
					};
					new_offset = new_offset + partition_recs;
					total_segments++;
				};
			}
			else {
				if(!int_check) {
					thrust::copy(h_columns_int[colname].begin() + offset, h_columns_int[colname].begin() + offset + mCount, d_col);
					pfor_compress( d, mCount*int_size, str, h_columns_int[colname], 0);
				}
				else {
					pfor_compress( thrust::raw_pointer_cast(d_columns_int[colname].data()), mCount*int_size, str, h_columns_int[colname], 0);
				};

			};
		}
		else
			if(type[colname] == 1) {
				if(decimal[colname]) {
					thrust::device_ptr<float_type> d_col((float_type*)d);
					if(!op_sort.empty()) {

						thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
						thrust::device_ptr<long long int> d_col_dec((long long int*)d);
						thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());

						for(unsigned int p = 0; p < partition_count; p++) {
							str = file_name + "." + colname;
							curr_file = str;
							str += "." + to_string(total_segments-1);
							if (p < partition_count - 1)
								pfor_compress( (int_type*)d + new_offset, partition_recs*float_size, str, h_columns_float[colname], 1);
							else
								pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*float_size, str, h_columns_float[colname], 1);
							new_offset = new_offset + partition_recs;
							total_segments++;
						};
					}
					else {
						thrust::copy(h_columns_float[colname].begin() + offset, h_columns_float[colname].begin() + offset + mCount, d_col);
						thrust::device_ptr<long long int> d_col_dec((long long int*)d);
						thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
						pfor_compress( d, mCount*float_size, str, h_columns_float[colname], 1);
					};
				}
				else { // do not compress -- float
					thrust::device_ptr<float_type> d_col((float_type*)d);
					if(!op_sort.empty()) {
						thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
						thrust::copy(d_col, d_col+mRecCount, h_columns_float[colname].begin());
						for(unsigned int p = 0; p < partition_count; p++) {
							str = file_name + "." + colname;
							curr_file = str;
							str += "." + to_string(total_segments-1);
							unsigned int curr_cnt;
							if (p < partition_count - 1)
								curr_cnt = partition_recs;
							else
								curr_cnt = mCount - partition_recs*p;

							fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
							binary_file.write((char *)&curr_cnt, 4);
							binary_file.write((char *)(h_columns_float[colname].data() + new_offset),curr_cnt*float_size);
							new_offset = new_offset + partition_recs;
							unsigned int comp_type = 3;
							binary_file.write((char *)&comp_type, 4);
							binary_file.close();
						};
					}
					else {
						fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
						binary_file.write((char *)&mCount, 4);
						binary_file.write((char *)(h_columns_float[colname].data() + offset),mCount*float_size);
						unsigned int comp_type = 3;
						binary_file.write((char *)&comp_type, 4);
						binary_file.close();
					};
				};
			}
			else { //char
				//populate char_hash
				if(append && total_segments == 1) {

					string s= file_name + "." + colname;
					ifstream binary_file(s.c_str(),ios::binary);
					if(binary_file) {
						char* strings = new char[oldCount*char_size[colname]];
						binary_file.read(strings, oldCount*char_size[colname]);
						binary_file.close();
						unsigned int ind = std::find(columnNames.begin(), columnNames.end(), colname) - columnNames.begin();
						for (unsigned int z = 0 ; z < oldCount; z++) {
							char_hash[ind][MurmurHash64A(&strings[z*char_size[colname]], char_size[colname], hash_seed)/2] = z;
						};
						delete [] strings;
					};
				};

				if(!op_sort.empty()) {
					unsigned int*  h_permutation = new unsigned int[mRecCount];
					thrust::copy(permutation.begin(), permutation.end(), h_permutation);
					char* t = new char[char_size[colname]*mRecCount];
					apply_permutation_char_host(h_columns_char[colname], h_permutation, mRecCount, t, char_size[colname]);

					delete [] h_permutation;
					thrust::copy(t, t+ char_size[colname]*mRecCount, h_columns_char[colname]);
					delete [] t;
					for(unsigned int p = 0; p < partition_count; p++) {
						str = file_name + "." + colname;
						curr_file = str;
						str += "." + to_string(total_segments-1);

						if (p < partition_count - 1)
							compress_char(str, colname, partition_recs, new_offset, total_segments-1);
						else
							compress_char(str, colname, mCount - partition_recs*p, new_offset, total_segments-1);
						new_offset = new_offset + partition_recs;
						total_segments++;
					};
				}
				else {
					compress_char(str, colname, mCount, offset, total_segments-1);
				};
			};

		if((check_type == 1 && fact_file_loaded) || (check_type == 1 && check_val == 0)) {
			if(!op_sort.empty())
				writeHeader(file_name, colname, total_segments-1);
			else {
				writeHeader(file_name, colname, total_segments);
			};
		};
		total_segments = old_segments;
	};

	hipFree(d);
	if(!op_sort.empty()) {
		total_segments = (old_segments-1)+partition_count;
	};
	permutation.resize(0);
	permutation.shrink_to_fit();
}



void CudaSet::writeHeader(string file_name, string colname, unsigned int tot_segs) {
	string str = file_name + "." + colname;
	string ff = str;
	str += ".header";

	fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
	binary_file.write((char *)&total_count, 8);
	binary_file.write((char *)&tot_segs, 4);
	binary_file.write((char *)&total_max, 4);
	binary_file.write((char *)&cnt_counts[ff], 4);
	//cout << "HEADER1 " << total_count << " " << tot_segs << " " << total_max << endl;
	binary_file.close();
};

void CudaSet::reWriteHeader(string file_name, string colname, unsigned int tot_segs, size_t newRecs, size_t maxRecs1) {
	string str = file_name + "." + colname;
	string ff = str;
	str += ".header";
	fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
	binary_file.write((char *)&newRecs, 8);
	binary_file.write((char *)&tot_segs, 4);
	binary_file.write((char *)&maxRecs1, 4);
	//cout << "HEADER2 " << newRecs << endl;
	binary_file.close();
};



void CudaSet::writeSortHeader(string file_name)
{
	string str(file_name);
	unsigned int idx;

	if(!op_sort.empty()) {
		str += ".sort";
		fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
		idx = (unsigned int)op_sort.size();
		binary_file.write((char *)&idx, 4);
		queue<string> os(op_sort);
		while(!os.empty()) {
			if(verbose)
				cout << "sorted on " << idx << endl;
			idx = os.front().size();
			binary_file.write((char *)&idx, 4);
			binary_file.write(os.front().data(), idx);
			os.pop();
		};
		binary_file.close();
	}
	else {
		str += ".sort";
		remove(str.c_str());
	};

	str = file_name;
	if(!op_presort.empty()) {
		str += ".presort";
		fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
		idx = (unsigned int)op_presort.size();
		binary_file.write((char *)&idx, 4);
		queue<string> os(op_presort);
		while(!os.empty()) {
			idx = os.front().size();
			binary_file.write((char *)&idx, 4);
			binary_file.write(os.front().data(), idx);
			os.pop();
		};
		binary_file.close();
	}
	else {
		str += ".presort";
		remove(str.c_str());
	};
}

using namespace mgpu;

void CudaSet::Display(unsigned int limit, bool binary, bool term)
{
#define MAXCOLS 128
#define MAXFIELDSIZE 1400

	//-- This should/will be converted to an array holding pointers of malloced sized structures--
	char    bigbuf[MAXCOLS * MAXFIELDSIZE];
	memset(bigbuf, 0, MAXCOLS * MAXFIELDSIZE);
	char    *fields[MAXCOLS];
	const   char *dcolumns[MAXCOLS];
	size_t  mCount;         // num records in play
	bool    print_all = 0;
	string  ss, str;
	int rows = 0;

	if(limit != 0 && limit < mRecCount)
		mCount = limit;
	else {
		mCount = mRecCount;
		print_all = 1;
	};

	cout << "mRecCount=" << mRecCount << " mcount = " << mCount << " term " << term <<  " limit=" << limit << " print_all=" << print_all << endl;

	unsigned int cc =0;
	unordered_map<string, FILE*> file_map;
	unordered_map<string, unsigned int> len_map;

	for(unsigned int i = 0; i < columnNames.size(); i++)
	{
		fields[cc] = &(bigbuf[cc*MAXFIELDSIZE]);                        // a hack to avoid malloc overheads     - refine later
		dcolumns[cc++] = columnNames[i].c_str();

		if(string_map.find(columnNames[i]) != string_map.end()) {
			auto s = string_map[columnNames[i]];
			auto pos = s.find_first_of(".");
			auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
			FILE *f;
			f = fopen(string_map[columnNames[i]].c_str(), "rb");
			file_map[string_map[columnNames[i]]] = f;
			len_map[string_map[columnNames[i]]] = len;
		};
	};

	// The goal here is to loop fast and avoid any double handling of outgoing data - pointers are good.
	if(not_compressed && prm_d.size() == 0) {
		for(unsigned int i=0; i < mCount; i++) {                            // for each record
			for(unsigned int j=0; j < columnNames.size(); j++) {                // for each col
				if (type[columnNames[j]] != 1) {
					if(string_map.find(columnNames[j]) == string_map.end()) {

						if(decimal_zeroes[columnNames[j]]) {
							str = std::to_string(h_columns_int[columnNames[j]][i]);
							//cout << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i] << endl;
							while(str.length() <= decimal_zeroes[columnNames[j]])
								str = '0' + str;
							str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
							sprintf(fields[j], "%s", str.c_str());
						}
						else {
							if(!ts_cols[columnNames[j]])
								sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
							else {

								time_t ts = (h_columns_int[columnNames[j]][i])/1000;
								auto ti = gmtime(&ts);
								char buffer[30];
								auto rem = (h_columns_int[columnNames[j]][i])%1000;
								strftime(buffer,30,"%Y-%m-%d %H.%M.%S", ti);
								//fprintf(file_pr, "%s", buffer);
								//fprintf(file_pr, ".%d", rem);
								sprintf(fields[j], "%s.%d", buffer,rem);


								/*time_t tt = h_columns_int[columnNames[j]][i];
								auto ti = localtime(&tt);
								char buffer[10];
								strftime(buffer,80,"%Y-%m-%d", ti);
								sprintf(fields[j], "%s", buffer);
								*/
							};
						};
					}
					else {
						fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
						fread(fields[j], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
						fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
					};
				}
				else
					sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
			};
			row_cb(mColumnCount, (char **)fields, (char **)dcolumns);
			rows++;
		};
	}
	else {
		queue<string> op_vx;
		for(unsigned int i = 0; i < columnNames.size(); i++)
			op_vx.push(columnNames[i]);


		if(prm_d.size() || source) {
			allocColumns(this, op_vx);
		};
		unsigned int curr_seg = 0;
		size_t cnt = 0;
		size_t curr_count, sum_printed = 0;
		resize(maxRecs);
		while(sum_printed < mCount || print_all) {

			if(prm_d.size() || source)  {                            // if host arrays are empty
				copyColumns(this, op_vx, curr_seg, cnt);
				size_t olRecs = mRecCount;
				mRecCount = olRecs;
				CopyToHost(0,mRecCount);
				if(sum_printed + mRecCount <= mCount || print_all)
					curr_count = mRecCount;
				else
					curr_count = mCount - sum_printed;
			}
			else
				curr_count = mCount;

			sum_printed = sum_printed + mRecCount;
			for(unsigned int i=0; i < curr_count; i++) {
				for(unsigned int j=0; j < columnNames.size(); j++) {
					if (type[columnNames[j]] != 1) {
						if(string_map.find(columnNames[j]) == string_map.end())
							sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
						else {
							fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
							fread(fields[j], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
							fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
						};
					}
					else
						sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
				};
				row_cb(mColumnCount, (char **)fields, (char**)dcolumns);
				rows++;
			};
			curr_seg++;
			if(curr_seg == segCount)
				print_all = 0;
		};
	};      // end else
	for(auto it = file_map.begin(); it != file_map.end(); it++)
		fclose(it->second);
}

void CudaSet::Store(const string file_name, const char* sep, const unsigned int limit, const bool binary, const bool append, const bool term)
{
	if (mRecCount == 0 && binary == 1 && !term) { // write tails
		for(unsigned int j=0; j < columnNames.size(); j++) {
			writeHeader(file_name, columnNames[j], total_segments);
		};
		return;
	};

	size_t mCount;
	bool print_all = 0;
	string str;


	if(limit != 0 && limit < mRecCount)
		mCount = limit;
	else {
		mCount = mRecCount;
		print_all = 1;
	};


	if(binary == 0) {

		unordered_map<string, FILE*> file_map;
		unordered_map<string, unsigned int> len_map;
		string bf;
		unsigned int max_len = 0;
		for(unsigned int j=0; j < columnNames.size(); j++) {
			if(string_map.find(columnNames[j]) != string_map.end()) {
				auto s = string_map[columnNames[j]];
				auto pos = s.find_first_of(".");
				auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
				if(len > max_len)
					max_len = len;
				FILE *f;
				f = fopen(string_map[columnNames[j]].c_str(), "rb");
				file_map[string_map[columnNames[j]]] = f;
				len_map[string_map[columnNames[j]]] = len;
			};
		};
		bf.reserve(max_len);

		FILE *file_pr;
		if(!term) {
			file_pr = fopen(file_name.c_str(), "w");
			if (!file_pr)
				cout << "Could not open file " << file_name << endl;
		}
		else
			file_pr = stdout;


		if(not_compressed && prm_d.size() == 0) {
			for(unsigned int i=0; i < mCount; i++) {
				for(unsigned int j=0; j < columnNames.size(); j++) {
					if (type[columnNames[j]] != 1 ) {
						if(string_map.find(columnNames[j]) == string_map.end()) {
							if(decimal_zeroes[columnNames[j]]) {
								str = std::to_string(h_columns_int[columnNames[j]][i]);
								//cout << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i] << endl;
								while(str.length() <= decimal_zeroes[columnNames[j]])
									str = '0' + str;
								str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
								fprintf(file_pr, "%s", str.c_str());
							}
							else {
								if(!ts_cols[columnNames[j]]) {
									fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
								}
								else {
									time_t ts = (h_columns_int[columnNames[j]][i])/1000;
									auto ti = gmtime(&ts);
									char buffer[30];
									auto rem = (h_columns_int[columnNames[j]][i])%1000;
									strftime(buffer,30,"%Y-%m-%d %H.%M.%S", ti);
									fprintf(file_pr, "%s", buffer);
									fprintf(file_pr, ".%d", rem);
								};
							};

						}
						else {
							//fprintf(file_pr, "%.*s", string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].size(), string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].c_str());
							fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
							fread(&bf[0], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
							fprintf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
						};
						fputs(sep, file_pr);
					}
					else {
						fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
						fputs(sep, file_pr);
					}
				};
				if (i != mCount -1 )
					fputs("\n",file_pr);
			};
			if(!term)
				fclose(file_pr);
		}
		else {

			queue<string> op_vx;
			string ss;
			for(unsigned int j=0; j < columnNames.size(); j++)
				op_vx.push(columnNames[j]);

			if(prm_d.size() || source) {
				allocColumns(this, op_vx);
			};

			unsigned int curr_seg = 0;
			size_t cnt = 0;
			size_t curr_count, sum_printed = 0;
			mRecCount = 0;
			resize(maxRecs);

			while(sum_printed < mCount || print_all) {

				if(prm_d.size() || source)  {
					copyColumns(this, op_vx, curr_seg, cnt);
					if(curr_seg == 0) {
						if(limit != 0 && limit < mRecCount) {
							mCount = limit;
							print_all = 0;
						}
						else {
							mCount = mRecCount;
							print_all = 1;
						};

					};

					// if host arrays are empty
					size_t olRecs = mRecCount;
					mRecCount = olRecs;
					CopyToHost(0,mRecCount);
					//cout << "start " << sum_printed << " " <<  mRecCount << " " <<  mCount << endl;
					if(sum_printed + mRecCount <= mCount || print_all) {
						curr_count = mRecCount;
					}
					else {
						curr_count = mCount - sum_printed;
					};
				}
				else {
					curr_count = mCount;
				};

				sum_printed = sum_printed + mRecCount;
				//cout << "sum printed " << sum_printed << " " << curr_count << " " << curr_seg << endl;

				for(unsigned int i=0; i < curr_count; i++) {
					for(unsigned int j=0; j < columnNames.size(); j++) {
						if (type[columnNames[j]] != 1) {
							if(string_map.find(columnNames[j]) == string_map.end()) {

								if(decimal_zeroes[columnNames[j]]) {
									str = std::to_string(h_columns_int[columnNames[j]][i]);
									//cout << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i] << endl;
									while(str.length() <= decimal_zeroes[columnNames[j]])
										str = '0' + str;
									str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
									fprintf(file_pr, "%s", str.c_str());
								}
								else {
									if(!ts_cols[columnNames[j]]) {
										fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
									}
									else {
										time_t ts = (h_columns_int[columnNames[j]][i])/1000;
										auto ti = gmtime(&ts);
										char buffer[30];
										auto rem = (h_columns_int[columnNames[j]][i])%1000;
										strftime(buffer,30,"%Y-%m-%d %H.%M.%S", ti);
										fprintf(file_pr, "%s", buffer);
										fprintf(file_pr, ".%d", rem);
									};
								};

							}
							else {
								fseek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
								fread(&bf[0], 1, len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
								fprintf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
							};
							fputs(sep, file_pr);
						}
						else  {
							fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
							fputs(sep, file_pr);
						};
					};
					if (i != mCount -1 && (curr_seg != segCount || i < curr_count))
						fputs("\n",file_pr);
				};
				curr_seg++;
				if(curr_seg == segCount)
					print_all = 0;
			};
			if(!term) {
				fclose(file_pr);
			};
		};
		for(auto it = file_map.begin(); it != file_map.end(); it++)
			fclose(it->second);
	}
	else {
		//lets update the data dictionary
		for(unsigned int j=0; j < columnNames.size(); j++) {

			data_dict[file_name][columnNames[j]].col_type = type[columnNames[j]];
			if(type[columnNames[j]] != 2) {
				if(decimal[columnNames[j]])
					data_dict[file_name][columnNames[j]].col_length = decimal_zeroes[columnNames[j]];
				else
					if (ts_cols[columnNames[j]])
						data_dict[file_name][columnNames[j]].col_length = UINT_MAX;
					else
						data_dict[file_name][columnNames[j]].col_length = 0;
			}
			else
				data_dict[file_name][columnNames[j]].col_length = char_size[columnNames[j]];
		};
		save_dict = 1;


		if(text_source) {  //writing a binary file using a text file as a source
			compress(file_name, 0, 1, 0, mCount, append);
			for(unsigned int i = 0; i< columnNames.size(); i++)
				if(type[columnNames[i]] == 2)
					deAllocColumnOnDevice(columnNames[i]);
		}
		else { //writing a binary file using a binary file as a source
			fact_file_loaded = 1;
			size_t offset = 0;

			if(!not_compressed) { // records are compressed, for example after filter op.
				//decompress to host
				queue<string> op_vx;
				for(unsigned int i = 0; i< columnNames.size(); i++) {
					op_vx.push(columnNames[i]);
				};

				allocColumns(this, op_vx);
				size_t oldCnt = mRecCount;
				mRecCount = 0;
				resize(oldCnt);
				mRecCount = oldCnt;
				for(unsigned int i = 0; i < segCount; i++) {
					size_t cnt = 0;
					copyColumns(this, op_vx, i, cnt);
					CopyToHost(0, mRecCount);
					offset = offset + mRecCount;
					compress(file_name, 0, 0, i - (segCount-1), mRecCount, append);
				};
			}
			else {
				// now we have decompressed records on the host
				//call setSegments and compress columns in every segment

				segCount = (mRecCount/process_count + 1);
				offset = 0;

				for(unsigned int z = 0; z < segCount; z++) {

					if(z < segCount-1) {
						if(mRecCount < process_count) {
							mCount = mRecCount;
						}
						else {
							mCount = process_count;
						}
					}
					else {
						mCount = mRecCount - (segCount-1)*process_count;
					};
					compress(file_name, offset, 0, z - (segCount-1), mCount, append);
					offset = offset + mCount;
				};
			};
		};
	};
}


void CudaSet::compress_char(const string file_name, const string colname, const size_t mCount, const size_t offset, const unsigned int segment)
{
	unsigned int len = char_size[colname];

	string h_name, i_name, file_no_seg = file_name.substr(0, file_name.find_last_of("."));
	i_name = file_no_seg + "." + to_string(segment) + ".idx";
	h_name = file_no_seg + "." + to_string(segment) + ".hash";
	fstream b_file_str, loc_hashes;

	fstream binary_file_h(h_name.c_str(),ios::out|ios::binary|ios::trunc);
	binary_file_h.write((char *)&mCount, 4);

	if(segment == 0) {
		b_file_str.open(file_no_seg.c_str(),ios::out|ios::binary|ios::trunc);
	}
	else {
		b_file_str.open(file_no_seg.c_str(),ios::out|ios::binary|ios::app);
	};

	if(h_columns_int.find(colname) == h_columns_int.end()) {
		h_columns_int[colname] = thrust::host_vector<int_type >(mCount);
	}
	else {
		if(h_columns_int[colname].size() < mCount)
			h_columns_int[colname].resize(mCount);
	};
	if(d_columns_int.find(colname) == d_columns_int.end()) {
		d_columns_int[colname] = thrust::device_vector<int_type >(mCount);
	}
	else {
		if(d_columns_int[colname].size() < mCount)
			d_columns_int[colname].resize(mCount);
	};


	size_t  cnt;
	long long int* hash_array = new long long int[mCount];
	map<unsigned long long int, size_t>::iterator iter;
	unsigned int ind = std::find(columnNames.begin(), columnNames.end(), colname) - columnNames.begin();

	for (unsigned int i = 0 ; i < mCount; i++) {
		hash_array[i] = MurmurHash64A(h_columns_char[colname] + (i+offset)*len, len, hash_seed)/2;
		iter = char_hash[ind].find(hash_array[i]);
		if(iter == char_hash[ind].end()) {
			cnt = char_hash[ind].size();
			char_hash[ind][hash_array[i]] = cnt;
			b_file_str.write((char *)h_columns_char[colname] + (i+offset)*len, len);
			h_columns_int[colname][i] = cnt;
		}
		else {
			h_columns_int[colname][i] = iter->second;
		};
	};

	binary_file_h.write((char *)hash_array, 8*mCount);
	delete [] hash_array;

	thrust::device_vector<int_type> d_col(mCount);
	thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mCount, d_col.begin());
	pfor_compress(thrust::raw_pointer_cast(d_col.data()), mCount*int_size, i_name, h_columns_int[colname], 0);
	binary_file_h.close();
	b_file_str.close();
};



bool first_time = 1;
size_t rec_sz = 0;
size_t process_piece;

bool CudaSet::LoadBigFile(FILE* file_p, thrust::device_vector<char>& d_readbuff, thrust::device_vector<char*>& dest,
                          thrust::device_vector<unsigned int>& ind, thrust::device_vector<unsigned int>& dest_len)
{
	const char* sep = separator.c_str();
	unsigned int maxx = cols.rbegin()->first;
	map<unsigned int, string>::iterator it;
	bool done = 0;
	std::clock_t start1 = std::clock();

	vector<int> types;
	vector<int> cl;
	types.push_back(0);
	for(int i = 0; i < maxx; i++) {
		auto iter = cols.find(i+1);
		if(iter != cols.end()) {
			types.push_back(type[iter->second]);
			cl.push_back(iter->first-1);
		}
		else
			types.push_back(0);
	};


	if(first_time)	{
		if(process_count*4 > getFreeMem()) {
			process_piece = getFreeMem()/4;
		}
		else
			process_piece = process_count;
		readbuff = new char[process_piece+1];
		d_readbuff.resize(process_piece+1);
		cout << "set a piece to " << process_piece << " " << getFreeMem() << endl;
	};


	thrust::device_vector<unsigned int> ind_cnt(1);
	thrust::device_vector<char> sepp(1);
	sepp[0] = *sep;

	long long int total_processed = 0;
	size_t recs_processed = 0;
	bool finished = 0;
	thrust::device_vector<long long int> dev_pos;
	long long int offset;
	unsigned int cnt = 1;
	const unsigned int max_len = 23;

	while(!done) {

		auto rb = fread(readbuff, 1, process_piece, file_p);

		if(rb < process_piece) {
			done = 1;
			finished = 1;
			fclose(file_p);
		};
		if(total_processed >= process_count)
			done = 1;
			
		thrust::fill(d_readbuff.begin(), d_readbuff.end(),0);
		thrust::copy(readbuff, readbuff+rb, d_readbuff.begin());

		
		auto curr_cnt = thrust::count(d_readbuff.begin(), d_readbuff.begin() + rb, '\n') - 1;

		if(recs_processed == 0 && first_time) {
			rec_sz = curr_cnt;
			if(finished)
				rec_sz++;
			total_max = curr_cnt;
		};
		

		if(first_time)	{
			for(unsigned int i=0; i < columnNames.size(); i++) {
				auto colname = columnNames[i];
				if (type[colname] == 0) {
					d_columns_int[colname].resize(d_columns_int[colname].size() + rec_sz);
					h_columns_int[colname].resize(h_columns_int[colname].size() + rec_sz);
				}
				else
					if (type[colname] == 1) {
						d_columns_float[colname].resize(d_columns_float[colname].size() + rec_sz);
						h_columns_float[colname].resize(h_columns_float[colname].size() + rec_sz);
					}
					else {
						char* c = new char[cnt*rec_sz*char_size[columnNames[i]]];
						if(recs_processed > 0) {
							memcpy(c, h_columns_char[columnNames[i]], recs_processed*char_size[columnNames[i]]);
							delete [] h_columns_char[columnNames[i]];
						};
						h_columns_char[columnNames[i]] = c;
						if(recs_processed == 0) {
							void* temp;
							CUDA_SAFE_CALL(hipMalloc((void **) &temp, char_size[columnNames[i]]*rec_sz));
							hipMemset(temp,0,char_size[columnNames[i]]*rec_sz);
							d_columns_char[columnNames[i]] = (char*)temp;
						};
					};

				if(recs_processed == 0) {
					ind[i] = cl[i];
					void* temp;
					if(type[columnNames[i]] != 2) {
						if(!ts_cols[columnNames[i]]) {
							CUDA_SAFE_CALL(hipMalloc((void **) &temp, max_len*rec_sz));
							dest_len[i] = max_len;
						}
						else {
							CUDA_SAFE_CALL(hipMalloc((void **) &temp, 23*rec_sz));
							dest_len[i] = 23;
						}
					}
					else {
						CUDA_SAFE_CALL(hipMalloc((void **) &temp, char_size[columnNames[i]]*rec_sz));
						dest_len[i] = char_size[columnNames[i]];
					};
					dest[i] = (char*)temp;
				};
			};
		};
		

		for(unsigned int i=0; i < columnNames.size(); i++) {
			if(type[columnNames[i]] != 2) {
				hipMemset(dest[i],0,max_len*rec_sz);
			}
			else {
				hipMemset(dest[i],0,char_size[columnNames[i]]*rec_sz);
			};
		};


		if(dev_pos.size() < curr_cnt+1)
			dev_pos.resize(curr_cnt+1);	//avoiding the unnecessary allocs
		dev_pos[0] = -1;
		thrust::copy_if(thrust::make_counting_iterator((unsigned long long int)0), thrust::make_counting_iterator((unsigned long long int)rb-1),
		                d_readbuff.begin(), dev_pos.begin()+1, _1 == '\n');


		if(!finished) {
			if(curr_cnt < rec_sz) {
				offset = (dev_pos[curr_cnt] - rb)+1;
				//cout << "PATH 1 " << dev_pos[curr_cnt] << " " << offset << endl;
				fseek(file_p, offset, SEEK_CUR);
				total_processed = total_processed + rb + offset;
				mRecCount = curr_cnt;
			}
			else {
				offset = (dev_pos[rec_sz] - rb)+1;
				//cout << "PATH 2 " << dev_pos[rec_sz] << " " << offset << endl;
				fseek(file_p, offset, SEEK_CUR);
				total_processed = total_processed + rb + offset;
				mRecCount = rec_sz;
			};
		}
		else {
			mRecCount = curr_cnt + 1;
		};
		

		thrust::counting_iterator<unsigned int> begin(0);
		ind_cnt[0] = mColumnCount;
		parse_functor ff((const char*)thrust::raw_pointer_cast(d_readbuff.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(ind.data()),
		                 thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(sepp.data()), thrust::raw_pointer_cast(dev_pos.data()), thrust::raw_pointer_cast(dest_len.data()));
		thrust::for_each(begin, begin + mRecCount, ff);


		ind_cnt[0] = max_len;
		for(int i =0; i < mColumnCount; i++) {
			if(type[columnNames[i]] == 0) {  //int
				thrust::device_ptr<char> p1((char*)dest[i]);
				if(p1[4] == '-') { //date
					if(!ts_cols[columnNames[i]]) {
						gpu_date date_ff((const char*)dest[i],(long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed);
						thrust::for_each(begin, begin + mRecCount, date_ff);
					}
					else {
						gpu_tdate date_ff((const char*)dest[i],(long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed);
						thrust::for_each(begin, begin + mRecCount, date_ff);
					}
				}
				else { //int
					if(decimal[columnNames[i]]) {
						thrust::device_vector<unsigned int> scale(1);
						scale[0] =  decimal_zeroes[columnNames[i]];
						gpu_atold atold((const char*)dest[i],(long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed,
						                thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(scale.data()));
						thrust::for_each(begin, begin + mRecCount, atold);
					}
					else {
						gpu_atoll atoll_ff((const char*)dest[i],(long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed,
						                   thrust::raw_pointer_cast(ind_cnt.data()));
						thrust::for_each(begin, begin + mRecCount, atoll_ff);
					};
				};
				thrust::copy(d_columns_int[columnNames[i]].begin() + recs_processed, d_columns_int[columnNames[i]].begin()+recs_processed+mRecCount, h_columns_int[columnNames[i]].begin() + recs_processed);
			}
			else
				if(type[columnNames[i]] == 1) {
					gpu_atof atof_ff((const char*)dest[i],(double*)thrust::raw_pointer_cast(d_columns_float[columnNames[i]].data()) + recs_processed,
					                 thrust::raw_pointer_cast(ind_cnt.data()));
					thrust::for_each(begin, begin + mRecCount, atof_ff);
					thrust::copy(d_columns_float[columnNames[i]].begin() + recs_processed, d_columns_float[columnNames[i]].begin()+recs_processed+mRecCount, h_columns_float[columnNames[i]].begin() + recs_processed);
				}
				else {//char is already done
					thrust::device_ptr<char> p1((char*)dest[i]);
					hipMemcpy( h_columns_char[columnNames[i]] + char_size[columnNames[i]]*recs_processed, (void *)dest[i] , char_size[columnNames[i]]*mRecCount, hipMemcpyDeviceToHost);
				};
		};
		recs_processed = recs_processed + mRecCount;
		cnt++;
	};


	if(finished) {
		for(int i =0; i < mColumnCount; i++) {
			if(dest[i]) {
				hipFree(dest[i]);
				dest[i] = nullptr;
			};
		};
		delete [] readbuff;
	};
	cout << "processed recs " << recs_processed << " " << getFreeMem() << endl;
	first_time = 0;
	mRecCount = recs_processed;
	return finished;
};


void CudaSet::free()  {
	for(unsigned int i = 0; i < columnNames.size(); i++ ) {
		if(type[columnNames[i]] == 0 && h_columns_int[columnNames[i]].size() ) {
			h_columns_int[columnNames[i]].resize(0);
			h_columns_int[columnNames[i]].shrink_to_fit();
		}
		else {
			h_columns_float[columnNames[i]].resize(0);
			h_columns_float[columnNames[i]].shrink_to_fit();
		};
	};
	if(prm_d.size()) {
		prm_d.resize(0);
		prm_d.shrink_to_fit();
	};
	deAllocOnDevice();
};

void alloc_pool(unsigned int maxRecs) {
	void* temp;
	CUDA_SAFE_CALL(hipMalloc((void **) &temp, 8*maxRecs));
	alloced_mem.push_back(temp);
};



bool* CudaSet::logical_and(bool* column1, bool* column2)
{
	thrust::device_ptr<bool> dev_ptr1(column1);
	thrust::device_ptr<bool> dev_ptr2(column2);

	thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());
	thrust::device_free(dev_ptr2);
	return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{
	thrust::device_ptr<bool> dev_ptr1(column1);
	thrust::device_ptr<bool> dev_ptr2(column2);

	thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
	thrust::device_free(dev_ptr2);
	return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
	bool res;

	if (op_type == 2) // >
		if(d>s)
			res = 1;
		else
			res = 0;
	else
		if (op_type == 1)  // <
			if(d<s)
				res = 1;
			else
				res = 0;
		else
			if (op_type == 6) // >=
				if(d>=s)
					res = 1;
				else
					res = 0;
			else
				if (op_type == 5)  // <=
					if(d<=s)
						res = 1;
					else
						res = 0;
				else
					if (op_type == 4)// =
						if(d==s)
							res = 1;
						else
							res = 0;
					else // !=
						if(d!=s)
							res = 1;
						else
							res = 0;

	thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
	thrust::sequence(p, p+mRecCount,res,(bool)0);

	return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
	bool res;

	if (op_type == 2) // >
		if ((d-s) > EPSILON)
			res = 1;
		else
			res = 0;
	else
		if (op_type == 1)  // <
			if ((s-d) > EPSILON)
				res = 1;
			else
				res = 0;
		else
			if (op_type == 6) // >=
				if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON)))
					res = 1;
				else
					res = 0;
			else
				if (op_type == 5)  // <=
					if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON)))
						res = 1;
					else
						res = 0;
				else
					if (op_type == 4)// =
						if (((d-s) < EPSILON) && ((d-s) > -EPSILON))
							res = 1;
						else
							res = 0;
					else // !=
						if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON)))
							res = 1;
						else
							res = 0;

	thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
	thrust::sequence(p, p+mRecCount,res,(bool)0);

	return thrust::raw_pointer_cast(p);
}



bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
	thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
	thrust::device_ptr<float_type> dev_ptr(column1);

	if (op_type == 2) // >
		thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
	else
		if (op_type == 1)  // <
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
		else
			if (op_type == 6) // >=
				thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
			else
				if (op_type == 5)  // <=
					thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
				else
					if (op_type == 4)// =
						thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
					else  // !=
						thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

	return thrust::raw_pointer_cast(res);
}

bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type, unsigned int p1, unsigned int p2)
{
	thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
	thrust::device_ptr<int_type> dev_ptr(column1);

	if(p2)
		d = d*(int_type)pow(10, p2);

	if (op_type == 2) // >
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
	else
		if (op_type == 1)  // <
			if(!p1)
				thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
		else
			if (op_type == 6) // >=
				if(!p1)
					thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
				else
					thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
			else
				if (op_type == 5)  // <=
					if(!p1)
						thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
					else
						thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
				else
					if (op_type == 4)// =
						if(!p1)
							thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
						else
							thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
					else // !=
						if(!p1)
							thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());
						else
							thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

	return thrust::raw_pointer_cast(temp);

}



bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type, unsigned int p1, unsigned int p2)
{
	thrust::device_ptr<int_type> dev_ptr1(column1);
	thrust::device_ptr<int_type> dev_ptr2(column2);
	thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

	if (op_type == 2) // >
		if(!p1 && !p2) {
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
		}
		else
			if(p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater<int_type>());
			else
				if(p1)
					thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::greater<int_type>());
				else
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater<int_type>());
	else
		if (op_type == 1)  // <
			if(!p1 && !p2)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
			else
				if(p1 && p2)
					thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less<int_type>());
				else
					if(p1)
						thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::less<int_type>());
					else
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less<int_type>());
		else
			if (op_type == 6) // >=
				if(!p1 && !p2)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
				else
					if(p1 && p2)
						thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater_equal<int_type>());
					else
						if(p1)
							thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::greater_equal<int_type>());
						else
							thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater_equal<int_type>());
			else
				if (op_type == 5)  // <=
					if(!p1 && !p2)
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
					else
						if(p1 && p2)
							thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less_equal<int_type>());
						else
							if(p1)
								thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::less_equal<int_type>());
							else
								thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less_equal<int_type>());
				else
					if (op_type == 4)// =
						if(!p1 && !p2)
							thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
						else
							if(p1 && p2)
								thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::equal_to<int_type>());
							else
								if(p1)
									thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::equal_to<int_type>());
								else
									thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::equal_to<int_type>());
					else // !=
						if(!p1 && !p2)
							thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());
						else
							if(p1 && p2)
								thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::not_equal_to<int_type>());
							else
								if(p1)
									thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::not_equal_to<int_type>());
								else
									thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::not_equal_to<int_type>());


	return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
	thrust::device_ptr<float_type> dev_ptr1(column1);
	thrust::device_ptr<float_type> dev_ptr2(column2);
	thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

	if (op_type == 2) // >
		thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
	else
		if (op_type == 1)  // <
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
		else
			if (op_type == 6) // >=
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
			else
				if (op_type == 5)  // <=
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
				else
					if (op_type == 4)// =
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
					else // !=
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

	return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
	thrust::device_ptr<float_type> dev_ptr1(column1);
	thrust::device_ptr<int_type> dev_ptr(column2);
	thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
	thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

	thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

	if (op_type == 2) // >
		thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
	else
		if (op_type == 1)  // <
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
		else
			if (op_type == 6) // >=
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
			else
				if (op_type == 5)  // <=
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
				else
					if (op_type == 4)// =
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
					else // !=
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

	thrust::device_free(dev_ptr2);
	return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, bool reverse)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
	thrust::device_ptr<int_type> dev_ptr(column1);

	thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation
	thrust::device_ptr<float_type> dev_ptr1(column2);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
				else
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
	}
	else {
		if (op_type.compare("MUL") == 0)
			thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
				else
					thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

	};
	alloced_mem.pop_back();
	return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, bool reverse, unsigned int p1, unsigned int p2)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	//cout << "OP " << d << " " << op_type << " " << p1 << " " << p2 << endl;
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());
	thrust::device_ptr<int_type> dev_ptr1(column1);
	unsigned int d1 = d;
	if(p2)
		d = d*(unsigned int)pow(10, p2);

	if(reverse == 0) {

		if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount,  thrust::make_constant_iterator(d1), temp, thrust::multiplies<int_type>());
		}
		else
			if (op_type.compare("ADD") == 0) {
				if(!p1)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::plus<int_type>());
				else
					thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::plus<int_type>());
			}
			else
				if (op_type.compare("MINUS") == 0) {
					if(!p1)
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::minus<int_type>());
					else
						thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::minus<int_type>());
				}
				else {
					if(!p1)
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::divides<int_type>());
					else
						thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::divides<int_type>());
				}
	}
	else {

		if (op_type.compare("MUL") == 0) {
			if(!p1)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
			else
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::multiplies<int_type>());
		}
		else
			if (op_type.compare("ADD") == 0) {
				if(!p1)
					thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
				else
					thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
			}
			else
				if (op_type.compare("MINUS") == 0) {
					if(!p1)
						thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
					else
						thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
				}
				else {
					if(!p1)
						thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
					else
						thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
				};
	};
	alloced_mem.pop_back();
	return thrust::raw_pointer_cast(temp);
}


int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, bool reverse, unsigned int p1, unsigned int p2)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());
	thrust::device_ptr<int_type> dev_ptr1(column1);
	thrust::device_ptr<int_type> dev_ptr2(column2);

	//cout << "OP " <<  op_type << " " << p1 << " " << p2 << " " << reverse << endl;

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
		}
		else
			if (op_type.compare("ADD") == 0) {
				if(!p1 && !p2)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
				else
					if(p1 && p2) {
						thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::plus<int_type>());
					}
					else
						if (p1)
							thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::plus<int_type>());
						else
							thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::plus<int_type>());

			}
			else
				if (op_type.compare("MINUS") == 0) {
					if(!p1 && !p2)
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
					else
						if(p1 && p2)
							thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::minus<int_type>());
						else
							if (p1)
								thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::minus<int_type>());
							else
								thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::minus<int_type>());

				}
				else {
					if(!p1 && !p2)
						thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
					else
						if(p1 && p2)
							thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::divides<int_type>());
						else
							if (p1)
								thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::divides<int_type>());
							else
								thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::divides<int_type>());
				}
	}
	else  {
		if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
		}
		else
			if (op_type.compare("ADD") == 0) {
				if(!p1 && !p2)
					thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
				else
					if(p1 && p2)
						thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
					else
						if (p1)
							thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
						else
							thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::plus<int_type>());

			}
			else
				if (op_type.compare("MINUS") == 0) {
					if(!p1 && !p2)
						thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
					else
						if(p1 && p2)
							thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
						else
							if (p1)
								thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
							else
								thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::minus<int_type>());
				}
				else {
					if(!p1 && !p2)
						thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
					else
						if(p1 && p2)
							thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
						else
							if (p1)
								thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
							else
								thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::divides<int_type>());
				}
	}
	alloced_mem.pop_back();
	return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, bool reverse)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
	thrust::device_ptr<float_type> dev_ptr1(column1);
	thrust::device_ptr<float_type> dev_ptr2(column2);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
				else
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
	}
	else {
		if (op_type.compare("MUL") == 0)
			thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
				else
					thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
	};
	alloced_mem.pop_back();
	return thrust::raw_pointer_cast(temp);
}




float_type* CudaSet::op(int_type* column1, float_type d, string op_type, bool reverse)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
	thrust::fill(temp, temp+mRecCount, d);

	thrust::device_ptr<int_type> dev_ptr(column1);
	thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
	thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
				else
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
	}
	else  {
		if (op_type.compare("MUL") == 0)
			thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
				else
					thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

	};
	thrust::device_free(dev_ptr1);
	alloced_mem.pop_back();
	return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(float_type* column1, float_type d, string op_type,bool reverse)
{
	if(alloced_mem.empty()) {
		alloc_pool(maxRecs);
	};
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
	thrust::device_ptr<float_type> dev_ptr1(column1);

	if(reverse == 0) {
		if (op_type.compare("MUL") == 0)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
				else
					thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
	}
	else	{
		if (op_type.compare("MUL") == 0)
			thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
		else
			if (op_type.compare("ADD") == 0)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
			else
				if (op_type.compare("MINUS") == 0)
					thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
				else
					thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
	};
	alloced_mem.pop_back();
	return (float_type*)thrust::raw_pointer_cast(temp);
}

char CudaSet::loadIndex(const string index_name, const unsigned int segment)
{
	FILE* f;
	unsigned int bits_encoded, fit_count, vals_count, sz, real_count;
	void* d_str;
	string f1 = index_name + "." + to_string(segment);
	char res;

	//interactive = 0;
	if(interactive) {
		if(index_buffers.find(f1) == index_buffers.end()) {
			f = fopen (f1.c_str(), "rb" );
			fseek(f, 0, SEEK_END);
			long fileSize = ftell(f);
			char* buff;
			hipHostAlloc(&buff, fileSize, hipHostMallocDefault);

			fseek(f, 0, SEEK_SET);
			fread(buff, fileSize, 1, f);
			fclose(f);
			index_buffers[f1] = buff;
		};
		sz = ((unsigned int*)index_buffers[f1])[0];

		idx_dictionary_int[index_name].clear();
		for(unsigned int i = 0; i < sz; i++) {
			idx_dictionary_int[index_name][((int_type*)(index_buffers[f1]+4+8*i))[0]] = i;
		};
		vals_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[2];
		real_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[3];
		mRecCount = real_count;

		if(idx_vals.count(index_name) == 0) {
			hipMalloc((void **) &d_str, (vals_count+2)*int_size);
			hipMemcpy( d_str, (void *) &((index_buffers[f1]+4 +8*sz)[0]), (vals_count+2)*int_size, hipMemcpyHostToDevice);
			idx_vals[index_name] = (unsigned long long int*)d_str;
		};

	}
	else {
		f = fopen (f1.c_str(), "rb" );
		fread(&sz, 4, 1, f);
		int_type* d_array = new int_type[sz];
		idx_dictionary_int[index_name].clear();
		fread((void*)d_array, sz*int_size, 1, f);
		for(unsigned int i = 0; i < sz; i++) {
			idx_dictionary_int[index_name][d_array[i]] = i;
		};
		delete [] d_array;

		fread(&fit_count, 4, 1, f);
		fread(&bits_encoded, 4, 1, f);
		fread(&vals_count, 4, 1, f);
		fread(&real_count, 4, 1, f);

		mRecCount = real_count;

		unsigned long long int* int_array = new unsigned long long int[vals_count+2];
		fseek ( f , -16 , SEEK_CUR );
		fread((void*)int_array, 1, vals_count*8 + 16, f);
		fread(&res, 1, 1, f);
		fclose(f);
		void* d_str;
		hipMalloc((void **) &d_str, (vals_count+2)*int_size);
		hipMemcpy( d_str, (void *) int_array, (vals_count+2)*int_size, hipMemcpyHostToDevice);
		if(idx_vals.count(index_name))
			hipFree(idx_vals[index_name]);
		idx_vals[index_name] = (unsigned long long int*)d_str;
	}
	return res;
}



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name) // compressed data for DIM tables
{
	mColumnCount = (unsigned int)nameRef.size();
	FILE* f;
	string f1;
	unsigned int cnt;
	char buffer[4000];
	string str;
	not_compressed = 0;
	mRecCount = Recs;
	hostRecCount = Recs;
	totalRecs = Recs;
	load_file_name = file_name;

	f1 = file_name + ".sort";
	f = fopen (f1.c_str() , "rb" );
	if(f) {
		unsigned int sz, idx;
		fread((char *)&sz, 4, 1, f);
		for(unsigned int j = 0; j < sz; j++) {
			fread((char *)&idx, 4, 1, f);
			fread(buffer, idx, 1, f);
			str.assign(buffer, idx);
			sorted_fields.push(str);
			if(verbose)
				cout << "segment sorted on " << str << endl;
		};
		fclose(f);
	};

	f1 = file_name + ".presort";
	f = fopen (f1.c_str() , "rb" );
	if(f) {
		unsigned int sz, idx;
		fread((char *)&sz, 4, 1, f);
		for(unsigned int j = 0; j < sz; j++) {
			fread((char *)&idx, 4, 1, f);
			fread(buffer, idx, 1, f);
			str.assign(buffer, idx);
			presorted_fields.push(str);
			if(verbose)
				cout << "presorted on " << str << endl;
		};
		fclose(f);
	};

	tmp_table = 0;
	filtered = 0;

	for(unsigned int i=0; i < mColumnCount; i++) {

		//f1 = file_name + "." + nameRef.front() + ".0";
		//f = fopen (f1.c_str() , "rb" );
		//fread((char *)&bytes, 4, 1, f); //need to read metadata such as type and length
		//fclose(f);

		columnNames.push_back(nameRef.front());
		cols[colsRef.front()] = nameRef.front();

		if (((typeRef.front()).compare("decimal") == 0) || ((typeRef.front()).compare("int") == 0)) {
			f1 = file_name + "." + nameRef.front() + ".0";
			f = fopen (f1.c_str() , "rb" );
			if(!f) {
				cout << "Couldn't find field " << nameRef.front() << endl;
				exit(0);
			};
			for(unsigned int j = 0; j < 6; j++)
				fread((char *)&cnt, 4, 1, f);
			fclose(f);
			compTypes[nameRef.front()] = cnt;
		};
		if((typeRef.front()).compare("timestamp") == 0)
			ts_cols[nameRef.front()] = 1;
		else
			ts_cols[nameRef.front()] = 0;


		if ((typeRef.front()).compare("int") == 0 || (typeRef.front()).compare("timestamp") == 0) {
			type[nameRef.front()] = 0;
			decimal[nameRef.front()] = 0;
			decimal_zeroes[nameRef.front()] = 0;
			h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
			d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
		}
		else
			if ((typeRef.front()).compare("float") == 0) {
				type[nameRef.front()] = 1;
				decimal[nameRef.front()] = 0;
				h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
				d_columns_float[nameRef.front()] = thrust::device_vector<float_type >();
			}
			else
				if ((typeRef.front()).compare("decimal") == 0) {
					type[nameRef.front()] = 0;
					decimal[nameRef.front()] = 1;
					decimal_zeroes[nameRef.front()] = sizeRef.front();
					h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
					d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
				}
				else {
					type[nameRef.front()] = 2;
					decimal[nameRef.front()] = 0;
					h_columns_char[nameRef.front()] = nullptr;
					d_columns_char[nameRef.front()] = nullptr;
					char_size[nameRef.front()] = sizeRef.front();
					string_map[nameRef.front()] = file_name + "." + nameRef.front();
				};
		nameRef.pop();
		typeRef.pop();
		sizeRef.pop();
		colsRef.pop();
	};
};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs)
{
	mColumnCount = (unsigned int)nameRef.size();
	tmp_table = 0;
	filtered = 0;
	mRecCount = 0;
	hostRecCount = Recs;
	segCount = 0;

	for(unsigned int i=0; i < mColumnCount; i++) {

		columnNames.push_back(nameRef.front());
		cols[colsRef.front()] = nameRef.front();

		if((typeRef.front()).compare("timestamp") == 0)
			ts_cols[nameRef.front()] = 1;
		else
			ts_cols[nameRef.front()] = 0;


		if ((typeRef.front()).compare("int") == 0 || (typeRef.front()).compare("timestamp") == 0) {
			type[nameRef.front()] = 0;
			decimal[nameRef.front()] = 0;
			decimal_zeroes[nameRef.front()] = 0;
			h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
			d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
		}
		else
			if ((typeRef.front()).compare("float") == 0) {
				type[nameRef.front()] = 1;
				decimal[nameRef.front()] = 0;
				h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
				d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
			}
			else
				if ((typeRef.front()).compare("decimal") == 0) {
					type[nameRef.front()] = 0;
					decimal[nameRef.front()] = 1;
					decimal_zeroes[nameRef.front()] = sizeRef.front();
					h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
					d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
				}
				else {
					type[nameRef.front()] = 2;
					decimal[nameRef.front()] = 0;
					h_columns_char[nameRef.front()] = nullptr;
					d_columns_char[nameRef.front()] = nullptr;
					char_size[nameRef.front()] = sizeRef.front();
				};
		nameRef.pop();
		typeRef.pop();
		sizeRef.pop();
		colsRef.pop();
	};
};

void CudaSet::initialize(const size_t RecordCount, const unsigned int ColumnCount)
{
	mRecCount = RecordCount;
	hostRecCount = RecordCount;
	mColumnCount = ColumnCount;
	filtered = 0;
};


void CudaSet::initialize(queue<string> op_sel, const queue<string> op_sel_as)
{
	mRecCount = 0;
	mColumnCount = (unsigned int)op_sel.size();
	segCount = 1;
	not_compressed = 1;
	filtered = 0;
	col_aliases = op_sel_as;
	unsigned int i = 0;
	CudaSet *a;
	while(!op_sel.empty()) {
		for(auto it = varNames.begin(); it != varNames.end(); it++) {
			a = it->second;
			if(std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) != a->columnNames.end())
				break;
		};

		type[op_sel.front()] = a->type[op_sel.front()];
		cols[i] = op_sel.front();
		decimal[op_sel.front()] = a->decimal[op_sel.front()];
		decimal_zeroes[op_sel.front()] = a->decimal_zeroes[op_sel.front()];
		columnNames.push_back(op_sel.front());

		if (a->type[op_sel.front()] == 0)  {
			d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
			//h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
			h_columns_int[op_sel.front()] = thrust::host_vector<int_type>();
		}
		else
			if (a->type[op_sel.front()] == 1) {
				d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
				//h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
				h_columns_float[op_sel.front()] = thrust::host_vector<float_type>();
			}
			else {
				h_columns_char[op_sel.front()] = nullptr;
				d_columns_char[op_sel.front()] = nullptr;
				char_size[op_sel.front()] = a->char_size[op_sel.front()];
			};
		i++;
		op_sel.pop();
	};
}



void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
	mRecCount = 0;
	mColumnCount = 0;
	queue<string> q_cnt(op_sel);
	unsigned int i = 0;
	set<string> field_names;
	while(!q_cnt.empty()) {
		if( std::find(a->columnNames.begin(), a->columnNames.end(), q_cnt.front()) !=  a->columnNames.end() ||
		        std::find(b->columnNames.begin(), b->columnNames.end(), q_cnt.front()) !=  b->columnNames.end())  {
			field_names.insert(q_cnt.front());
		};
		q_cnt.pop();
	}
	mColumnCount = (unsigned int)field_names.size();
	maxRecs = b->maxRecs;
	segCount = 1;
	filtered = 0;
	not_compressed = 1;

	col_aliases = op_sel_as;
	i = 0;
	while(!op_sel.empty()) {
		if(std::find(columnNames.begin(), columnNames.end(), op_sel.front()) ==  columnNames.end()) {
			if(std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) !=  a->columnNames.end()) {
				cols[i] = op_sel.front();
				decimal[op_sel.front()] = a->decimal[op_sel.front()];
				columnNames.push_back(op_sel.front());
				type[op_sel.front()] = a->type[op_sel.front()];
				ts_cols[op_sel.front()] = a->ts_cols[op_sel.front()];

				if (a->type[op_sel.front()] == 0)  {
					d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
					h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
					if(a->string_map.find(op_sel.front()) != a->string_map.end()) {
						string_map[op_sel.front()] = a->string_map[op_sel.front()];
					};
					decimal[op_sel.front()] = a->decimal[op_sel.front()];
					decimal_zeroes[op_sel.front()] = a->decimal_zeroes[op_sel.front()];
				}
				else
					if (a->type[op_sel.front()] == 1) {
						d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
						h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
					}
					else {
						h_columns_char[op_sel.front()] = nullptr;
						d_columns_char[op_sel.front()] = nullptr;
						char_size[op_sel.front()] = a->char_size[op_sel.front()];
						string_map[op_sel.front()] = a->string_map[op_sel.front()];
					};
				i++;
			}
			else
				if(std::find(b->columnNames.begin(), b->columnNames.end(), op_sel.front()) !=  b->columnNames.end()) {
					columnNames.push_back(op_sel.front());
					cols[i] = op_sel.front();
					decimal[op_sel.front()] = b->decimal[op_sel.front()];
					type[op_sel.front()] = b->type[op_sel.front()];
					ts_cols[op_sel.front()] = b->ts_cols[op_sel.front()];

					if (b->type[op_sel.front()] == 0) {
						d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
						h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
						if(b->string_map.find(op_sel.front()) != b->string_map.end()) {
							string_map[op_sel.front()] = b->string_map[op_sel.front()];
						};
						decimal[op_sel.front()] = b->decimal[op_sel.front()];
						decimal_zeroes[op_sel.front()] = b->decimal_zeroes[op_sel.front()];
					}
					else
						if (b->type[op_sel.front()] == 1) {
							d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
							h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
						}
						else {
							h_columns_char[op_sel.front()] = nullptr;
							d_columns_char[op_sel.front()] = nullptr;
							char_size[op_sel.front()] = b->char_size[op_sel.front()];
							string_map[op_sel.front()] = b->string_map[op_sel.front()];
						};
					i++;
				}
		}
		op_sel.pop();
	};
};



int_type reverse_op(int_type op_type)
{
	if (op_type == 2) // >
		return 1;
	else
		if (op_type == 1)  // <
			return 2;
		else
			if (op_type == 6) // >=
				return 5;
			else
				if (op_type == 5)  // <=
					return 6;
				else
					return op_type;
}


size_t getFreeMem()
{
	size_t available, total;
	hipMemGetInfo(&available, &total);
	return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
	if(a->filtered) {
		CudaSet* t;
		if(a->filtered)
			t = varNames[a->source_name];
		else
			t = a;

		if(int_size*t->maxRecs > alloced_sz) {
			if(alloced_sz) {
				hipFree(alloced_tmp);
			};
			hipMalloc((void **) &alloced_tmp, int_size*t->maxRecs);
			alloced_sz = int_size*t->maxRecs;
		}
	}
	else {
		while(!fields.empty()) {
			if(var_exists(a, fields.front()) && !a->onDevice(fields.front())) {
				a->allocColumnOnDevice(fields.front(), a->maxRecs);
			}
			fields.pop();
		};
	};
}



void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, size_t& count)
{
	if(!a->onDevice(field)) {
		a->allocColumnOnDevice(field, a->maxRecs);
	};
	if(a->prm_index == 'R') {
		mygather(field, a, t, count, a->mRecCount);
	}
	else {
		mycopy(field, a, t, count, t->mRecCount);
		a->mRecCount = t->mRecCount;
	};
}


void copyFinalize(CudaSet* a, queue<string> fields, bool ts)
{
	set<string> uniques;
	if(scratch.size() < a->mRecCount*8)
		scratch.resize(a->mRecCount*8);
	thrust::device_ptr<int_type> tmp((int_type*)thrust::raw_pointer_cast(scratch.data()));

	while(!fields.empty()) {
		if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()) && cpy_bits.find(fields.front()) != cpy_bits.end() && (!a->ts_cols[fields.front()] || ts))	{

			if(cpy_bits[fields.front()] == 8) {
				if(a->type[fields.front()] != 1) {
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned char>());
				}
				else {
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
					thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned char>());
				};
			}
			else
				if(cpy_bits[fields.front()] == 16) {
					if(a->type[fields.front()] != 1) {
						thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
						thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned short>());
					}
					else {
						thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
						thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned short>());
					};
				}
				else
					if(cpy_bits[fields.front()] == 32) {
						if(a->type[fields.front()] != 1) {
							thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
							thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned int>());
						}
						else {
							thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
							thrust::transform(src, src+a->mRecCount, tmp, to_int64<unsigned int>());
						};
					}
					else {
						if(a->type[fields.front()] != 1) {
							thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_int[fields.front()].data()));
							thrust::copy(src, src+a->mRecCount, tmp);
						}
						else {
							thrust::device_ptr<int_type> src((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
							thrust::copy(src, src+a->mRecCount, tmp);
						};
					};
			thrust::constant_iterator<int_type> iter(cpy_init_val[fields.front()]);
			if(a->type[fields.front()] != 1) {
				thrust::transform(tmp, tmp + a->mRecCount, iter, a->d_columns_int[fields.front()].begin(), thrust::plus<int_type>());
			}
			else {
				thrust::device_ptr<int_type> dest((int_type*)thrust::raw_pointer_cast(a->d_columns_float[fields.front()].data()));
				thrust::transform(tmp, tmp + a->mRecCount, iter, dest, thrust::plus<int_type>());
				thrust::transform(dest, dest+a->mRecCount, a->d_columns_float[fields.front()].begin(), long_to_float());
			};
		};
		uniques.insert(fields.front());
		fields.pop();
	};
}


void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt)
{
	//std::clock_t start1 = std::clock();
	set<string> uniques;
	if(a->filtered) { //filter the segment
		if(flt) {
			filter_op(a->fil_s, a->fil_f, segment);
		};
		if(rsz && a->mRecCount) {
			queue<string> fields1(fields);
			while(!fields1.empty()) {
				a->resizeDeviceColumn(a->devRecCount + a->mRecCount, fields1.front());
				fields1.pop();
			};
			a->devRecCount = a->devRecCount + a->mRecCount;
		};
	};
	cpy_bits.clear();
	cpy_init_val.clear();
	auto f(fields);

	while(!fields.empty()) {
		if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{
			if(a->filtered) {
				if(a->mRecCount) {
					CudaSet *t = varNames[a->source_name];
					alloced_switch = 1;
					t->CopyColumnToGpu(fields.front(), segment);
					gatherColumns(a, t, fields.front(), segment, count);
					alloced_switch = 0;
				};
			}
			else {
				if(a->mRecCount) {
					a->CopyColumnToGpu(fields.front(), segment, count);
				};
			};
			uniques.insert(fields.front());
		};
		fields.pop();
	};
	//std::cout<< "copy time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void mygather(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
	if(t->type[colname] != 1 ) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
				thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
				thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
				thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else
				if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
				}
				else
					if(cpy_bits[colname] == 32) {
						thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
						thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
						thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
					}
					else
						if(cpy_bits[colname] == 64) {
							thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
							thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);
						};
		}
		else {
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_int[colname].begin() + offset);
		};

	}
	else  {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
				thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
				thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
				thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
			}
			else
				if(cpy_bits[colname] == 16) {
					thrust::device_ptr<unsigned short int> d_col_source((unsigned short int*)alloced_tmp);
					thrust::device_ptr<unsigned short int> d_col_dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
					thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
				}
				else
					if(cpy_bits[colname] == 32) {
						thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
						thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
						thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col_source, d_col_dest + offset);
					}
					else
						if(cpy_bits[colname] == 64) {
							thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
							thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
						};
		}
		else {
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size, d_col, a->d_columns_float[colname].begin() + offset);
		};
	}
};


void mycopy(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
	if(t->type[colname] != 1) {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
				thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
				thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
				thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else
				if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
				}
				else
					if(cpy_bits[colname] == 32) {
						thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
						thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[colname].data()));
						thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
					}
					else
						if(cpy_bits[colname] == 64) {
							thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
							thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_int[colname].begin() + offset);
						};
		}
		else {
			thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size, a->d_columns_int[colname].begin() + offset);
		};
	}
	else {
		if(cpy_bits.find(colname) != cpy_bits.end()) { // non-delta compression
			if(cpy_bits[colname] == 8) {
				thrust::device_ptr<unsigned char> d_col_source((unsigned char*)alloced_tmp);
				thrust::device_ptr<unsigned char> d_col_dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
				thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
			}
			else
				if(cpy_bits[colname] == 16) {
					thrust::device_ptr<short int> d_col_source((short int*)alloced_tmp);
					thrust::device_ptr<short int> d_col_dest((short int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()+offset));
					thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
				}
				else
					if(cpy_bits[colname] == 32) {
						thrust::device_ptr<unsigned int> d_col_source((unsigned int*)alloced_tmp);
						thrust::device_ptr<unsigned int> d_col_dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_float[colname].data()));
						thrust::copy(d_col_source, d_col_source + g_size, d_col_dest + offset);
					}
					else
						if(cpy_bits[colname] == 64) {
							thrust::device_ptr<int_type> d_col_source((int_type*)alloced_tmp);
							thrust::copy(d_col_source, d_col_source + g_size, a->d_columns_float[colname].begin() + offset);
						};
		}
		else {
			thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
			thrust::copy(d_col, d_col + g_size,	a->d_columns_float[colname].begin() + offset);
		};
	};
};



size_t load_queue(queue<string> c1, CudaSet* right, string f2, size_t &rcount,
                  unsigned int start_segment, unsigned int end_segment, bool rsz, bool flt)
{
	queue<string> cc;
	while(!c1.empty()) {
		if(std::find(right->columnNames.begin(), right->columnNames.end(), c1.front()) !=  right->columnNames.end()) {
			if(f2 != c1.front() ) {
				cc.push(c1.front());
			};
		};
		c1.pop();
	};
	if(std::find(right->columnNames.begin(), right->columnNames.end(), f2) !=  right->columnNames.end()) {
		cc.push(f2);
	};

	if(right->filtered) {
		allocColumns(right, cc);
	};

	rcount = right->maxRecs;
	queue<string> ct(cc);

	while(!ct.empty()) {
		if(right->filtered && rsz) {
			right->mRecCount = 0;
		}
		else {
			right->allocColumnOnDevice(ct.front(), rcount*right->segCount);
		};
		ct.pop();
	};

	size_t cnt_r = 0;
	right->devRecCount = 0;
	for(unsigned int i = start_segment; i < end_segment; i++) {
		if(!right->filtered)
			copyColumns(right, cc, i, cnt_r, rsz, 0);
		else
			copyColumns(right, cc, i, cnt_r, rsz, flt);
		cnt_r = cnt_r + right->mRecCount;
	};

	right->mRecCount = cnt_r;
	return cnt_r;

}

size_t max_char(CudaSet* a)
{
	size_t max_char1 = 8;
	for(unsigned int i = 0; i < a->columnNames.size(); i++) {
		if(a->type[a->columnNames[i]] == 2) {
			if (a->char_size[a->columnNames[i]] > max_char1)
				max_char1 = a->char_size[a->columnNames[i]];
		}
		else
			if(a->type[a->columnNames[i]] == 0 && a->string_map.find(a->columnNames[i]) != a->string_map.end()) {
				auto s = a->string_map[a->columnNames[i]];
				auto pos = s.find_first_of(".");
				auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;
				if (len > max_char1)
					max_char1 = len;
			};
	};
	return max_char1;
};


size_t max_char(CudaSet* a, queue<string> field_names)
{
	size_t max_char = 8;
	while (!field_names.empty()) {
		if (a->type[field_names.front()] == 2) {
			if (a->char_size[field_names.front()] > max_char)
				max_char = a->char_size[field_names.front()];
		};
		field_names.pop();
	};
	return max_char;
};


void setSegments(CudaSet* a, queue<string> cols)
{
	size_t mem_available = getFreeMem();
	size_t tot_sz = 0;
	while(!cols.empty()) {
		if(a->type[cols.front()] != 2)
			tot_sz = tot_sz + int_size;
		else
			tot_sz = tot_sz + a->char_size[cols.front()];
		cols.pop();
	};
	if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
		a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;
		a->maxRecs = (a->mRecCount/a->segCount)+1;
	};
};


void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{
	str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

	if (SortType.compare("DESC") == 0 )
		str_sort_host(tmp, RecCount, permutation, 1, len);
	else
		str_sort_host(tmp, RecCount, permutation, 0, len);
}


void apply_permutation_char(char* key, unsigned int* permutation, size_t RecCount, char* tmp, unsigned int len)
{
	// copy keys to temporary vector
	hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
	// permute the keys
	str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len)
{
	str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}


void filter_op(const char *s, const char *f, unsigned int segment)
{
	CudaSet *a, *b;

	a = varNames.find(f)->second;
	a->name = f;
	//std::clock_t start1 = std::clock();

	if(a->mRecCount == 0 && !a->filtered) {
		b = new CudaSet(0,1);
	}
	else {
		if(verbose)
			cout << "FILTER " << s << " " << f << " " << getFreeMem() << '\xd';

		b = varNames[s];
		b->name = s;
		b->string_map = a->string_map;
		size_t cnt = 0;
		b->sorted_fields = a->sorted_fields;
		b->ts_cols = a->ts_cols;
		allocColumns(a, b->fil_value);

		if (b->prm_d.size() == 0) {
			b->prm_d.resize(a->maxRecs);
		};

		cout << endl << "MAP CHECK start " << segment <<  endl;
		char map_check = zone_map_check(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, b->fil_nums_precision, a, segment);
		cout << endl << "MAP CHECK segment " << segment << " " << map_check <<  endl;

		if(map_check == 'R') {
			auto old_ph = phase_copy;
			phase_copy = 0;
			copyColumns(a, b->fil_value, segment, cnt);
			phase_copy = old_ph;
			bool* res = filter(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, b->fil_nums_precision, a, segment);
			thrust::device_ptr<bool> bp((bool*)res);
			b->prm_index = 'R';
			b->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 1);
			thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
			                bp, b->prm_d.begin(), thrust::identity<bool>());

			hipFree(res);
		}
		else  {
			b->prm_index = map_check;
			if(map_check == 'A')
				b->mRecCount = a->mRecCount;
			else
				b->mRecCount = 0;
		};
		if(segment == a->segCount-1)
			a->deAllocOnDevice();
	}
	if(verbose)
		cout << endl << "filter result " << b->mRecCount << endl;
}




size_t load_right(CudaSet* right, string f2, queue<string> op_g, queue<string> op_alt, size_t& rcount, unsigned int start_seg, unsigned int end_seg) {

	size_t cnt_r = 0;
	//if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

	// need to allocate all right columns
	if(right->not_compressed) {
		queue<string> op_alt1;
		op_alt1.push(f2);
		cnt_r = load_queue(op_alt1, right, "", rcount, start_seg, end_seg, 1, 1);

		queue<string> op_alt2;
		while(!op_alt.empty()) {
			if(f2.compare(op_alt.front())) {
				if (std::find(right->columnNames.begin(), right->columnNames.end(), op_alt.front()) != right->columnNames.end()) {
					op_alt2.push(op_alt.front());
				};
			};
			op_alt.pop();
		};
		if(!op_alt2.empty())
			cnt_r = load_queue(op_alt2, right, "", rcount, start_seg, end_seg, 0, 0);
	}
	else {
		cnt_r = load_queue(op_alt, right, f2, rcount, start_seg, end_seg, 1, 1);
	};

	return cnt_r;
};



void insert_records(const char* f, const char* s) {
	char buf[4096];
	size_t size, maxRecs, cnt = 0;
	string str_s, str_d;

	if(varNames.find(s) == varNames.end()) {
		process_error(3, "couldn't find " + string(s) );
	};
	CudaSet *a;
	a = varNames.find(s)->second;
	a->name = s;

	if(varNames.find(f) == varNames.end()) {
		process_error(3, "couldn't find " + string(f) );
	};

	CudaSet *b;
	b = varNames.find(f)->second;
	b->name = f;

	// if both source and destination are on disk
	cout << "SOURCES " << a->source << ":" << b->source << endl;
	if(a->source && b->source) {
		for(unsigned int i = 0; i < a->segCount; i++) {
			for(unsigned int z = 0; z < a->columnNames.size(); z++) {

				if(a->type[a->columnNames[z]] != 2) {
					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
					str_d = b->load_file_name + "." + a->columnNames[z] + "." + to_string(b->segCount + i);
					cout << str_s << " " << str_d << endl;
					FILE* source = fopen(str_s.c_str(), "rb");
					FILE* dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);
				}
				else { //merge strings
					//read b's strings
					str_s = b->load_file_name + "." + b->columnNames[z];
					FILE* dest = fopen(str_s.c_str(), "rb");
					auto len = b->char_size[b->columnNames[z]];
					map<string, unsigned long long int> map_d;
					buf[len] = 0;
					unsigned long long cnt = 0;
					while (fread(buf, len, 1, dest)) {
						map_d[buf] = cnt;
						cnt++;
					};
					fclose(dest);
					unsigned long long int cct = cnt;

					str_s = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i) + ".hash";
					str_d = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".hash";
					FILE* source = fopen(str_s.c_str(), "rb");
					dest = fopen(str_d.c_str(), "wb");
					while (size = fread(buf, 1, BUFSIZ, source)) {
						fwrite(buf, 1, size, dest);
					}
					fclose(source);
					fclose(dest);

					str_s = a->load_file_name + "." + a->columnNames[z];
					source = fopen(str_s.c_str(), "rb");
					map<unsigned long long int, string> map_s;
					buf[len] = 0;
					cnt = 0;
					while (fread(buf, len, 1, source)) {
						map_s[cnt] = buf;
						cnt++;
					};
					fclose(source);

					queue<string> op_vx;
					op_vx.push(a->columnNames[z]);
					allocColumns(a, op_vx);
					a->resize(a->maxRecs);
					a->CopyColumnToGpu(a->columnNames[z], z, 0);
					a->CopyColumnToHost(a->columnNames[z]);

					str_d = b->load_file_name + "." + b->columnNames[z];
					fstream f_file;
					f_file.open(str_d.c_str(), ios::out|ios::app|ios::binary);

					for(auto j = 0; j < a->mRecCount; j++) {
						auto ss = map_s[a->h_columns_int[a->columnNames[z]][j]];
						if(map_d.find(ss) == map_d.end()) { //add
							f_file.write((char *)ss.c_str(), len);
							a->h_columns_int[a->columnNames[z]][j] = cct;
							cct++;
						}
						else {
							a->h_columns_int[a->columnNames[z]][j] = map_d[ss];
						};
					};
					f_file.close();

					thrust::device_vector<int_type> d_col(a->mRecCount);
					thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, d_col.begin());
					auto i_name = b->load_file_name + "." + b->columnNames[z] + "." + to_string(b->segCount + i) + ".idx";
					pfor_compress(thrust::raw_pointer_cast(d_col.data()), a->mRecCount*int_size, i_name, a->h_columns_int[a->columnNames[z]], 0);
				};
			};
		};

		if(a->maxRecs > b->maxRecs)
			maxRecs = a->maxRecs;
		else
			maxRecs = b->maxRecs;

		for(unsigned int i = 0; i < b->columnNames.size(); i++) {
			b->reWriteHeader(b->load_file_name, b->columnNames[i], a->segCount + b->segCount, a->totalRecs + b->totalRecs, maxRecs);
		};
	}
	else
		if(!a->source && !b->source) { //if both source and destination are in memory
			size_t oldCount = b->mRecCount;
			b->resize(a->mRecCount);
			for(unsigned int z = 0; z< b->mColumnCount; z++) {
				if(b->type[a->columnNames[z]] == 0) {
					thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_int[b->columnNames[z]].begin() + oldCount);
				}
				else
					if(b->type[a->columnNames[z]] == 1) {
						thrust::copy(a->h_columns_float[a->columnNames[z]].begin(), a->h_columns_float[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_float[b->columnNames[z]].begin() + oldCount);
					}
					else {
						hipMemcpy(b->h_columns_char[b->columnNames[z]] + b->char_size[b->columnNames[z]]*oldCount, a->h_columns_char[a->columnNames[z]], a->char_size[a->columnNames[z]]*a->mRecCount, hipMemcpyHostToHost);
					};
			};
		}
		else
			if(!a->source && b->source) {

				total_segments = b->segCount;
				total_count = b->mRecCount;
				total_max = b->maxRecs;;

				queue<string> op_vx;
				for(unsigned int i=0; i < a->columnNames.size(); i++)
					op_vx.push(a->columnNames[i]);

				allocColumns(a, op_vx);
				a->resize(a->maxRecs);
				for(unsigned int i = 0; i < a->segCount; i++) {
					if (a->filtered) {
						copyColumns(a, op_vx, i, cnt);
						a->CopyToHost(0, a->mRecCount);
					};
					a->compress(b->load_file_name, 0, 1, i - (a->segCount-1), a->mRecCount, 0);
				};
				for(unsigned int i = 0; i < b->columnNames.size(); i++) {
					b->writeHeader(b->load_file_name, b->columnNames[i], total_segments);
				};
			};
};



void delete_records(const char* f) {

	CudaSet *a;
	a = varNames.find(f)->second;
	a->name = f;
	size_t totalRemoved = 0;
	size_t maxRecs = 0;

	if(!a->keep) { // temporary variable
		process_error(2, "Delete operator is only applicable to disk based sets\nfor deleting records from derived sets please use filter operator ");
	}
	else {  // read matching segments, delete, compress and write on a disk replacing the original segments

		string str, str_old;
		queue<string> op_vx;
		size_t cnt;
		for ( auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
			op_vx.push((*it).first);
			if (std::find(a->columnNames.begin(), a->columnNames.end(), (*it).first) == a->columnNames.end()) {

				if ((*it).second.col_type == 0) {
					a->type[(*it).first] = 0;
					a->decimal[(*it).first] = 0;
					//a->h_columns_int[(*it).first] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
					a->h_columns_int[(*it).first] = thrust::host_vector<int_type>();
					a->d_columns_int[(*it).first] = thrust::device_vector<int_type>();
				}
				else
					if((*it).second.col_type == 1) {
						a->type[(*it).first] = 1;
						a->decimal[(*it).first] = 0;
						//a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
						a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
						a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
					}
					else
						if ((*it).second.col_type == 3) {
							a->type[(*it).first] = 1;
							a->decimal[(*it).first] = 1;
							//a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
							a->h_columns_float[(*it).first] = thrust::host_vector<float_type>();
							a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
						}
						else {
							a->type[(*it).first] = 2;
							a->decimal[(*it).first] = 0;
							a->h_columns_char[(*it).first] = nullptr;
							a->d_columns_char[(*it).first] = nullptr;
							a->char_size[(*it).first] = (*it).second.col_length;
						};
				a->columnNames.push_back((*it).first);
			}
		};

		allocColumns(a, op_vx);
		a->resize(a->maxRecs);
		a->prm_d.resize(a->maxRecs);
		size_t cc = a->mRecCount;
		size_t tmp;

		void* d;
		CUDA_SAFE_CALL(hipMalloc((void **) &d, a->maxRecs*float_size));
		unsigned int new_seg_count = 0;
		char map_check;

		for(unsigned int i = 0; i < a->segCount; i++) {

			map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, op_nums_precision, a, i);
			if(verbose)
				cout << "MAP CHECK segment " << i << " " << map_check <<  endl;
			if(map_check != 'N') {

				cnt = 0;
				copyColumns(a, op_vx, i, cnt);
				tmp = a->mRecCount;

				if(a->mRecCount) {
					bool* res = filter(op_type,op_value,op_nums, op_nums_f, op_nums_precision, a, i);
					thrust::device_ptr<bool> bp((bool*)res);
					thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
					                bp, a->prm_d.begin(), thrust::logical_not<bool>());

					a->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 0);
					hipFree(res);

//					cout << "Remained recs count " << a->mRecCount << endl;
					if(a->mRecCount > maxRecs)
						maxRecs = a->mRecCount;

					if (a->mRecCount) {

						totalRemoved = totalRemoved + (tmp - a->mRecCount);
						if (a->mRecCount == tmp) { //none deleted
							if(new_seg_count != i) {
								for (auto it=data_dict[a->load_file_name].begin() ; it != data_dict[a->load_file_name].end(); ++it ) {
									auto colname = (*it).first;
									str_old = a->load_file_name + "." + colname + "." + to_string(i);
									str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);
									remove(str.c_str());
									rename(str_old.c_str(), str.c_str());
								};
							};
							new_seg_count++;

						}
						else { //some deleted
							//cout << "writing segment " << new_seg_count << endl;

							map<string, col_data> s = data_dict[a->load_file_name];
							for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
								string colname = (*it).first;
								str = a->load_file_name + "." + colname + "." + to_string(new_seg_count);

								if(a->type[colname] == 0) {
									thrust::device_ptr<int_type> d_col((int_type*)d);
									thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
									pfor_compress( d, a->mRecCount*int_size, str, a->h_columns_int[colname], 0);
								}
								else
									if(a->type[colname] == 1) {
										thrust::device_ptr<float_type> d_col((float_type*)d);
										if(a->decimal[colname]) {
											thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
											thrust::device_ptr<long long int> d_col_dec((long long int*)d);
											thrust::transform(d_col,d_col+a->mRecCount, d_col_dec, float_to_long());
											pfor_compress( d, a->mRecCount*float_size, str, a->h_columns_float[colname], 1);
										}
										else {
											thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
											thrust::copy(d_col, d_col + a->mRecCount, a->h_columns_float[colname].begin());
											fstream binary_file(str.c_str(),ios::out|ios::binary);
											binary_file.write((char *)&a->mRecCount, 4);
											binary_file.write((char *)(a->h_columns_float[colname].data()),a->mRecCount*float_size);
											unsigned int comp_type = 3;
											binary_file.write((char *)&comp_type, 4);
											binary_file.close();

										};
									}
									else {
										thrust::device_ptr<int_type> d_col((int_type*)d);
										thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
										pfor_compress( d, a->mRecCount*int_size, str + ".hash", a->h_columns_int[colname], 0);
									};
							};
							new_seg_count++;
						};
					}
					else {
						totalRemoved = totalRemoved + tmp;
					};
				}
			}
			else {
				if(new_seg_count != i) {
					for(unsigned int z = 0; z < a->columnNames.size(); z++) {
						str_old = a->load_file_name + "." + a->columnNames[z] + "." + to_string(i);
						str = a->load_file_name + "." + a->columnNames[z] + "." + to_string(new_seg_count);
						remove(str.c_str());
						rename(str_old.c_str(), str.c_str());
					};
				};
				new_seg_count++;
				maxRecs	= a->maxRecs;
			};
		};

		if (new_seg_count < a->segCount) {
			for(unsigned int i = new_seg_count; i < a->segCount; i++) {
				//cout << "delete segment " << i << endl;
				for(unsigned int z = 0; z < a->columnNames.size(); z++) {
					str = a->load_file_name + "." + a->columnNames[z];
					str += "." + to_string(i);
					remove(str.c_str());
				};
			};
		};

		for(unsigned int i = new_seg_count; i < a->segCount; i++) {
			a->reWriteHeader(a->load_file_name, a->columnNames[i], new_seg_count, a->totalRecs-totalRemoved, maxRecs);
		};


		a->mRecCount = cc;
		a->prm_d.resize(0);
		a->segCount = new_seg_count;
		a->deAllocOnDevice();
		hipFree(d);
	};


};


void save_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
	size_t str_len;
	fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
	size_t len = data_dict.size();
	binary_file.write((char *)&len, 8);
	for (auto it=data_dict.begin() ; it != data_dict.end(); ++it ) {
		str_len = (*it).first.size();
		binary_file.write((char *)&str_len, 8);
		binary_file.write((char *)(*it).first.data(), str_len);
		map<string, col_data> s = (*it).second;
		size_t len1 = s.size();
		binary_file.write((char *)&len1, 8);

		for (auto sit=s.begin() ; sit != s.end(); ++sit ) {
			str_len = (*sit).first.size();
			binary_file.write((char *)&str_len, 8);
			binary_file.write((char *)(*sit).first.data(), str_len);
			binary_file.write((char *)&(*sit).second.col_type, 4);
			binary_file.write((char *)&(*sit).second.col_length, 4);
		};
	};
	binary_file.close();
}

void load_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
	size_t str_len, recs, len1;
	string str1, str2;
	char buffer[4000];
	unsigned int col_type, col_length;
	fstream binary_file;
	binary_file.open(file_name.c_str(),ios::in|ios::binary);
	if(binary_file.is_open()) {
		binary_file.read((char*)&recs, 8);
		for(unsigned int i = 0; i < recs; i++) {
			binary_file.read((char*)&str_len, 8);
			binary_file.read(buffer, str_len);
			str1.assign(buffer, str_len);
			binary_file.read((char*)&len1, 8);

			for(unsigned int j = 0; j < len1; j++) {
				binary_file.read((char*)&str_len, 8);
				binary_file.read(buffer, str_len);
				str2.assign(buffer, str_len);
				binary_file.read((char*)&col_type, 4);
				binary_file.read((char*)&col_length, 4);
				data_dict[str1][str2].col_type = col_type;
				data_dict[str1][str2].col_length = col_length;
				//cout << "data DICT " << str1 << " " << str2 << " " << col_type << " " << col_length << endl;
			};
		};
		binary_file.close();
	}
	else {
		cout << "Couldn't open data dictionary" << endl;
	};
}

bool var_exists(CudaSet* a, string name) {

	if(std::find(a->columnNames.begin(), a->columnNames.end(), name) !=  a->columnNames.end())
		return 1;
	else

		return 0;
}

int file_exist (const char *filename)
{
	std::ifstream infile(filename);
	return infile.good();
}

bool check_bitmap_file_exist(CudaSet* left, CudaSet* right)
{
	queue<string> cols(right->fil_value);
	bool bitmaps_exist = 1;

	if(cols.size() == 0) {
		bitmaps_exist = 0;
	};
	while(cols.size() ) {
		if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
			string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
			if( !file_exist(fname.c_str())) {
				bitmaps_exist = 0;
			};
		};
		cols.pop();
	};
	return bitmaps_exist;
}

bool check_bitmaps_exist(CudaSet* left, CudaSet* right)
{
	//check if there are join bitmap indexes
	queue<string> cols(right->fil_value);
	bool bitmaps_exist = 1;

	if(cols.size() == 0) {
		bitmaps_exist = 1;
		return 1;
	};
	while(cols.size() ) {
		if (std::find(right->columnNames.begin(), right->columnNames.end(), cols.front()) != right->columnNames.end()) {
			string fname = left->load_file_name + "."  + right->load_file_name + "." + cols.front() + ".0";
			if( !file_exist(fname.c_str())) {
				bitmaps_exist = 0;
			};
		};
		cols.pop();
	};
	if(bitmaps_exist) {
		while(!right->fil_nums.empty() ) {
			left->fil_nums.push(right->fil_nums.front());
			right->fil_nums.pop();
		};
		while(!right->fil_nums_precision.empty() ) {
			left->fil_nums_precision.push(right->fil_nums_precision.front());
			right->fil_nums_precision.pop();
		};
		while(!right->fil_nums_f.empty() ) {
			left->fil_nums_f.push(right->fil_nums_f.front());
			right->fil_nums_f.pop();
		};
		while(!right->fil_value.empty() ) {
			if (std::find(right->columnNames.begin(), right->columnNames.end(), right->fil_value.front()) != right->columnNames.end()) {
				string fname = left->load_file_name + "."  + right->load_file_name + "." + right->fil_value.front();
				left->fil_value.push(fname);
			}
			else
				left->fil_value.push(right->fil_value.front());
			right->fil_value.pop();
		};
		bool add_and = 1;
		if(left->fil_type.empty())
			add_and = 0;
		while(!right->fil_type.empty() ) {
			left->fil_type.push(right->fil_type.front());
			right->fil_type.pop();
		};
		if(add_and) {
			left->fil_type.push("AND");
		};
		return 1;
	}
	else {
		return 0;
	};
}


void check_sort(const string str, const char* rtable, const char* rid)
{
	CudaSet* right = varNames.find(rtable)->second;
	fstream binary_file(str.c_str(),ios::out|ios::binary|ios::app);
	binary_file.write((char *)&right->sort_check, 1);
	binary_file.close();
}

void update_char_permutation(CudaSet* a, string colname, unsigned int* raw_ptr, string ord, void* temp, bool host)
{
	auto s = a->string_map[colname];
	auto pos = s.find_first_of(".");
	auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

	a->h_columns_char[colname] = new char[a->mRecCount*len];
	memset(a->h_columns_char[colname], 0, a->mRecCount*len);

	thrust::device_ptr<unsigned int> perm(raw_ptr);
	thrust::device_ptr<int_type> temp_int((int_type*)temp);
	thrust::gather(perm, perm+a->mRecCount, a->d_columns_int[colname].begin(), temp_int);

	//for(int z = 0 ; z < a->mRecCount; z++) {
	//cout << "Init vals " << a->d_columns_int[colname][z] << " " << perm[z] << " " << temp_int[z] << endl;
	//};

	//cout << "sz " << a->h_columns_int[colname].size() << " " << a->d_columns_int[colname].size() <<  " " << len << endl;
	hipMemcpy(thrust::raw_pointer_cast(a->h_columns_int[colname].data()), temp, 8*a->mRecCount, hipMemcpyDeviceToHost);

	FILE *f;
	f = fopen(a->string_map[colname].c_str(), "rb");

	for(int z = 0 ; z < a->mRecCount; z++) {
		fseek(f, a->h_columns_int[colname][z] * len, SEEK_SET);
		fread(a->h_columns_char[colname] + z*len, 1, len, f);
	};
	fclose(f);

	if(!host) {
		void *d;
		hipMalloc((void **) &d, a->mRecCount*len);
		a->d_columns_char[colname] = (char*)d;

		hipMemcpy(a->d_columns_char[colname], a->h_columns_char[colname], len*a->mRecCount, hipMemcpyHostToDevice);

		if (ord.compare("DESC") == 0 )
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort(a->d_columns_char[colname], a->mRecCount, raw_ptr, 0, len);

		hipFree(d);
	}
	else {
		if (ord.compare("DESC") == 0 )
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 1, len);
		else
			str_sort_host(a->h_columns_char[colname], a->mRecCount, raw_ptr, 0, len);
	};
}



void compress_int(const string file_name, const thrust::host_vector<int_type>& res)
{
	std::vector<unsigned int> dict_val;
	unsigned int bits_encoded;
	set<int_type> dict_s;
	map<int_type, unsigned int> d_ordered;

	for (unsigned int i = 0 ; i < res.size(); i++) {
		int_type f = res[i];
		dict_s.insert(f);
	};

	unsigned int i = 0;
	for (auto it = dict_s.begin(); it != dict_s.end(); it++) {
		d_ordered[*it] = i++;
	};

	for (unsigned int i = 0 ; i < res.size(); i++) {
		int_type f = res[i];
		dict_val.push_back(d_ordered[f]);
	};

	bits_encoded = (unsigned int)ceil(log2(double(d_ordered.size()+1)));
	//cout << "bits " << bits_encoded << endl;

	unsigned int sz = (unsigned int)d_ordered.size();
	// write to a file
	fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
	binary_file.write((char *)&sz, 4);

	for (auto it = d_ordered.begin(); it != d_ordered.end(); it++) {
		binary_file.write((char*)(&(it->first)), int_size);
	};

	unsigned int fit_count = 64/bits_encoded;
	unsigned long long int val = 0;
	binary_file.write((char *)&fit_count, 4);
	binary_file.write((char *)&bits_encoded, 4);
	unsigned int curr_cnt = 1;
	unsigned int vals_count = (unsigned int)dict_val.size()/fit_count;
	if(!vals_count || dict_val.size()%fit_count)
		vals_count++;
	binary_file.write((char *)&vals_count, 4);
	unsigned int real_count = (unsigned int)dict_val.size();
	binary_file.write((char *)&real_count, 4);

	for(unsigned int i = 0; i < dict_val.size(); i++) {

		val = val | dict_val[i];

		if(curr_cnt < fit_count)
			val = val << bits_encoded;

		if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
			if (curr_cnt < fit_count) {
				val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
			};
			curr_cnt = 1;
			binary_file.write((char *)&val, int_size);
			val = 0;
		}
		else
			curr_cnt = curr_cnt + 1;
	};
	binary_file.close();
};

int_type* get_vec(CudaSet* a, string s1_val, stack<int_type*>& exe_vectors) {
	int_type* t;
	if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end())
		t = a->get_int_by_name(s1_val);
	else {
		t = exe_vectors.top();
		exe_vectors.pop();
	}
	return t;
};

int_type* get_host_vec(CudaSet* a, string s1_val, stack<int_type*>& exe_vectors) {
	int_type* t;
	if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end()) {
		t = a->get_host_int_by_name(s1_val);
	}
	else {
		t = exe_vectors.top();


		thrust::device_ptr<int_type> st1((int_type*)t);
		for(int z = 0; z < 10; z++)
			cout << "RESVEC " << st1[z] << endl;

		exe_vectors.pop();
	}
	return t;
};


unsigned int get_decimals(CudaSet* a, string s1_val, stack<unsigned int>& exe_precision) {
	unsigned int t;
	if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end())
		t = a->decimal_zeroes[s1_val];
	else {
		t = exe_precision.top();
		exe_precision.pop();
	}
	return t;
};


#ifdef _WIN64
size_t getTotalSystemMemory()
{
	MEMORYSTATUSEX status;
	status.dwLength = sizeof(status);
	GlobalMemoryStatusEx(&status);
	return status.ullTotalPhys;
}
#else
size_t getTotalSystemMemory()
{
	long pages = sysconf(_SC_PHYS_PAGES);
	long page_size = sysconf(_SC_PAGE_SIZE);
	return pages * page_size;
}
#endif

